#include "hip/hip_runtime.h"
#define GLM_FORCE_CUDA
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <glm/glm.hpp>
#include "utilityCore.hpp"
#include "kernel.h"

// LOOK-2.1 potentially useful for doing grid-based neighbor search
#ifndef imax
#define imax( a, b ) ( ((a) > (b)) ? (a) : (b) )
#endif

#ifndef imin
#define imin( a, b ) ( ((a) < (b)) ? (a) : (b) )
#endif

#define checkCUDAErrorWithLine(msg) checkCUDAError(msg, __LINE__)

/**
* Check for CUDA errors; print and exit if there was a problem.
*/
void checkCUDAError(const char *msg, int line = -1) {
  hipError_t err = hipGetLastError();
  if (hipSuccess != err) {
    if (line >= 0) {
      fprintf(stderr, "Line %d: ", line);
    }
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}


/*****************
* Configuration *
*****************/

/*! Block size used for CUDA kernel launch. */
#define blockSize 128

// LOOK-1.2 Parameters for the boids algorithm.
// These worked well in our reference implementation.
#define rule1Distance 5.0f
#define rule2Distance 3.0f
#define rule3Distance 5.0f

#define rule1Scale 0.01f
#define rule2Scale 0.1f
#define rule3Scale 0.1f

#define maxSpeed 1.0f

/*! Size of the starting area in simulation space. */
#define scene_scale 100.0f

/***********************************************
* Kernel state (pointers are device pointers) *
***********************************************/

int numObjects;
dim3 threadsPerBlock(blockSize);

// LOOK-1.2 - These buffers are here to hold all your boid information.
// These get allocated for you in Boids::initSimulation.
// Consider why you would need two velocity buffers in a simulation where each
// boid cares about its neighbors' velocities.
// These are called ping-pong buffers.
glm::vec3 *dev_pos;
glm::vec3 *dev_vel1;
glm::vec3 *dev_vel2;

// LOOK-2.1 - these are NOT allocated for you. You'll have to set up the thrust
// pointers on your own too.

// For efficient sorting and the uniform grid. These should always be parallel.
int *dev_particleArrayIndices; // What index in dev_pos and dev_velX represents this particle?
int *dev_particleGridIndices; // What grid cell is this particle in?
// needed for use with thrust
thrust::device_ptr<int> dev_thrust_particleArrayIndices;
thrust::device_ptr<int> dev_thrust_particleGridIndices;

int *dev_gridCellStartIndices; // What part of dev_particleArrayIndices belongs
int *dev_gridCellEndIndices;   // to this cell?

// TODO-2.3 - consider what additional buffers you might need to reshuffle
// the position and velocity data to be coherent within cells.

glm::vec3* dev_coherentPos;
glm::vec3* dev_coherentVel;

// LOOK-2.1 - Grid parameters based on simulation parameters.
// These are automatically computed for you in Boids::initSimulation
int gridCellCount;
int gridSideCount;
float gridCellWidth;
float gridInverseCellWidth;
glm::vec3 gridMinimum;

/******************
* initSimulation *
******************/

__host__ __device__ unsigned int hash(unsigned int a) {
  a = (a + 0x7ed55d16) + (a << 12);
  a = (a ^ 0xc761c23c) ^ (a >> 19);
  a = (a + 0x165667b1) + (a << 5);
  a = (a + 0xd3a2646c) ^ (a << 9);
  a = (a + 0xfd7046c5) + (a << 3);
  a = (a ^ 0xb55a4f09) ^ (a >> 16);
  return a;
}

/**
* LOOK-1.2 - this is a typical helper function for a CUDA kernel.
* Function for generating a random vec3.
*/
__host__ __device__ glm::vec3 generateRandomVec3(float time, int index) {
  thrust::default_random_engine rng(hash((int)(index * time)));
  thrust::uniform_real_distribution<float> unitDistrib(-1, 1);

  return glm::vec3((float)unitDistrib(rng), (float)unitDistrib(rng), (float)unitDistrib(rng));
}

/**
* LOOK-1.2 - This is a basic CUDA kernel.
* CUDA kernel for generating boids with a specified mass randomly around the star.
*/
__global__ void kernGenerateRandomPosArray(int time, int N, glm::vec3 * arr, float scale) {
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (index < N) {
    glm::vec3 rand = generateRandomVec3(time, index);
    arr[index].x = scale * rand.x;
    arr[index].y = scale * rand.y;
    arr[index].z = scale * rand.z;
  }
}

/**
* Initialize memory, update some globals
*/
void Boids::initSimulation(int N) {
  numObjects = N;
  dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);

  // LOOK-1.2 - This is basic CUDA memory management and error checking.
  // Don't forget to hipFree in  Boids::endSimulation.
  hipMalloc((void**)&dev_pos, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_pos failed!");

  hipMalloc((void**)&dev_vel1, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_vel1 failed!");

  hipMalloc((void**)&dev_vel2, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_vel2 failed!");

  // LOOK-1.2 - This is a typical CUDA kernel invocation.
  kernGenerateRandomPosArray<<<fullBlocksPerGrid, blockSize>>>(1, numObjects,
    dev_pos, scene_scale);
  checkCUDAErrorWithLine("kernGenerateRandomPosArray failed!");

  // LOOK-2.1 computing grid params
  gridCellWidth = 2.0f * std::max(std::max(rule1Distance, rule2Distance), rule3Distance);
  int halfSideCount = (int)(scene_scale / gridCellWidth) + 1;
  gridSideCount = 2 * halfSideCount;

  gridCellCount = gridSideCount * gridSideCount * gridSideCount;
  gridInverseCellWidth = 1.0f / gridCellWidth;
  float halfGridWidth = gridCellWidth * halfSideCount;
  gridMinimum.x -= halfGridWidth;
  gridMinimum.y -= halfGridWidth;
  gridMinimum.z -= halfGridWidth;

  // TODO-2.1 TODO-2.3 - Allocate additional buffers here.

  hipMalloc((void**)&dev_particleArrayIndices, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_particleArrayIndices failed!");

  hipMalloc((void**)&dev_particleGridIndices, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_particleGridIndices failed!");

  hipMalloc((void**)&dev_gridCellStartIndices, gridCellCount * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_gridCellStartIndices failed!");

  hipMalloc((void**)&dev_gridCellEndIndices, gridCellCount * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_gridCellEndIndices failed!");

  hipMalloc((void**)&dev_coherentPos, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_coherentPos failed!");

  hipMalloc((void**)&dev_coherentVel, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_coherentVel failed!");


  hipDeviceSynchronize();
}


/******************
* copyBoidsToVBO *
******************/

/**
* Copy the boid positions into the VBO so that they can be drawn by OpenGL.
*/
__global__ void kernCopyPositionsToVBO(int N, glm::vec3 *pos, float *vbo, float s_scale) {
  int index = threadIdx.x + (blockIdx.x * blockDim.x);

  float c_scale = -1.0f / s_scale;

  if (index < N) {
    vbo[4 * index + 0] = pos[index].x * c_scale;
    vbo[4 * index + 1] = pos[index].y * c_scale;
    vbo[4 * index + 2] = pos[index].z * c_scale;
    vbo[4 * index + 3] = 1.0f;
  }
}

__global__ void kernCopyVelocitiesToVBO(int N, glm::vec3 *vel, float *vbo, float s_scale) {
  int index = threadIdx.x + (blockIdx.x * blockDim.x);

  if (index < N) {
    vbo[4 * index + 0] = vel[index].x + 0.3f;
    vbo[4 * index + 1] = vel[index].y + 0.3f;
    vbo[4 * index + 2] = vel[index].z + 0.3f;
    vbo[4 * index + 3] = 1.0f;
  }
}

/**
* Wrapper for call to the kernCopyboidsToVBO CUDA kernel.
*/
void Boids::copyBoidsToVBO(float *vbodptr_positions, float *vbodptr_velocities) {
  dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);

  kernCopyPositionsToVBO << <fullBlocksPerGrid, blockSize >> >(numObjects, dev_pos, vbodptr_positions, scene_scale);
  kernCopyVelocitiesToVBO << <fullBlocksPerGrid, blockSize >> >(numObjects, dev_vel1, vbodptr_velocities, scene_scale);

  checkCUDAErrorWithLine("copyBoidsToVBO failed!");

  hipDeviceSynchronize();
}


/******************
* stepSimulation *
******************/

/**
* LOOK-1.2 You can use this as a helper for kernUpdateVelocityBruteForce.
* __device__ code can be called from a __global__ context
* Compute the new velocity on the body with index `iSelf` due to the `N` boids
* in the `pos` and `vel` arrays.
*/
__device__ glm::vec3 computeVelocityChange(int N, int iSelf, const glm::vec3 *pos, const glm::vec3 *vel) {

	glm::vec3 result(0.0f);
	glm::vec3 perceived_center(0.0f), c( 0.0f), perceived_velocity(0.0f);

    int r1_num_neighbors = 0, r3_num_neighbors = 0;

	for (int i = 0; i < N; i++) {

		float distance = glm::distance(pos[i], pos[iSelf]);

		// Rule 1: boids fly towards their local perceived center of mass, which excludes themselves
		if (i != iSelf && distance < rule1Distance) {
			perceived_center += pos[i];
			r1_num_neighbors++;
		}

		// Rule 2: boids try to stay a distance d away from each other
		if (i != iSelf && distance < rule2Distance) {
			c -= (pos[i] - pos[iSelf]);
		}

		// Rule 3: boids try to match the speed of surrounding boids
		if (i != iSelf && distance < rule3Distance) {
			perceived_velocity += vel[i];
			r3_num_neighbors++;
		}
	}

	// Finalize Rule 1 and Rule 3
	if (r1_num_neighbors > 0) {
		perceived_center /= r1_num_neighbors;
		result += (perceived_center - pos[iSelf]) * rule1Scale;
	}

	if (r3_num_neighbors > 0) {
		perceived_velocity /= r3_num_neighbors;
		result += perceived_velocity * rule3Scale;
	}

	// Finalize Rule 2
	result += c * rule2Scale;

	return result;
}

/**
* TODO-1.2 implement basic flocking
* For each of the `N` bodies, update its position based on its current velocity.
*/
__global__ void kernUpdateVelocityBruteForce(int N, glm::vec3 *pos,
  glm::vec3 *vel1, glm::vec3 *vel2) {
  // Compute a new velocity based on pos and vel1
  // Clamp the speed
  // Record the new velocity into vel2. Question: why NOT vel1?
	int index = threadIdx.x + (blockIdx.x * blockDim.x);
	if (index < N) {
		glm::vec3 velocity = vel1[index] + computeVelocityChange(N, index, pos, vel1);
		float speed = glm::length(velocity);
		if (speed > maxSpeed) {
			velocity = glm::normalize(velocity) * maxSpeed;
		}
		vel2[index] = velocity;
	}
}

/**
* LOOK-1.2 Since this is pretty trivial, we implemented it for you.
* For each of the `N` bodies, update its position based on its current velocity.
*/
__global__ void kernUpdatePos(int N, float dt, glm::vec3 *pos, glm::vec3 *vel) {
  // Update position by velocity
  int index = threadIdx.x + (blockIdx.x * blockDim.x);
  if (index >= N) {
    return;
  }
  glm::vec3 thisPos = pos[index];
  thisPos += vel[index] * dt;

  // Wrap the boids around so we don't lose them
  thisPos.x = thisPos.x < -scene_scale ? scene_scale : thisPos.x;
  thisPos.y = thisPos.y < -scene_scale ? scene_scale : thisPos.y;
  thisPos.z = thisPos.z < -scene_scale ? scene_scale : thisPos.z;

  thisPos.x = thisPos.x > scene_scale ? -scene_scale : thisPos.x;
  thisPos.y = thisPos.y > scene_scale ? -scene_scale : thisPos.y;
  thisPos.z = thisPos.z > scene_scale ? -scene_scale : thisPos.z;

  pos[index] = thisPos;
}

// LOOK-2.1 Consider this method of computing a 1D index from a 3D grid index.
// LOOK-2.3 Looking at this method, what would be the most memory efficient
//          order for iterating over neighboring grid cells?
//          for(x)
//            for(y)
//             for(z)? Or some other order?
__device__ int gridIndex3Dto1D(int x, int y, int z, int gridResolution) {
  return x + y * gridResolution + z * gridResolution * gridResolution;
}

__global__ void kernComputeIndices(int N, int gridResolution,
  glm::vec3 gridMin, float inverseCellWidth,
  glm::vec3 *pos, int *indices, int *gridIndices) {
    // TODO-2.1
    // - Label each boid with the index of its grid cell.
    // - Set up a parallel array of integer indices as pointers to the actual
    //   boid data in pos and vel1/vel2
	int index = threadIdx.x + (blockIdx.x * blockDim.x);
	if (index < N) {
		indices[index] = index;
		glm::vec3 gridIndex3D = glm::floor((pos[index] - gridMin) * inverseCellWidth);
		gridIndices[index] = gridIndex3Dto1D(gridIndex3D.x, gridIndex3D.y, gridIndex3D.z, gridResolution);
	}
}

// LOOK-2.1 Consider how this could be useful for indicating that a cell
//          does not enclose any boids
__global__ void kernResetIntBuffer(int N, int *intBuffer, int value) {
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (index < N) {
    intBuffer[index] = value;
  }
}

__global__ void kernIdentifyCellStartEnd(int N, int *particleGridIndices,
  int *gridCellStartIndices, int *gridCellEndIndices) {
  // TODO-2.1
  // Identify the start point of each cell in the gridIndices array.
  // This is basically a parallel unrolling of a loop that goes
  // "this index doesn't match the one before it, must be a new cell!"

	int index = threadIdx.x + (blockIdx.x * blockDim.x);

	if (index < N) {
		// If at the start, set the start index as current
		if (index == 0) {
			gridCellStartIndices[particleGridIndices[index]] = index;
		}
		// If at the end, set the end index as current
		else if (index == N - 1) {
			gridCellEndIndices[particleGridIndices[index]] = index;
		}
		else {
			// If the current grid index is different from the previous one, set the end index 
			// of the previous grid index as the current index - 1 and the start index of the 
			// current grid index as the current index
			if (particleGridIndices[index] != particleGridIndices[index - 1]) {
				gridCellEndIndices[particleGridIndices[index - 1]] = index - 1;
				gridCellStartIndices[particleGridIndices[index]] = index;
			}
		}
	}

}

__global__ void kernReshuffleBuffer(int N, int* particleArrayIndices, glm::vec3* pos, glm::vec3* vel, glm::vec3* coherentPos, glm::vec3* coherentVel) {
	int index = threadIdx.x + (blockIdx.x * blockDim.x);
	if (index < N) {
		int coherentIndex = particleArrayIndices[index];
		coherentPos[index] = pos[coherentIndex];
		coherentVel[index] = vel[coherentIndex];
	}
}

__global__ void kernUpdateVelNeighborSearchScattered(
  int N, int gridResolution, glm::vec3 gridMin,
  float inverseCellWidth, float cellWidth,
  int *gridCellStartIndices, int *gridCellEndIndices,
  int *particleArrayIndices,
  glm::vec3 *pos, glm::vec3 *vel1, glm::vec3 *vel2) {
  // TODO-2.1 - Update a boid's velocity using the uniform grid to reduce
  // the number of boids that need to be checked.
  // - Identify the grid cell that this particle is in
  // - Identify which cells may contain neighbors. This isn't always 8.
  // - For each cell, read the start/end indices in the boid pointer array.
  // - Access each boid in the cell and compute velocity change from
  //   the boids rules, if this boid is within the neighborhood distance.
  // - Clamp the speed change before putting the new speed in vel2
	int index = threadIdx.x + (blockIdx.x * blockDim.x);
	if (index < N) {
		glm::vec3 perceived_center(0.0f), c(0.0f), perceived_velocity(0.0f), result(0.0f);
		int num_neighbors_rule1 = 0, num_neighbors_rule3 = 0;

		glm::vec3 position = pos[index], gridPos = glm::floor((position - gridMin) * inverseCellWidth);

		// Iterate over the 3x3x3 grid of cells around the current cell (upto 8 cells)
		for (int x = imax(gridPos.x - 1, 0); x < imin(gridPos.x + 1, gridResolution - 1); x++) {
			for (int y = imax(gridPos.y - 1, 0); y < imin(gridPos.y + 1, gridResolution - 1); y++) {
				for (int z = imax(gridPos.z - 1, 0); z < imin(gridPos.z + 1, gridResolution - 1); z++) {
					
					int currGridIndex = gridIndex3Dto1D(x, y, z, gridResolution);
					int currStartIndex = gridCellStartIndices[currGridIndex];
					int currEndIndex = gridCellEndIndices[currGridIndex];

					// If no boids are in the current cell, continue
					if (currStartIndex == -1 || currEndIndex == -1) {
						continue;
					}


					for (int i = currStartIndex; i <= currEndIndex; i++) {

						int boidIndex = particleArrayIndices[i];

						if (boidIndex != index) {
							float distance = glm::distance(pos[boidIndex], position);

							// Rule 1
							if (distance < rule1Distance) {
								perceived_center += pos[boidIndex];
								num_neighbors_rule1++;
							}
							// Rule 2
							if (distance < rule2Distance) {
								c -= (pos[boidIndex] - position);
							}
							// Rule 3
							if (distance < rule3Distance) {
								perceived_velocity += vel1[boidIndex];
								num_neighbors_rule3++;
							}
						}
					}
				}
			}
		}	

		// Finalize Rule 1 and Rule 3

		if (num_neighbors_rule1 > 0) {
			perceived_center /= num_neighbors_rule1;
			result += (perceived_center - position) * rule1Scale;
		}

		if (num_neighbors_rule3 > 0) {
			perceived_velocity /= num_neighbors_rule3;
			result += perceived_velocity * rule3Scale;
		}

		// Finalize Rule 2
		result += c * rule2Scale;

		result += vel1[index];

		// Clamp the speed
		float speed = glm::length(result);
		if (speed > maxSpeed) {
			result = glm::normalize(result) * maxSpeed;
		}

		vel2[index] = result;
	}
}

__global__ void kernUpdateVelNeighborSearchCoherent(
  int N, int gridResolution, glm::vec3 gridMin,
  float inverseCellWidth, float cellWidth,
  int *gridCellStartIndices, int *gridCellEndIndices,
  glm::vec3 *pos, glm::vec3 *vel1, glm::vec3 *vel2) {
  // TODO-2.3 - This should be very similar to kernUpdateVelNeighborSearchScattered,
  // except with one less level of indirection.
  // This should expect gridCellStartIndices and gridCellEndIndices to refer
  // directly to pos and vel1.
  // - Identify the grid cell that this particle is in
  // - Identify which cells may contain neighbors. This isn't always 8.
  // - For each cell, read the start/end indices in the boid pointer array.
  //   DIFFERENCE: For best results, consider what order the cells should be
  //   checked in to maximize the memory benefits of reordering the boids data.
  // - Access each boid in the cell and compute velocity change from
  //   the boids rules, if this boid is within the neighborhood distance.
  // - Clamp the speed change before putting the new speed in vel2

	int index = threadIdx.x + (blockIdx.x * blockDim.x);
	if (index < N) {
		glm::vec3 perceived_center(0.0f), c(0.0f), perceived_velocity(0.0f), result(0.0f);
		int num_neighbors_rule1 = 0, num_neighbors_rule3 = 0;

		glm::vec3 position = pos[index], gridPos = glm::floor((position - gridMin) * inverseCellWidth);

		// Iterate over the 3x3x3 grid of cells around the current cell (upto 8 cells)
		for (int x = imax(gridPos.x - 1, 0); x < imin(gridPos.x + 1, gridResolution - 1); x++) {
			for (int y = imax(gridPos.y - 1, 0); y < imin(gridPos.y + 1, gridResolution - 1); y++) {
				for (int z = imax(gridPos.z - 1, 0); z < imin(gridPos.z + 1, gridResolution - 1); z++) {

					int currGridIndex = gridIndex3Dto1D(x, y, z, gridResolution);
					int currStartIndex = gridCellStartIndices[currGridIndex];
					int currEndIndex = gridCellEndIndices[currGridIndex];

					// If no boids are in the current cell, continue
					if (currStartIndex == -1 || currEndIndex == -1) {
						continue;
					}


					for (int i = currStartIndex; i <= currEndIndex; i++) {

						if (i != index) {
							float distance = glm::distance(pos[i], position);

							// Rule 1
							if (distance < rule1Distance) {
								perceived_center += pos[i];
								num_neighbors_rule1++;
							}
							// Rule 2
							if (distance < rule2Distance) {
								c -= (pos[i] - position);
							}
							// Rule 3
							if (distance < rule3Distance) {
								perceived_velocity += vel1[i];
								num_neighbors_rule3++;
							}
						}
					}
				}
			}
		}

		// Finalize Rule 1 and Rule 3

		if (num_neighbors_rule1 > 0) {
			perceived_center /= num_neighbors_rule1;
			result += (perceived_center - position) * rule1Scale;
		}

		if (num_neighbors_rule3 > 0) {
			perceived_velocity /= num_neighbors_rule3;
			result += perceived_velocity * rule3Scale;
		}

		// Finalize Rule 2
		result += c * rule2Scale;

		result += vel1[index];

		// Clamp the speed
		float speed = glm::length(result);
		if (speed > maxSpeed) {
			result = glm::normalize(result) * maxSpeed;
		}

		vel2[index] = result;
	}
}

/**
* Step the entire N-body simulation by `dt` seconds.
*/
void Boids::stepSimulationNaive(float dt) {
  // TODO-1.2 - use the kernels you wrote to step the simulation forward in time.
  // TODO-1.2 ping-pong the velocity buffers
	dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);
	kernUpdateVelocityBruteForce <<<fullBlocksPerGrid, blockSize >>> (numObjects, dev_pos, dev_vel1, dev_vel2);
	checkCUDAErrorWithLine("kernUpdateVelocityBruteForce failed!");

	kernUpdatePos <<<fullBlocksPerGrid, blockSize >>> (numObjects, dt, dev_pos, dev_vel2);
	checkCUDAErrorWithLine("kernUpdatePos failed!");

	std::swap(dev_vel1, dev_vel2);
}

void Boids::stepSimulationScatteredGrid(float dt) {
  // TODO-2.1
  // Uniform Grid Neighbor search using Thrust sort.
  // In Parallel:
  // - label each particle with its array index as well as its grid index.
  //   Use 2x width grids.
  // - Unstable key sort using Thrust. A stable sort isn't necessary, but you
  //   are welcome to do a performance comparison.
  // - Naively unroll the loop for finding the start and end indices of each
  //   cell's data pointers in the array of boid indices
  // - Perform velocity updates using neighbor search
  // - Update positions
  // - Ping-pong buffers as needed

	dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);	
	dim3 fullCellsPerGrid((gridCellCount + blockSize - 1) / blockSize);

	// Compute indices
	kernComputeIndices <<<fullBlocksPerGrid, blockSize >>> (numObjects, gridSideCount, gridMinimum, gridInverseCellWidth, dev_pos, dev_particleArrayIndices, dev_particleGridIndices);
	checkCUDAErrorWithLine("kernComputeIndices failed!");

	// Unstable key sort
	thrust::device_ptr<int> dev_thrust_keys(dev_particleGridIndices);
	thrust::device_ptr<int> dev_thrust_values(dev_particleArrayIndices);
	thrust::sort_by_key(dev_thrust_keys, dev_thrust_keys + numObjects, dev_thrust_values);

	// Reset buffers to -1
	kernResetIntBuffer <<<fullCellsPerGrid, blockSize >>> (gridCellCount, dev_gridCellStartIndices, -1);
	checkCUDAErrorWithLine("kernResetIntBuffer for Start failed!");
	kernResetIntBuffer <<<fullCellsPerGrid, blockSize >>> (gridCellCount, dev_gridCellEndIndices, -1);
	checkCUDAErrorWithLine("kernResetIntBuffer for End failed!");

	// Identify cell start and end indices
	kernIdentifyCellStartEnd <<<fullBlocksPerGrid, blockSize >>> (numObjects, dev_particleGridIndices, dev_gridCellStartIndices, dev_gridCellEndIndices);
	checkCUDAErrorWithLine("kernIdentifyCellStartEnd failed!");

	// Update velocity using neighbor search
	kernUpdateVelNeighborSearchScattered <<<fullBlocksPerGrid, blockSize >>> (numObjects, gridSideCount, gridMinimum, gridInverseCellWidth, gridCellWidth, dev_gridCellStartIndices, dev_gridCellEndIndices, dev_particleArrayIndices, dev_pos, dev_vel1, dev_vel2);
	checkCUDAErrorWithLine("kernUpdateVelNeighborSearchScattered failed!");

	// Update positions
	kernUpdatePos <<<fullBlocksPerGrid, blockSize >>> (numObjects, dt, dev_pos, dev_vel2);
	checkCUDAErrorWithLine("kernUpdatePos failed!");
	// Ping Pong buffers
	std::swap(dev_vel1, dev_vel2);
}

void Boids::stepSimulationCoherentGrid(float dt) {
  // TODO-2.3 - start by copying Boids::stepSimulationNaiveGrid
  // Uniform Grid Neighbor search using Thrust sort on cell-coherent data.
  // In Parallel:
  // - Label each particle with its array index as well as its grid index.
  //   Use 2x width grids
  // - Unstable key sort using Thrust. A stable sort isn't necessary, but you
  //   are welcome to do a performance comparison.
  // - Naively unroll the loop for finding the start and end indices of each
  //   cell's data pointers in the array of boid indices
  // - BIG DIFFERENCE: use the rearranged array index buffer to reshuffle all
  //   the particle data in the simulation array.
  //   CONSIDER WHAT ADDITIONAL BUFFERS YOU NEED
  // - Perform velocity updates using neighbor search
  // - Update positions
  // - Ping-pong buffers as needed. THIS MAY BE DIFFERENT FROM BEFORE.

	dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);
	dim3 fullCellsPerGrid((gridCellCount + blockSize - 1) / blockSize);

	// Compute indices
	kernComputeIndices << <fullBlocksPerGrid, blockSize >> > (numObjects, gridSideCount, gridMinimum, gridInverseCellWidth, dev_pos, dev_particleArrayIndices, dev_particleGridIndices);
	checkCUDAErrorWithLine("kernComputeIndices failed!");

	// Unstable key sort
	thrust::device_ptr<int> dev_thrust_keys(dev_particleGridIndices);
	thrust::device_ptr<int> dev_thrust_values(dev_particleArrayIndices);
	thrust::sort_by_key(dev_thrust_keys, dev_thrust_keys + numObjects, dev_thrust_values);

	// Reset buffers to -1
	kernResetIntBuffer <<<fullCellsPerGrid, blockSize >>> (gridCellCount, dev_gridCellStartIndices, -1);
	checkCUDAErrorWithLine("kernResetIntBuffer for Start failed!");
	kernResetIntBuffer <<<fullCellsPerGrid, blockSize >>> (gridCellCount, dev_gridCellEndIndices, -1);
	checkCUDAErrorWithLine("kernResetIntBuffer for End failed!");

	// Identify cell start and end indices
	kernIdentifyCellStartEnd <<<fullBlocksPerGrid, blockSize >>> (numObjects, dev_particleGridIndices, dev_gridCellStartIndices, dev_gridCellEndIndices);
	checkCUDAErrorWithLine("kernIdentifyCellStartEnd failed!");

	// Reshuffle buffers
	kernReshuffleBuffer <<<fullBlocksPerGrid, blockSize >>> (numObjects, dev_particleArrayIndices, dev_pos, dev_vel1, dev_coherentPos, dev_coherentVel);
	checkCUDAErrorWithLine("kernReshuffleBuffer failed!");

	// Update velocity using neighbor search
	kernUpdateVelNeighborSearchCoherent <<<fullBlocksPerGrid, blockSize >>> (numObjects, gridSideCount, gridMinimum, gridInverseCellWidth, gridCellWidth, dev_gridCellStartIndices, dev_gridCellEndIndices, dev_coherentPos, dev_coherentVel, dev_vel2);
	checkCUDAErrorWithLine("kernUpdateVelNeighborSearchCoherent failed!");

	// Update positions
	kernUpdatePos <<<fullBlocksPerGrid, blockSize >>> (numObjects, dt, dev_coherentPos, dev_vel2);
	checkCUDAErrorWithLine("kernUpdatePos failed!");

	// Ping Pong buffers
	std::swap(dev_pos, dev_coherentPos);
	std::swap(dev_vel1, dev_coherentVel);
	std::swap(dev_vel1, dev_vel2);



}

void Boids::endSimulation() {
  hipFree(dev_vel1);
  hipFree(dev_vel2);
  hipFree(dev_pos);

  // TODO-2.1 TODO-2.3 - Free any additional buffers here.
  hipFree(dev_particleArrayIndices);
  hipFree(dev_particleGridIndices);
  hipFree(dev_gridCellStartIndices);
  hipFree(dev_gridCellStartIndices);

  hipFree(dev_coherentPos);
  hipFree(dev_coherentVel);
}

void Boids::unitTest() {
  // LOOK-1.2 Feel free to write additional tests here.

  // test unstable sort
  int *dev_intKeys;
  int *dev_intValues;
  int N = 10;

  std::unique_ptr<int[]>intKeys{ new int[N] };
  std::unique_ptr<int[]>intValues{ new int[N] };

  intKeys[0] = 0; intValues[0] = 0;
  intKeys[1] = 1; intValues[1] = 1;
  intKeys[2] = 0; intValues[2] = 2;
  intKeys[3] = 3; intValues[3] = 3;
  intKeys[4] = 0; intValues[4] = 4;
  intKeys[5] = 2; intValues[5] = 5;
  intKeys[6] = 2; intValues[6] = 6;
  intKeys[7] = 0; intValues[7] = 7;
  intKeys[8] = 5; intValues[8] = 8;
  intKeys[9] = 6; intValues[9] = 9;

  hipMalloc((void**)&dev_intKeys, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_intKeys failed!");

  hipMalloc((void**)&dev_intValues, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_intValues failed!");

  dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);

  std::cout << "before unstable sort: " << std::endl;
  for (int i = 0; i < N; i++) {
    std::cout << "  key: " << intKeys[i];
    std::cout << " value: " << intValues[i] << std::endl;
  }

  // How to copy data to the GPU
  hipMemcpy(dev_intKeys, intKeys.get(), sizeof(int) * N, hipMemcpyHostToDevice);
  hipMemcpy(dev_intValues, intValues.get(), sizeof(int) * N, hipMemcpyHostToDevice);

  // Wrap device vectors in thrust iterators for use with thrust.
  thrust::device_ptr<int> dev_thrust_keys(dev_intKeys);
  thrust::device_ptr<int> dev_thrust_values(dev_intValues);
  // LOOK-2.1 Example for using thrust::sort_by_key
  thrust::sort_by_key(dev_thrust_keys, dev_thrust_keys + N, dev_thrust_values);

  // How to copy data back to the CPU side from the GPU
  hipMemcpy(intKeys.get(), dev_intKeys, sizeof(int) * N, hipMemcpyDeviceToHost);
  hipMemcpy(intValues.get(), dev_intValues, sizeof(int) * N, hipMemcpyDeviceToHost);
  checkCUDAErrorWithLine("memcpy back failed!");

  std::cout << "after unstable sort: " << std::endl;
  for (int i = 0; i < N; i++) {
    std::cout << "  key: " << intKeys[i];
    std::cout << " value: " << intValues[i] << std::endl;
  }

  // cleanup
  hipFree(dev_intKeys);
  hipFree(dev_intValues);
  checkCUDAErrorWithLine("hipFree failed!");
  return;
}
