#include "hip/hip_runtime.h"
#define GLM_FORCE_CUDA
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <glm/glm.hpp>
#include "utilityCore.hpp"
#include "kernel.h"

// LOOK-2.1 potentially useful for doing grid-based neighbor search
#ifndef imax
#define imax( a, b ) ( ((a) > (b)) ? (a) : (b) )
#endif

#ifndef imin
#define imin( a, b ) ( ((a) < (b)) ? (a) : (b) )
#endif

#define checkCUDAErrorWithLine(msg) checkCUDAError(msg, __LINE__)

/**
* Check for CUDA errors; print and exit if there was a problem.
*/
void checkCUDAError(const char *msg, int line = -1) {
  hipError_t err = hipGetLastError();
  if (hipSuccess != err) {
    if (line >= 0) {
      fprintf(stderr, "Line %d: ", line);
    }
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}


/*****************
* Configuration *
*****************/

/*! Block size used for CUDA kernel launch. */
#define blockSize 128

// LOOK-1.2 Parameters for the boids algorithm.
// These worked well in our reference implementation.
#define rule1Distance 5.0f
#define rule2Distance 3.0f
#define rule3Distance 5.0f

#define rule1Scale 0.01f
#define rule2Scale 0.1f
#define rule3Scale 0.1f

#define maxSpeed 1.0f

/*! Size of the starting area in simulation space. */
#define scene_scale 100.0f

/***********************************************
* Kernel state (pointers are device pointers) *
***********************************************/

int numObjects;
dim3 threadsPerBlock(blockSize);

// LOOK-1.2 - These buffers are here to hold all your boid information.
// These get allocated for you in Boids::initSimulation.
// Consider why you would need two velocity buffers in a simulation where each
// boid cares about its neighbors' velocities.
// These are called ping-pong buffers.
glm::vec3 *dev_pos;
glm::vec3 *dev_vel1;
glm::vec3 *dev_vel2;

// LOOK-2.1 - these are NOT allocated for you. You'll have to set up the thrust
// pointers on your own too.

// For efficient sorting and the uniform grid. These should always be parallel.
int *dev_particleArrayIndices; // What index in dev_pos and dev_velX represents this particle?
int *dev_particleGridIndices; // What grid cell is this particle in?
// needed for use with thrust
thrust::device_ptr<int> dev_thrust_particleArrayIndices;
thrust::device_ptr<int> dev_thrust_particleGridIndices;

int *dev_gridCellStartIndices; // What part of dev_particleArrayIndices belongs
int *dev_gridCellEndIndices;   // to this cell?

// TODO-2.3 - consider what additional buffers you might need to reshuffle
// the position and velocity data to be coherent within cells.

// LOOK-2.1 - Grid parameters based on simulation parameters.
// These are automatically computed for you in Boids::initSimulation
int gridCellCount;
int gridSideCount;
float gridCellWidth;
float gridInverseCellWidth;
glm::vec3 gridMinimum;

/******************
* initSimulation *
******************/

__host__ __device__ unsigned int hash(unsigned int a) {
  a = (a + 0x7ed55d16) + (a << 12);
  a = (a ^ 0xc761c23c) ^ (a >> 19);
  a = (a + 0x165667b1) + (a << 5);
  a = (a + 0xd3a2646c) ^ (a << 9);
  a = (a + 0xfd7046c5) + (a << 3);
  a = (a ^ 0xb55a4f09) ^ (a >> 16);
  return a;
}

/**
* LOOK-1.2 - this is a typical helper function for a CUDA kernel.
* Function for generating a random vec3.
*/
__host__ __device__ glm::vec3 generateRandomVec3(float time, int index) {
  thrust::default_random_engine rng(hash((int)(index * time)));
  thrust::uniform_real_distribution<float> unitDistrib(-1, 1);

  return glm::vec3((float)unitDistrib(rng), (float)unitDistrib(rng), (float)unitDistrib(rng));
}

/**
* LOOK-1.2 - This is a basic CUDA kernel.
* CUDA kernel for generating boids with a specified mass randomly around the star.
*/
__global__ void kernGenerateRandomPosArray(int time, int N, glm::vec3 * arr, float scale) {
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (index < N) {
    glm::vec3 rand = generateRandomVec3(time, index);
    arr[index].x = scale * rand.x;
    arr[index].y = scale * rand.y;
    arr[index].z = scale * rand.z;
  }
}

/**
* Initialize memory, update some globals
*/
void Boids::initSimulation(int N) {
  numObjects = N;
  dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);

  // LOOK-1.2 - This is basic CUDA memory management and error checking.
  // Don't forget to hipFree in  Boids::endSimulation.
  hipMalloc((void**)&dev_pos, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_pos failed!");

  hipMalloc((void**)&dev_vel1, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_vel1 failed!");

  hipMalloc((void**)&dev_vel2, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_vel2 failed!");

  // LOOK-1.2 - This is a typical CUDA kernel invocation.
  kernGenerateRandomPosArray<<<fullBlocksPerGrid, blockSize>>>(1, numObjects,
    dev_pos, scene_scale);
  checkCUDAErrorWithLine("kernGenerateRandomPosArray failed!");

  // LOOK-2.1 computing grid params
  gridCellWidth = 2.0f * std::max(std::max(rule1Distance, rule2Distance), rule3Distance);
  int halfSideCount = (int)(scene_scale / gridCellWidth) + 1;
  gridSideCount = 2 * halfSideCount;

  gridCellCount = gridSideCount * gridSideCount * gridSideCount;
  gridInverseCellWidth = 1.0f / gridCellWidth;
  float halfGridWidth = gridCellWidth * halfSideCount;
  gridMinimum.x -= halfGridWidth;
  gridMinimum.y -= halfGridWidth;
  gridMinimum.z -= halfGridWidth;

  // DONE-2.1 TODO-2.3 - Allocate additional buffers here. 
  hipMalloc((void**)&dev_particleArrayIndices, N * sizeof(int)); 
  checkCUDAErrorWithLine("hipMalloc dev_particleArrayIndices failed!");

  hipMalloc((void**)&dev_particleGridIndices, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_particleGridIndices failed!");

  hipMalloc((void**)&dev_gridCellStartIndices, gridCellCount * sizeof(int)); 
  checkCUDAErrorWithLine("hipMalloc dev_gridCellStartIndices failed!");

  hipMalloc((void**)&dev_gridCellEndIndices, gridCellCount * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_gridCellEndIndices failed!");

  dev_thrust_particleArrayIndices = thrust::device_ptr<int>(dev_particleArrayIndices); 
  dev_thrust_particleGridIndices = thrust::device_ptr<int>(dev_particleGridIndices);


  /*dim3 blocksPerGrid((gridCellCount + blockSize - 1) / blockSize);

  kernResetIntBuffer<<<blocksPerGrid, blockSize>>>(gridCellCount, &dev_gridCellStartIndices, -1); 
  checkCUDAErrorWithLine("kernResetIntBuffer failed!");
  kernResetIntBuffer<<<blocksPerGrid, blockSize>>>(gridCellCount, &dev_gridCellEndIndices, -1);
  checkCUDAErrorWithLine("kernResetIntBuffer failed!");*/

  hipDeviceSynchronize();
}


/******************
* copyBoidsToVBO *
******************/

/**
* Copy the boid positions into the VBO so that they can be drawn by OpenGL.
*/
__global__ void kernCopyPositionsToVBO(int N, glm::vec3 *pos, float *vbo, float s_scale) {
  int index = threadIdx.x + (blockIdx.x * blockDim.x);

  float c_scale = -1.0f / s_scale;

  if (index < N) {
    vbo[4 * index + 0] = pos[index].x * c_scale;
    vbo[4 * index + 1] = pos[index].y * c_scale;
    vbo[4 * index + 2] = pos[index].z * c_scale;
    vbo[4 * index + 3] = 1.0f;
  }
}

__global__ void kernCopyVelocitiesToVBO(int N, glm::vec3 *vel, float *vbo, float s_scale) {
  int index = threadIdx.x + (blockIdx.x * blockDim.x);

  if (index < N) {
    vbo[4 * index + 0] = vel[index].x + 0.3f;
    vbo[4 * index + 1] = vel[index].y + 0.3f;
    vbo[4 * index + 2] = vel[index].z + 0.3f;
    vbo[4 * index + 3] = 1.0f;
  }
}

/**
* Wrapper for call to the kernCopyboidsToVBO CUDA kernel.
*/
void Boids::copyBoidsToVBO(float *vbodptr_positions, float *vbodptr_velocities) {
  dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);

  kernCopyPositionsToVBO << <fullBlocksPerGrid, blockSize >> >(numObjects, dev_pos, vbodptr_positions, scene_scale);
  kernCopyVelocitiesToVBO << <fullBlocksPerGrid, blockSize >> >(numObjects, dev_vel1, vbodptr_velocities, scene_scale);

  checkCUDAErrorWithLine("copyBoidsToVBO failed!");

  hipDeviceSynchronize();
}


/******************
* stepSimulation *
******************/

/**
* LOOK-1.2 You can use this as a helper for kernUpdateVelocityBruteForce.
* __device__ code can be called from a __global__ context
* Compute the new velocity on the boid with index `iSelf` due to the `N` boids
* in the `pos` and `vel` arrays.
*/
__device__ glm::vec3 computeVelocityChange(int N, int iSelf, const glm::vec3 *pos, const glm::vec3 *vel) {
  glm::vec3     perceived_center(0.f, 0.f, 0.f);
  glm::vec3                    c(0.f, 0.f, 0.f); 
  glm::vec3   perceived_velocity(0.f, 0.f, 0.f); 
  int r1_number_of_neighbours = 0; 
  int r3_number_of_neighbours = 0;

  const glm::vec3& boidpos = pos[iSelf]; 

  for (int i = 0; i < N; i++) {
    const glm::vec3& b = pos[i]; 
    const glm::vec3& bvel = vel[i]; 

    // Rule 1: boids fly towards their local perceived center of mass, which excludes themselves
    if (i != iSelf && glm::distance(b, boidpos) < rule1Distance) {
      r1_number_of_neighbours++;
      perceived_center += b; 
    }

    // Rule 2: boids try to stay a distance d away from each other
    if (i != iSelf && glm::distance(b, boidpos) < rule2Distance) {
      c -= (b - boidpos); 
    }

    // Rule 3: boids try to match the speed of surrounding boids
    if (i != iSelf && glm::distance(b, boidpos) < rule3Distance) {
      r3_number_of_neighbours++; 
      perceived_velocity += bvel; 
    }
  }

  glm::vec3 rule1(0.f); 
  glm::vec3 rule2(0.f);
  glm::vec3 rule3(0.f);

  if (r1_number_of_neighbours > 0) {
    perceived_center /= glm::vec3(r1_number_of_neighbours);
    rule1 = (perceived_center - boidpos) * rule1Scale;
  }
  
  rule2 = c * rule2Scale; 

  if (r3_number_of_neighbours > 0) {
    perceived_velocity /= glm::vec3(r3_number_of_neighbours);
    rule3 = perceived_velocity * rule3Scale;
  }
 
  return vel[iSelf] + (rule1 + rule2 + rule3);
}

/**
* TODO-1.2 implement basic flocking
* For each of the `N` bodies, update its position based on its current velocity.
*/
__global__ void kernUpdateVelocityBruteForce(int N, glm::vec3 *pos,
  glm::vec3 *vel1, glm::vec3 *vel2) {
  int idx = threadIdx.x + (blockIdx.x * blockDim.x);
  // Compute a new velocity based on pos and vel1
  glm::vec3 newVel = computeVelocityChange(N, idx, pos, vel1); 
  // Clamp the speed
  if (glm::length(newVel) > maxSpeed) {
    newVel = glm::normalize(newVel) * maxSpeed;  // Normalize and scale to maxVelocity
  }
  // Record the new velocity into vel2. Question: why NOT vel1?
  vel2[idx] = newVel; 
}

/**
* LOOK-1.2 Since this is pretty trivial, we implemented it for you.
* For each of the `N` bodies, update its position based on its current velocity.
*/
__global__ void kernUpdatePos(int N, float dt, glm::vec3 *pos, glm::vec3 *vel) {
  // Update position by velocity
  int index = threadIdx.x + (blockIdx.x * blockDim.x);
  if (index >= N) {
    return;
  }
  glm::vec3 thisPos = pos[index];
  thisPos += vel[index] * dt;

  // Wrap the boids around so we don't lose them
  thisPos.x = thisPos.x < -scene_scale ? scene_scale : thisPos.x;
  thisPos.y = thisPos.y < -scene_scale ? scene_scale : thisPos.y;
  thisPos.z = thisPos.z < -scene_scale ? scene_scale : thisPos.z;

  thisPos.x = thisPos.x > scene_scale ? -scene_scale : thisPos.x;
  thisPos.y = thisPos.y > scene_scale ? -scene_scale : thisPos.y;
  thisPos.z = thisPos.z > scene_scale ? -scene_scale : thisPos.z;

  pos[index] = thisPos;
}

// LOOK-2.1 Consider this method of computing a 1D index from a 3D grid index.
// LOOK-2.3 Looking at this method, what would be the most memory efficient
//          order for iterating over neighboring grid cells?
//          for(x)
//            for(y)
//             for(z)? Or some other order?
__device__ int gridIndex3Dto1D(int x, int y, int z, int gridResolution) {
  return x + y * gridResolution + z * gridResolution * gridResolution;
}

__global__ void kernComputeIndices(int N, int gridResolution,
  glm::vec3 gridMin, float inverseCellWidth,
  glm::vec3 *pos, int *indices, int *gridIndices) {
  // DONE-2.1

  /*int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;
  int z = (blockIdx.z * blockDim.z) + threadIdx.z;*/

  // - Label each boid with the index of its grid cell.
  int index = (blockIdx.x * blockDim.x) + threadIdx.x; 

  if (index < N) { 
    glm::vec3 gridPos = pos[index] - gridMin;
    gridPos *= inverseCellWidth; 
    gridIndices[index] = gridIndex3Dto1D(gridPos.x, gridPos.y, gridPos.z, gridResolution);
    // - Set up a parallel array of integer indices as pointers to the actual
    //   boid data in pos and vel1/vel2
    indices[index] = index; 
  }
}

// LOOK-2.1 Consider how this could be useful for indicating that a cell
//          does not enclose any boids
__global__ void kernResetIntBuffer(int N, int *intBuffer, int value) {
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (index < N) {
    intBuffer[index] = value;
  }
}

// looping through the the particle grid indices (N boids)
__global__ void kernIdentifyCellStartEnd(int N, int *particleGridIndices,
  int *gridCellStartIndices, int *gridCellEndIndices) {
  // DONE-2.1
  int index = (blockIdx.x * blockDim.x) + threadIdx.x; 
  if (index < N) {
    // Identify the start point of each cell in the gridIndices array.
    // This is basically a parallel unrolling of a loop that goes
    // "this index doesn't match the one before it, must be a new cell!"
    if (index == 0 || particleGridIndices[index] != particleGridIndices[index - 1]) {  // start index
      gridCellStartIndices[particleGridIndices[index]] = index;
    }
    if (index == (N - 1) || particleGridIndices[index] != particleGridIndices[index + 1]) {  // end index
      gridCellEndIndices[particleGridIndices[index]] = index;
    }
  }
}

__global__ void kernUpdateVelNeighborSearchScattered(
  int N, int gridResolution, glm::vec3 gridMin,
  float inverseCellWidth, float cellWidth,
  int *gridCellStartIndices, int *gridCellEndIndices,
  int *particleArrayIndices,
  glm::vec3 *pos, glm::vec3 *vel1, glm::vec3 *vel2) {
  // TODO-2.1 - Update a boid's velocity using the uniform grid to reduce
  // the number of boids that need to be checked.
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (index >= N) {
    return;
  }

  float neighbourhoodDistance = fmax(fmax(rule1Distance, rule2Distance), rule3Distance); 

  // - Identify the grid cell that this particle is in
  glm::vec3 gridPos = pos[index] - gridMin;
  int gridIdx = gridIndex3Dto1D(
    gridPos.x *= inverseCellWidth, gridPos.y *= inverseCellWidth, gridPos.z *= inverseCellWidth, gridResolution
  );

  // - Identify which cells may contain neighbors. This isn't always 8.
  int minX = imax(0, static_cast<int>((gridPos.x - neighbourhoodDistance) * inverseCellWidth));
  int maxX = imin(gridResolution - 1, static_cast<int>((gridPos.x + neighbourhoodDistance) * inverseCellWidth));
  int minY = imax(0, static_cast<int>((gridPos.y - neighbourhoodDistance) * inverseCellWidth));
  int maxY = imin(gridResolution - 1, static_cast<int>((gridPos.y + neighbourhoodDistance) * inverseCellWidth));
  int minZ = imax(0, static_cast<int>((gridPos.z - neighbourhoodDistance) * inverseCellWidth));
  int maxZ = imin(gridResolution - 1, static_cast<int>((gridPos.z + neighbourhoodDistance) * inverseCellWidth));

  glm::vec3     perceived_center(0.f, 0.f, 0.f);
  glm::vec3                    c(0.f, 0.f, 0.f);
  glm::vec3   perceived_velocity(0.f, 0.f, 0.f);
  int r1_number_of_neighbours = 0;
  int r3_number_of_neighbours = 0;
  glm::vec3 boidpos = pos[index];
  glm::vec3 boidvel = vel1[index];

  for (int i = minX; i <= maxX; i++) {
    for (int j = minY; j <= maxY; j++) {
      for (int k = minZ; k <= maxZ; k++) {
        glm::vec3 cell(i * cellWidth, j * cellWidth, k * cellWidth); // position of current cell (corner)

        // find the closest point on current cell to our boid
        glm::vec3 closest; 
        closest.x = fmax(cell.x, fmin(gridPos.x, cell.x + cellWidth));
        closest.y = fmax(cell.y, fmin(gridPos.y, cell.y + cellWidth));
        closest.z = fmax(cell.z, fmin(gridPos.z, cell.z + cellWidth));

        // if closest point is within neighbourhood distance, consider the neighb boids in that cell
        if (glm::distance(closest, gridPos) <= neighbourhoodDistance) {
          int startEndIdx = gridIndex3Dto1D(i, j, k, gridResolution);

          int gridCellStart = gridCellStartIndices[startEndIdx];
          int gridCellEnd = gridCellEndIndices[startEndIdx];

          if (gridCellStart == -1) {
            continue;
          }

          for (int boidIdx = gridCellStart; boidIdx <= gridCellEnd; boidIdx++) {
            glm::vec3 bpos = pos[boidIdx];
            glm::vec3 bvel = vel1[boidIdx];

            if (boidIdx != index) {

              // Rule 1: boids fly towards their local perceived center of mass, which excludes themselves
              if (glm::distance(bpos, boidpos) < rule1Distance) {
                r1_number_of_neighbours++;
                perceived_center += bpos;
              }

              // Rule 2: boids try to stay a distance d away from each other
              if (glm::distance(bpos, boidpos) < rule2Distance) {
                c -= (bpos - boidpos);
              }

              // Rule 3: boids try to match the speed of surrounding boids
              if (glm::distance(bpos, boidpos) < rule3Distance) {
                r3_number_of_neighbours++;
                perceived_velocity += bvel;
              }
            }
          }
        }
      }
    }
  }

  glm::vec3 rule1(0.f);
  glm::vec3 rule2(0.f);
  glm::vec3 rule3(0.f);

  if (r1_number_of_neighbours > 0) {
    perceived_center /= glm::vec3(r1_number_of_neighbours);
    rule1 = (perceived_center - boidpos) * rule1Scale;
  }

  rule2 = c * rule2Scale;

  if (r3_number_of_neighbours > 0) {
    perceived_velocity /= glm::vec3(r3_number_of_neighbours);
    rule3 = perceived_velocity * rule3Scale;
  }

  glm::vec3 newVel = vel1[index] + (rule1 + rule2 + rule3);

  // - Clamp the speed change before putting the new speed in vel2
  if (glm::length(newVel) > maxSpeed) {
    newVel = glm::normalize(newVel) * maxSpeed;
  }
  
  vel2[index] = newVel; 
}

__global__ void kernUpdateVelNeighborSearchCoherent(
  int N, int gridResolution, glm::vec3 gridMin,
  float inverseCellWidth, float cellWidth,
  int *gridCellStartIndices, int *gridCellEndIndices,
  glm::vec3 *pos, glm::vec3 *vel1, glm::vec3 *vel2) {
  // TODO-2.3 - This should be very similar to kernUpdateVelNeighborSearchScattered,
  // except with one less level of indirection.
  // This should expect gridCellStartIndices and gridCellEndIndices to refer
  // directly to pos and vel1.
  // - Identify the grid cell that this particle is in
  // - Identify which cells may contain neighbors. This isn't always 8.
  // - For each cell, read the start/end indices in the boid pointer array.
  //   DIFFERENCE: For best results, consider what order the cells should be
  //   checked in to maximize the memory benefits of reordering the boids data.
  // - Access each boid in the cell and compute velocity change from
  //   the boids rules, if this boid is within the neighborhood distance.
  // - Clamp the speed change before putting the new speed in vel2
}

/**
* Step the entire N-body simulation by `dt` seconds.
*/
void Boids::stepSimulationNaive(float dt) {
  int blocksPerGrid = (numObjects + threadsPerBlock.x - 1) / threadsPerBlock.x;

  kernUpdateVelocityBruteForce<<<blocksPerGrid, threadsPerBlock>>>(numObjects, dev_pos, dev_vel1, dev_vel2);
  checkCUDAErrorWithLine("kernUpdateVelocityBruteForce failed.");

  kernUpdatePos<<<blocksPerGrid, threadsPerBlock>>>(numObjects, dt, dev_pos, dev_vel2); 
  checkCUDAErrorWithLine("kernUpdateVelocityBruteForce failed.");

  // ping-pong the velocity buffers
  hipMemcpy(dev_vel1, dev_vel2, numObjects * sizeof(glm::vec3), hipMemcpyDeviceToDevice);
}

void Boids::stepSimulationScatteredGrid(float dt) {
  // DONE-2.1
  // Uniform Grid Neighbor search using Thrust sort.
  // In Parallel:

  int blocksPerGrid = (numObjects + threadsPerBlock.x - 1) / threadsPerBlock.x;
  
  // - label each particle with its array index as well as its grid index.
  //   Use 2x width grids.

  kernComputeIndices<<<blocksPerGrid, threadsPerBlock>>>(numObjects, gridCellCount, gridMinimum, gridInverseCellWidth, dev_pos, dev_particleArrayIndices, dev_particleGridIndices);

  // - Unstable key sort using Thrust. A stable sort isn't necessary, but you
  //   are welcome to do a performance comparison.

  thrust::sort_by_key(dev_thrust_particleGridIndices, dev_thrust_particleGridIndices + numObjects, dev_thrust_particleArrayIndices);
  
  // - Naively unroll the loop for finding the start and end indices of each
  //   cell's data pointers in the array of boid indices
  
  kernIdentifyCellStartEnd<<<blocksPerGrid, threadsPerBlock>>>(numObjects, dev_particleGridIndices, dev_gridCellStartIndices, dev_gridCellEndIndices); 

  // - Perform velocity updates using neighbor search
  
  kernUpdateVelNeighborSearchScattered<<<blocksPerGrid, threadsPerBlock>>>(
    numObjects, 
    gridCellCount, 
    gridMinimum, 
    gridInverseCellWidth, 
    gridCellWidth,
    dev_gridCellStartIndices, 
    dev_gridCellEndIndices,
    dev_particleGridIndices,
    dev_pos, 
    dev_vel1, 
    dev_vel2
  );

  // - Update positions

  kernUpdatePos<<<blocksPerGrid, threadsPerBlock>>>(numObjects, dt, dev_pos, dev_vel2); 

  // - Ping-pong buffers as needed

  hipMemcpy(dev_vel1, dev_vel2, numObjects * sizeof(glm::vec3), hipMemcpyDeviceToDevice);
}

void Boids::stepSimulationCoherentGrid(float dt) {
  // TODO-2.3 - start by copying Boids::stepSimulationNaiveGrid
  // Uniform Grid Neighbor search using Thrust sort on cell-coherent data.
  // In Parallel:
  // - Label each particle with its array index as well as its grid index.
  //   Use 2x width grids
  // - Unstable key sort using Thrust. A stable sort isn't necessary, but you
  //   are welcome to do a performance comparison.
  // - Naively unroll the loop for finding the start and end indices of each
  //   cell's data pointers in the array of boid indices
  // - BIG DIFFERENCE: use the rearranged array index buffer to reshuffle all
  //   the particle data in the simulation array.
  //   CONSIDER WHAT ADDITIONAL BUFFERS YOU NEED
  // - Perform velocity updates using neighbor search
  // - Update positions
  // - Ping-pong buffers as needed. THIS MAY BE DIFFERENT FROM BEFORE.
}

void Boids::endSimulation() {
  hipFree(dev_vel1);
  hipFree(dev_vel2);
  hipFree(dev_pos);

  // DONE-2.1 TODO-2.3 - Free any additional buffers here.
  hipFree(dev_particleArrayIndices); 
  hipFree(dev_particleGridIndices); 
  hipFree(dev_gridCellStartIndices);
  hipFree(dev_gridCellEndIndices); 
}

void Boids::unitTest() {
  // LOOK-1.2 Feel free to write additional tests here.

  // test unstable sort
  int *dev_intKeys;
  int *dev_intValues;
  int N = 10;

  std::unique_ptr<int[]>intKeys{ new int[N] };
  std::unique_ptr<int[]>intValues{ new int[N] };

  intKeys[0] = 0; intValues[0] = 0;
  intKeys[1] = 1; intValues[1] = 1;
  intKeys[2] = 0; intValues[2] = 2;
  intKeys[3] = 3; intValues[3] = 3;
  intKeys[4] = 0; intValues[4] = 4;
  intKeys[5] = 2; intValues[5] = 5;
  intKeys[6] = 2; intValues[6] = 6;
  intKeys[7] = 0; intValues[7] = 7;
  intKeys[8] = 5; intValues[8] = 8;
  intKeys[9] = 6; intValues[9] = 9;

  hipMalloc((void**)&dev_intKeys, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_intKeys failed!");

  hipMalloc((void**)&dev_intValues, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_intValues failed!");

  // dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);

  std::cout << "before unstable sort: " << std::endl;
  for (int i = 0; i < N; i++) {
    std::cout << "  key: " << intKeys[i];
    std::cout << " value: " << intValues[i] << std::endl;
  }

  // How to copy data to the GPU
  hipMemcpy(dev_intKeys, intKeys.get(), sizeof(int) * N, hipMemcpyHostToDevice);
  hipMemcpy(dev_intValues, intValues.get(), sizeof(int) * N, hipMemcpyHostToDevice);

  // Wrap device vectors in thrust iterators for use with thrust.
  thrust::device_ptr<int> dev_thrust_keys(dev_intKeys);
  thrust::device_ptr<int> dev_thrust_values(dev_intValues);
  // LOOK-2.1 Example for using thrust::sort_by_key
  thrust::sort_by_key(dev_thrust_keys, dev_thrust_keys + N, dev_thrust_values);

  // How to copy data back to the CPU side from the GPU
  hipMemcpy(intKeys.get(), dev_intKeys, sizeof(int) * N, hipMemcpyDeviceToHost);
  hipMemcpy(intValues.get(), dev_intValues, sizeof(int) * N, hipMemcpyDeviceToHost);
  checkCUDAErrorWithLine("memcpy back failed!");

  std::cout << "after unstable sort: " << std::endl;
  for (int i = 0; i < N; i++) {
    std::cout << "  key: " << intKeys[i];
    std::cout << " value: " << intValues[i] << std::endl;
  }

  // cleanup
  hipFree(dev_intKeys);
  hipFree(dev_intValues);
  checkCUDAErrorWithLine("hipFree failed!");
  return;
}
