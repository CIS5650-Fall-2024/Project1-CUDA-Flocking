#include "hip/hip_runtime.h"
#define GLM_FORCE_CUDA
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <glm/glm.hpp>
#include "utilityCore.hpp"
#include "kernel.h"

// LOOK-2.1 potentially useful for doing grid-based neighbor search
#ifndef imax
#define imax( a, b ) ( ((a) > (b)) ? (a) : (b) )
#endif

#ifndef imin
#define imin( a, b ) ( ((a) < (b)) ? (a) : (b) )
#endif

#define checkCUDAErrorWithLine(msg) checkCUDAError(msg, __LINE__)

/**
* Check for CUDA errors; print and exit if there was a problem.
*/
void checkCUDAError(const char *msg, int line = -1) {
  hipError_t err = hipGetLastError();
  if (hipSuccess != err) {
    if (line >= 0) {
      fprintf(stderr, "Line %d: ", line);
    }
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}


/*****************
* Configuration *
*****************/

/*! Block size used for CUDA kernel launch. */
#define blockSize 128

// LOOK-1.2 Parameters for the boids algorithm.
// These worked well in our reference implementation.
#define rule1Distance 5.0f
#define rule2Distance 3.0f
#define rule3Distance 5.0f

#define rule1Scale 0.01f
#define rule2Scale 0.1f
#define rule3Scale 0.1f

#define maxSpeed 1.0f

/*! Size of the starting area in simulation space. */
#define scene_scale 100.0f

/***********************************************
* Kernel state (pointers are device pointers) *
***********************************************/

int numObjects;
dim3 threadsPerBlock(blockSize);

// LOOK-1.2 - These buffers are here to hold all your boid information.
// These get allocated for you in Boids::initSimulation.
// Consider why you would need two velocity buffers in a simulation where each
// boid cares about its neighbors' velocities.
// These are called ping-pong buffers.
glm::vec3 *dev_pos;
glm::vec3 *dev_vel1;
glm::vec3 *dev_vel2;

// LOOK-2.1 - these are NOT allocated for you. You'll have to set up the thrust
// pointers on your own too.

// For efficient sorting and the uniform grid. These should always be parallel.
int *dev_particleArrayIndices; // What index in dev_pos and dev_velX represents this particle?
int *dev_particleGridIndices; // What grid cell is this particle in?
// needed for use with thrust
thrust::device_ptr<int> dev_thrust_particleArrayIndices;
thrust::device_ptr<int> dev_thrust_particleGridIndices;

int *dev_gridCellStartIndices; // What part of dev_particleArrayIndices belongs
int *dev_gridCellEndIndices;   // to this cell?

// TODO-2.3 - consider what additional buffers you might need to reshuffle
// the position and velocity data to be coherent within cells.
glm::vec3 *dev_coherentPos;
glm::vec3* dev_coherentVel;

// LOOK-2.1 - Grid parameters based on simulation parameters.
// These are automatically computed for you in Boids::initSimulation
int gridCellCount;
int gridSideCount;
float gridCellWidth;
float gridInverseCellWidth;
glm::vec3 gridMinimum;

/******************
* initSimulation *
******************/

__host__ __device__ unsigned int hash(unsigned int a) {
  a = (a + 0x7ed55d16) + (a << 12);
  a = (a ^ 0xc761c23c) ^ (a >> 19);
  a = (a + 0x165667b1) + (a << 5);
  a = (a + 0xd3a2646c) ^ (a << 9);
  a = (a + 0xfd7046c5) + (a << 3);
  a = (a ^ 0xb55a4f09) ^ (a >> 16);
  return a;
}

/**
* LOOK-1.2 - this is a typical helper function for a CUDA kernel.
* Function for generating a random vec3.
*/
__host__ __device__ glm::vec3 generateRandomVec3(float time, int index) {
  thrust::default_random_engine rng(hash((int)(index * time)));
  thrust::uniform_real_distribution<float> unitDistrib(-1, 1);

  return glm::vec3((float)unitDistrib(rng), (float)unitDistrib(rng), (float)unitDistrib(rng));
}

/**
* LOOK-1.2 - This is a basic CUDA kernel.
* CUDA kernel for generating boids with a specified mass randomly around the star.
*/
__global__ void kernGenerateRandomPosArray(int time, int N, glm::vec3 * arr, float scale) {
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (index < N) {
    glm::vec3 rand = generateRandomVec3(time, index);
    arr[index].x = scale * rand.x;
    arr[index].y = scale * rand.y;
    arr[index].z = scale * rand.z;
  }
}

/**
* Initialize memory, update some globals
*/
void Boids::initSimulation(int N, float gridCellWidthScale) {
  numObjects = N;
  dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);

  // LOOK-1.2 - This is basic CUDA memory management and error checking.
  // Don't forget to hipFree in  Boids::endSimulation.
  hipMalloc((void**)&dev_pos, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_pos failed!");

  hipMalloc((void**)&dev_vel1, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_vel1 failed!");

  hipMalloc((void**)&dev_vel2, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_vel2 failed!");

  // LOOK-1.2 - This is a typical CUDA kernel invocation.
  kernGenerateRandomPosArray<<<fullBlocksPerGrid, blockSize>>>(1, numObjects,
    dev_pos, scene_scale);
  checkCUDAErrorWithLine("kernGenerateRandomPosArray failed!");

  // LOOK-2.1 computing grid params
  gridCellWidth = gridCellWidthScale * std::max(std::max(rule1Distance, rule2Distance), rule3Distance);
  int halfSideCount = (int)(scene_scale / gridCellWidth) + 1;
  gridSideCount = 2 * halfSideCount;

  gridCellCount = gridSideCount * gridSideCount * gridSideCount;
  gridInverseCellWidth = 1.0f / gridCellWidth;
  float halfGridWidth = gridCellWidth * halfSideCount;
  gridMinimum.x -= halfGridWidth;
  gridMinimum.y -= halfGridWidth;
  gridMinimum.z -= halfGridWidth;

  // TODO-2.1 TODO-2.3 - Allocate additional buffers here.
  hipMalloc((void**)&dev_particleArrayIndices, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_particleArrayIndices failed!");

  hipMalloc((void**)&dev_particleGridIndices, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_particleGridIndices failed!");

  hipMalloc((void**)&dev_gridCellStartIndices, gridCellCount * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_gridCellStartIndices failed!");

  hipMalloc((void**)&dev_gridCellEndIndices, gridCellCount * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_gridCellEndIndices failed!");

  hipMalloc((void**)&dev_coherentPos, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_coherentPos failed!");

  hipMalloc((void**)&dev_coherentVel, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_coherentVel failed!");

  hipDeviceSynchronize();
}


/******************
* copyBoidsToVBO *
******************/

/**
* Copy the boid positions into the VBO so that they can be drawn by OpenGL.
*/
__global__ void kernCopyPositionsToVBO(int N, glm::vec3 *pos, float *vbo, float s_scale) {
  int index = threadIdx.x + (blockIdx.x * blockDim.x);

  float c_scale = -1.0f / s_scale;

  if (index < N) {
    vbo[4 * index + 0] = pos[index].x * c_scale;
    vbo[4 * index + 1] = pos[index].y * c_scale;
    vbo[4 * index + 2] = pos[index].z * c_scale;
    vbo[4 * index + 3] = 1.0f;
  }
}

__global__ void kernCopyVelocitiesToVBO(int N, glm::vec3 *vel, float *vbo, float s_scale) {
  int index = threadIdx.x + (blockIdx.x * blockDim.x);

  if (index < N) {
    vbo[4 * index + 0] = vel[index].x + 0.3f;
    vbo[4 * index + 1] = vel[index].y + 0.3f;
    vbo[4 * index + 2] = vel[index].z + 0.3f;
    vbo[4 * index + 3] = 1.0f;
  }
}

/**
* Wrapper for call to the kernCopyboidsToVBO CUDA kernel.
*/
void Boids::copyBoidsToVBO(float *vbodptr_positions, float *vbodptr_velocities) {
  dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);

  kernCopyPositionsToVBO << <fullBlocksPerGrid, blockSize >> >(numObjects, dev_pos, vbodptr_positions, scene_scale);
  kernCopyVelocitiesToVBO << <fullBlocksPerGrid, blockSize >> >(numObjects, dev_vel1, vbodptr_velocities, scene_scale);

  checkCUDAErrorWithLine("copyBoidsToVBO failed!");

  hipDeviceSynchronize();
}


/******************
* stepSimulation *
******************/

/**
* LOOK-1.2 You can use this as a helper for kernUpdateVelocityBruteForce.
* __device__ code can be called from a __global__ context
* Compute the new velocity on the body with index `iSelf` due to the `N` boids
* in the `pos` and `vel` arrays.
*/
__device__ glm::vec3 computeVelocityChange(int N, int iSelf, const glm::vec3 *pos, const glm::vec3 *vel) {
  // Rule 1: boids fly towards their local perceived center of mass, which excludes themselves
  // Rule 2: boids try to stay a distance d away from each other
  // Rule 3: boids try to match the speed of surrounding boids
  return glm::vec3(0.0f, 0.0f, 0.0f);
}

// return the Euclidean distance between two 3D points
__device__ float computeDistance(glm::vec3 pos1, glm::vec3 pos2) {
  float dX = pos2.x - pos1.x;
  float dY = pos2.y - pos1.y;
  float dZ = pos2.z - pos1.z;

  return sqrtf(dX * dX + dY * dY + dZ * dZ);
}

/**
* TODO-1.2 implement basic flocking
* For each of the `N` bodies, update its position based on its current velocity.
*/
__global__ void kernUpdateVelocityBruteForce(int N, glm::vec3 *pos,
  glm::vec3 *vel1, glm::vec3 *vel2) {
  // Compute a new velocity based on pos and vel1
  // Clamp the speed
  // Record the new velocity into vel2. Question: why NOT vel1?

  // this code is heavily inspired by the provided 2D example
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index >= N) return;

  glm::vec3 currPos = pos[index];

  int neighborCountCenter = 0;
  glm::vec3 centerPos = glm::vec3(0.f);
  
  glm::vec3 separatePos = glm::vec3(0.f);

  int neighborCountCohesion = 0;
  glm::vec3 cohesionVel = glm::vec3(0.f);

  for (int j = 0; j < N; j++) {
    if (j == index) continue;

    float distance = computeDistance(pos[j], currPos);
    // Rule 1: Cohesion: boids fly towards the center of mass of neighboring boids
    if (distance < rule1Distance) {
      neighborCountCenter++;
      centerPos += pos[j];
    }

    // Rule 2: Separation: boids try to keep a small distance away from each other
    if (distance < rule2Distance) separatePos -= pos[j] - currPos;

    // Rule 3: Alignment: boids try to match the velocities of neighboring boids
    if (distance < rule3Distance) {
      neighborCountCohesion++;
      cohesionVel += vel1[j];
    }
  }

  glm::vec3 deltaVel = vel1[index];
  if (neighborCountCenter) {
    centerPos = centerPos / (float)neighborCountCenter;
    deltaVel += (centerPos - currPos) * rule1Scale;
  }
  if (neighborCountCohesion) {
    cohesionVel = cohesionVel / (float)neighborCountCohesion;
    deltaVel += cohesionVel * rule3Scale;
  }
  deltaVel += separatePos * rule2Scale;

  float speed = sqrtf(dot(deltaVel, deltaVel));
  if (speed > maxSpeed) deltaVel = (deltaVel / speed) * maxSpeed;

  vel2[index] = deltaVel;
}

/**
* LOOK-1.2 Since this is pretty trivial, we implemented it for you.
* For each of the `N` bodies, update its position based on its current velocity.
*/
__global__ void kernUpdatePos(int N, float dt, glm::vec3 *pos, glm::vec3 *vel) {
  // Update position by velocity
  int index = threadIdx.x + (blockIdx.x * blockDim.x);
  if (index >= N) {
    return;
  }
  glm::vec3 thisPos = pos[index];
  thisPos += vel[index] * dt;

  // Wrap the boids around so we don't lose them
  thisPos.x = thisPos.x < -scene_scale ? scene_scale : thisPos.x;
  thisPos.y = thisPos.y < -scene_scale ? scene_scale : thisPos.y;
  thisPos.z = thisPos.z < -scene_scale ? scene_scale : thisPos.z;

  thisPos.x = thisPos.x > scene_scale ? -scene_scale : thisPos.x;
  thisPos.y = thisPos.y > scene_scale ? -scene_scale : thisPos.y;
  thisPos.z = thisPos.z > scene_scale ? -scene_scale : thisPos.z;

  pos[index] = thisPos;
}

// LOOK-2.1 Consider this method of computing a 1D index from a 3D grid index.
// LOOK-2.3 Looking at this method, what would be the most memory efficient
//          order for iterating over neighboring grid cells?
//          for(x)
//            for(y)
//             for(z)? Or some other order?
__device__ int gridIndex3Dto1D(int x, int y, int z, int gridResolution) {
  return x + y * gridResolution + z * gridResolution * gridResolution;
}

__global__ void kernComputeIndices(int N, int gridResolution,
  glm::vec3 gridMin, float inverseCellWidth,
  glm::vec3 *pos, int *indices, int *gridIndices) {
    // TODO-2.1
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= N) return;

    glm::vec3 normalizedPos = (pos[index] - gridMin) * inverseCellWidth;

    // - Label each boid with the index of its grid cell.
    gridIndices[index] = gridIndex3Dto1D((int)normalizedPos.x, 
        (int)normalizedPos.y, (int)normalizedPos.z, gridResolution);

    // - Set up a parallel array of integer indices as pointers to the actual
    //   boid data in pos and vel1/vel2
    indices[index] = index;
}

// LOOK-2.1 Consider how this could be useful for indicating that a cell
//          does not enclose any boids
// value == -1 means no boids
__global__ void kernResetIntBuffer(int N, int *intBuffer, int value) {
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (index < N) {
    intBuffer[index] = value;
  }
}

__global__ void kernIdentifyCellStartEnd(int N, int *particleGridIndices,
  int *gridCellStartIndices, int *gridCellEndIndices) {
  // TODO-2.1
  // Identify the start point of each cell in the gridIndices array.
  // This is basically a parallel unrolling of a loop that goes
  // "this index doesn't match the one before it, must be a new cell!"
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= N) return;

    // need to have as separate cases instead of if/else for edge cases
    if (index == 0) 
        gridCellStartIndices[particleGridIndices[index]] = 0;
    
    if (index == N - 1)
        gridCellEndIndices[particleGridIndices[index]] = N - 1;
    
    if (index != N - 1 && particleGridIndices[index] != particleGridIndices[index + 1]) {
        gridCellEndIndices[particleGridIndices[index]] = index;
        gridCellStartIndices[particleGridIndices[index + 1]] = index + 1;
    }
}

__global__ void kernUpdateVelNeighborSearchScattered(
  int N, int gridResolution, glm::vec3 gridMin,
  float inverseCellWidth, float cellWidth,
  int *gridCellStartIndices, int *gridCellEndIndices,
  int *particleArrayIndices,
  glm::vec3 *pos, glm::vec3 *vel1, glm::vec3 *vel2) {
  // TODO-2.1 - Update a boid's velocity using the uniform grid to reduce
  // the number of boids that need to be checked.

    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= N) return;

    int neighborCountCenter = 0;
    glm::vec3 centerPos = glm::vec3(0.f);

    glm::vec3 separatePos = glm::vec3(0.f);

    int neighborCountCohesion = 0;
    glm::vec3 cohesionVel = glm::vec3(0.f);

    // - Identify the grid cell that this particle is in
    glm::vec3 currPos = pos[index];

    // - Identify which cells may contain neighbors. This isn't always 8.
    // Extra Credit
    // set lower/upper bounds on neighboring cell exploration per axis
    // at minimum it's based on the number of cells that fit in max radius
    float radiusOfInflu = max(max(rule1Distance, rule2Distance), rule3Distance) * inverseCellWidth;
    glm::vec3 normalizedPos = (currPos - gridMin) * inverseCellWidth;
    int dzlb = (int)floorf(normalizedPos.z - radiusOfInflu),
        dylb = (int)floorf(normalizedPos.y - radiusOfInflu),
        dxlb = (int)floorf(normalizedPos.x - radiusOfInflu),
        dzub = (int)floorf(normalizedPos.z + radiusOfInflu) + 1,
        dyub = (int)floorf(normalizedPos.y + radiusOfInflu) + 1,
        dxub = (int)floorf(normalizedPos.x + radiusOfInflu) + 1;

    // skip out of bounds neighboring cell indices
    dzlb = imin(imax(dzlb, 0), gridResolution - 1);
    dzub = imin(imax(dzub, 0), gridResolution - 1);
    dylb = imin(imax(dylb, 0), gridResolution - 1);
    dyub = imin(imax(dyub, 0), gridResolution - 1);
    dxlb = imin(imax(dxlb, 0), gridResolution - 1);
    dxub = imin(imax(dxub, 0), gridResolution - 1);

    // now guaranteed currX, currY, currZ in bounds
    int currX, currY, currZ, startIndex, endIndex, 
        neighborIndex, cellBoidIndex, actualBoidIndex;
    float distance;
    for (currZ = dzlb; currZ < dzub; currZ++) {
        for (currY = dylb; currY < dyub; currY++) {
            for (currX = dxlb; currX < dxub; currX++) {
                neighborIndex = gridIndex3Dto1D(currX, currY, currZ, gridResolution);
                // skip to next iteration if no boid in this cell
                if (gridCellStartIndices[neighborIndex] == -1) continue;

                // - For each cell, read the start/end indices in the boid pointer array.
                startIndex = gridCellStartIndices[neighborIndex];
                endIndex = gridCellEndIndices[neighborIndex];

                // - Access each boid in the cell and compute velocity change from
                //   the boids rules, if this boid is within the neighborhood distance.
                for (cellBoidIndex = startIndex; cellBoidIndex <= endIndex; cellBoidIndex++) {
                    actualBoidIndex = particleArrayIndices[cellBoidIndex];
                    if (actualBoidIndex == index) continue;

                    distance = computeDistance(pos[actualBoidIndex], currPos);
                    // Rule 1: Cohesion: boids fly towards the center of mass of neighboring boids
                    if (distance < rule1Distance) {
                        neighborCountCenter++;
                        centerPos += pos[actualBoidIndex];
                    }

                    // Rule 2: Separation: boids try to keep a small distance away from each other
                    if (distance < rule2Distance) separatePos -= pos[actualBoidIndex] - currPos;

                    // Rule 3: Alignment: boids try to match the velocities of neighboring boids
                    if (distance < rule3Distance) {
                        neighborCountCohesion++;
                        cohesionVel += vel1[actualBoidIndex];
                    }
                }
            }
        }
    }

    glm::vec3 deltaVel = vel1[index];
    if (neighborCountCenter) {
        centerPos = centerPos / (float)neighborCountCenter;
        deltaVel += (centerPos - currPos) * rule1Scale;
    }
    if (neighborCountCohesion) {
        cohesionVel = cohesionVel / (float)neighborCountCohesion;
        deltaVel += cohesionVel * rule3Scale;
    }
    deltaVel += separatePos * rule2Scale;

    // - Clamp the speed change before putting the new speed in vel2
    float speed = sqrtf(dot(deltaVel, deltaVel));
    if (speed > maxSpeed) deltaVel = (deltaVel / speed) * maxSpeed;

    vel2[index] = deltaVel;
}

__global__ void kernUpdateVelNeighborSearchCoherent(
  int N, int gridResolution, glm::vec3 gridMin,
  float inverseCellWidth, float cellWidth,
  int *gridCellStartIndices, int *gridCellEndIndices,
  glm::vec3 *pos, glm::vec3 *vel1, glm::vec3 *vel2) {
  // TODO-2.3 - This should be very similar to kernUpdateVelNeighborSearchScattered,
  // except with one less level of indirection.
  // This should expect gridCellStartIndices and gridCellEndIndices to refer
  // directly to pos and vel1.

    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= N) return;

    int neighborCountCenter = 0;
    glm::vec3 centerPos = glm::vec3(0.f);

    glm::vec3 separatePos = glm::vec3(0.f);

    int neighborCountCohesion = 0;
    glm::vec3 cohesionVel = glm::vec3(0.f);

    // - Identify the grid cell that this particle is in
    glm::vec3 currPos = pos[index];

    // - Identify which cells may contain neighbors. This isn't always 8.
    // Extra Credit
    // set lower/upper bounds on neighboring cell exploration per axis
    // at minimum it's based on the number of cells that fit in max radius
    float radiusOfInflu = max(max(rule1Distance, rule2Distance), rule3Distance) * inverseCellWidth;
    glm::vec3 normalizedPos = (currPos - gridMin) * inverseCellWidth;
    int dzlb = (int)floorf(normalizedPos.z - radiusOfInflu),
        dylb = (int)floorf(normalizedPos.y - radiusOfInflu),
        dxlb = (int)floorf(normalizedPos.x - radiusOfInflu),
        dzub = (int)floorf(normalizedPos.z + radiusOfInflu) + 1,
        dyub = (int)floorf(normalizedPos.y + radiusOfInflu) + 1,
        dxub = (int)floorf(normalizedPos.x + radiusOfInflu) + 1;

    // skip out of bounds neighboring cell indices
    dzlb = imin(imax(dzlb, 0), gridResolution - 1);
    dzub = imin(imax(dzub, 0), gridResolution - 1);
    dylb = imin(imax(dylb, 0), gridResolution - 1);
    dyub = imin(imax(dyub, 0), gridResolution - 1);
    dxlb = imin(imax(dxlb, 0), gridResolution - 1);
    dxub = imin(imax(dxub, 0), gridResolution - 1);

    // now guaranteed currX, currY, currZ in bounds
    int currX, currY, currZ, startIndex, endIndex,
        neighborIndex, cellBoidIndex, actualBoidIndex;
    float distance;
    for (currZ = dzlb; currZ < dzub; currZ++) {
        for (currY = dylb; currY < dyub; currY++) {
            for (currX = dxlb; currX < dxub; currX++) {
                neighborIndex = gridIndex3Dto1D(currX, currY, currZ, gridResolution);
                // skip to next iteration if no boid in this cell
                if (gridCellStartIndices[neighborIndex] == -1) continue;

                // - For each cell, read the start/end indices in the boid pointer array.
                //   DIFFERENCE: For best results, consider what order the cells should be
                //   checked in to maximize the memory benefits of reordering the boids data.
                startIndex = gridCellStartIndices[neighborIndex];
                endIndex = gridCellEndIndices[neighborIndex];

                // - Access each boid in the cell and compute velocity change from
                //   the boids rules, if this boid is within the neighborhood distance.
                for (actualBoidIndex = startIndex; cellBoidIndex <= endIndex; cellBoidIndex++) {
                    if (actualBoidIndex == index) continue;

                    distance = computeDistance(pos[actualBoidIndex], currPos);
                    // Rule 1: Cohesion: boids fly towards the center of mass of neighboring boids
                    if (distance < rule1Distance) {
                        neighborCountCenter++;
                        centerPos += pos[actualBoidIndex];
                    }

                    // Rule 2: Separation: boids try to keep a small distance away from each other
                    if (distance < rule2Distance) separatePos -= pos[actualBoidIndex] - currPos;

                    // Rule 3: Alignment: boids try to match the velocities of neighboring boids
                    if (distance < rule3Distance) {
                        neighborCountCohesion++;
                        cohesionVel += vel1[actualBoidIndex];
                    }
                }
            }
        }
    }

    glm::vec3 deltaVel = vel1[index];
    if (neighborCountCenter) {
        centerPos = centerPos / (float)neighborCountCenter;
        deltaVel += (centerPos - currPos) * rule1Scale;
    }
    if (neighborCountCohesion) {
        cohesionVel = cohesionVel / (float)neighborCountCohesion;
        deltaVel += cohesionVel * rule3Scale;
    }
    deltaVel += separatePos * rule2Scale;

    // - Clamp the speed change before putting the new speed in vel2
    float speed = sqrtf(dot(deltaVel, deltaVel));
    if (speed > maxSpeed) deltaVel = (deltaVel / speed) * maxSpeed;

    vel2[index] = deltaVel;
}

// 
__global__ void kernReshufflePosVel(int N, int* particleArrayIndices,
    glm::vec3* pos, glm::vec3* vel, glm::vec3* cohPos, glm::vec3* cohVel) {

    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= N) return;

    cohPos[index] = pos[particleArrayIndices[index]];
    cohVel[index] = vel[particleArrayIndices[index]];
}

/**
* Step the entire N-body simulation by `dt` seconds.
*/
void Boids::stepSimulationNaive(float dt) {
  // TODO-1.2 - use the kernels you wrote to step the simulation forward in time.
  // TODO-1.2 ping-pong the velocity buffers
  dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);
  kernUpdateVelocityBruteForce<<<fullBlocksPerGrid, blockSize>>>(numObjects,
    dev_pos, dev_vel1, dev_vel2);
  checkCUDAErrorWithLine("kernUpdateVelocityBruteForce failed!");

  kernUpdatePos<<<fullBlocksPerGrid, blockSize>>> (numObjects,
      dt, dev_pos, dev_vel2);
  std::swap(dev_vel1, dev_vel2);
}

void Boids::stepSimulationScatteredGrid(float dt) {
  // TODO-2.1
  // Uniform Grid Neighbor search using Thrust sort.
  // In Parallel:
    dim3 gridSizePerBoid((numObjects + blockSize - 1) / blockSize);
    dim3 gridSizePerCell((gridCellCount + blockSize - 1) / blockSize);

    // - label each particle with its array index as well as its grid index.
    //   Use 2x width grids.
    // reset buffers in case of boid movements
    kernResetIntBuffer<<<gridSizePerCell, blockSize>>>(gridCellCount, 
        dev_gridCellStartIndices, -1);
    kernResetIntBuffer<<<gridSizePerCell, blockSize>>>(gridCellCount, 
        dev_gridCellEndIndices, -1);

    // compute new grid indices each boid belongs to
    kernComputeIndices<<<gridSizePerBoid, blockSize>>>(numObjects, 
        gridSideCount, gridMinimum, gridInverseCellWidth, dev_pos, 
        dev_particleArrayIndices, dev_particleGridIndices);

    // - Unstable key sort using Thrust. A stable sort isn't necessary, but you
    //   are welcome to do a performance comparison.
    dev_thrust_particleArrayIndices = thrust::device_ptr<int>(dev_particleArrayIndices);
    dev_thrust_particleGridIndices = thrust::device_ptr<int>(dev_particleGridIndices);
    thrust::sort_by_key(dev_thrust_particleGridIndices, 
        dev_thrust_particleGridIndices + numObjects, 
        dev_thrust_particleArrayIndices);

    // - Naively unroll the loop for finding the start and end indices of each
    //   cell's data pointers in the array of boid indices
    kernIdentifyCellStartEnd<<<gridSizePerBoid, blockSize>>>(
        numObjects, dev_particleGridIndices, 
        dev_gridCellStartIndices, dev_gridCellEndIndices);

    // - Perform velocity updates using neighbor search
    kernUpdateVelNeighborSearchScattered<<<gridSizePerBoid, blockSize>>>(
        numObjects, gridSideCount, gridMinimum, gridInverseCellWidth, 
        gridCellWidth, dev_gridCellStartIndices, dev_gridCellEndIndices, 
        dev_particleArrayIndices, dev_pos, dev_vel1, dev_vel2);

    // - Update positions
    kernUpdatePos<<<gridSizePerBoid, blockSize>>>(numObjects, dt, dev_pos, dev_vel2);

    // - Ping-pong buffers as needed
    std::swap(dev_vel1, dev_vel2);
}

void Boids::stepSimulationCoherentGrid(float dt) {
  // TODO-2.3 - start by copying Boids::stepSimulationNaiveGrid
  // Uniform Grid Neighbor search using Thrust sort on cell-coherent data.
  // In Parallel:

    dim3 gridSizePerBoid((numObjects + blockSize - 1) / blockSize);
    dim3 gridSizePerCell((gridCellCount + blockSize - 1) / blockSize);

    // - Label each particle with its array index as well as its grid index.
    //   Use 2x width grids
    // reset buffers in case of boid movements
    kernResetIntBuffer<<<gridSizePerCell, blockSize>>>(gridCellCount,
        dev_gridCellStartIndices, -1);
    kernResetIntBuffer<<<gridSizePerCell, blockSize>>>(gridCellCount,
        dev_gridCellEndIndices, -1);

    // compute new grid indices each boid belongs to
    kernComputeIndices<<<gridSizePerBoid, blockSize>>>(numObjects,
        gridSideCount, gridMinimum, gridInverseCellWidth, dev_pos,
        dev_particleArrayIndices, dev_particleGridIndices);

    // - Unstable key sort using Thrust. A stable sort isn't necessary, but you
    //   are welcome to do a performance comparison.
    dev_thrust_particleArrayIndices = thrust::device_ptr<int>(dev_particleArrayIndices);
    dev_thrust_particleGridIndices = thrust::device_ptr<int>(dev_particleGridIndices);
    thrust::sort_by_key(dev_thrust_particleGridIndices,
        dev_thrust_particleGridIndices + numObjects,
        dev_thrust_particleArrayIndices);

    // - Naively unroll the loop for finding the start and end indices of each
    //   cell's data pointers in the array of boid indices
    kernIdentifyCellStartEnd<<<gridSizePerBoid, blockSize>>>(
        numObjects, dev_particleGridIndices,
        dev_gridCellStartIndices, dev_gridCellEndIndices);

    // - BIG DIFFERENCE: use the rearranged array index buffer to reshuffle all
    //   the particle data in the simulation array.
    //   CONSIDER WHAT ADDITIONAL BUFFERS YOU NEED
    kernReshufflePosVel<<<gridSizePerBoid, blockSize>>>(numObjects, 
        dev_particleArrayIndices, dev_pos, dev_vel1, dev_coherentPos, dev_coherentVel);

    // - Perform velocity updates using neighbor search
    kernUpdateVelNeighborSearchScattered<<<gridSizePerBoid, blockSize>>>(
        numObjects, gridSideCount, gridMinimum, gridInverseCellWidth,
        gridCellWidth, dev_gridCellStartIndices, dev_gridCellEndIndices,
        dev_particleArrayIndices, dev_coherentPos, dev_coherentVel, dev_vel2);

    // - Update positions
    kernUpdatePos<<<gridSizePerBoid, blockSize>>>(numObjects, dt, dev_coherentPos, dev_vel2);

    // - Ping-pong buffers as needed. THIS MAY BE DIFFERENT FROM BEFORE.
    std::swap(dev_vel1, dev_vel2);
    std::swap(dev_pos, dev_coherentPos);
}

void Boids::endSimulation() {
  hipFree(dev_vel1);
  hipFree(dev_vel2);
  hipFree(dev_pos);

  // TODO-2.1 TODO-2.3 - Free any additional buffers here.
  hipFree(dev_particleArrayIndices);
  hipFree(dev_particleGridIndices);
  hipFree(dev_gridCellStartIndices);
  hipFree(dev_gridCellEndIndices);

  hipFree(dev_coherentPos);
  hipFree(dev_coherentVel);
}

void Boids::unitTest() {
  // LOOK-1.2 Feel free to write additional tests here.

  // test unstable sort
  int *dev_intKeys;
  int *dev_intValues;
  int N = 10;

  std::unique_ptr<int[]>intKeys{ new int[N] };
  std::unique_ptr<int[]>intValues{ new int[N] };

  intKeys[0] = 0; intValues[0] = 0;
  intKeys[1] = 1; intValues[1] = 1;
  intKeys[2] = 0; intValues[2] = 2;
  intKeys[3] = 3; intValues[3] = 3;
  intKeys[4] = 0; intValues[4] = 4;
  intKeys[5] = 2; intValues[5] = 5;
  intKeys[6] = 2; intValues[6] = 6;
  intKeys[7] = 0; intValues[7] = 7;
  intKeys[8] = 5; intValues[8] = 8;
  intKeys[9] = 6; intValues[9] = 9;

  hipMalloc((void**)&dev_intKeys, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_intKeys failed!");

  hipMalloc((void**)&dev_intValues, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_intValues failed!");

  dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);

  std::cout << "before unstable sort: " << std::endl;
  for (int i = 0; i < N; i++) {
    std::cout << "  key: " << intKeys[i];
    std::cout << " value: " << intValues[i] << std::endl;
  }

  // How to copy data to the GPU
  hipMemcpy(dev_intKeys, intKeys.get(), sizeof(int) * N, hipMemcpyHostToDevice);
  hipMemcpy(dev_intValues, intValues.get(), sizeof(int) * N, hipMemcpyHostToDevice);

  // Wrap device vectors in thrust iterators for use with thrust.
  thrust::device_ptr<int> dev_thrust_keys(dev_intKeys);
  thrust::device_ptr<int> dev_thrust_values(dev_intValues);
  // LOOK-2.1 Example for using thrust::sort_by_key
  thrust::sort_by_key(dev_thrust_keys, dev_thrust_keys + N, dev_thrust_values);

  // How to copy data back to the CPU side from the GPU
  hipMemcpy(intKeys.get(), dev_intKeys, sizeof(int) * N, hipMemcpyDeviceToHost);
  hipMemcpy(intValues.get(), dev_intValues, sizeof(int) * N, hipMemcpyDeviceToHost);
  checkCUDAErrorWithLine("memcpy back failed!");

  std::cout << "after unstable sort: " << std::endl;
  for (int i = 0; i < N; i++) {
    std::cout << "  key: " << intKeys[i];
    std::cout << " value: " << intValues[i] << std::endl;
  }

  // cleanup
  hipFree(dev_intKeys);
  hipFree(dev_intValues);
  checkCUDAErrorWithLine("hipFree failed!");
  return;
}
