#include "hip/hip_runtime.h"
#define GLM_FORCE_CUDA
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <glm/glm.hpp>
#include "utilityCore.hpp"
#include "kernel.h"

// LOOK-2.1 potentially useful for doing grid-based neighbor search
#ifndef imax
#define imax( a, b ) ( ((a) > (b)) ? (a) : (b) )
#endif

#ifndef imin
#define imin( a, b ) ( ((a) < (b)) ? (a) : (b) )
#endif

#define checkCUDAErrorWithLine(msg) checkCUDAError(msg, __LINE__)

/**
* Check for CUDA errors; print and exit if there was a problem.
*/
void checkCUDAError(const char *msg, int line = -1) {
  hipError_t err = hipGetLastError();
  if (hipSuccess != err) {
    if (line >= 0) {
      fprintf(stderr, "Line %d: ", line);
    }
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}


/*****************
* Configuration *
*****************/

/*! Block size used for CUDA kernel launch. */
#define blockSize 128

// LOOK-1.2 Parameters for the boids algorithm.
// These worked well in our reference implementation.
#define rule1Distance 5.0f
#define rule2Distance 3.0f
#define rule3Distance 5.0f

#define rule1Scale 0.01f
#define rule2Scale 0.1f
#define rule3Scale 0.1f

#define maxSpeed 1.0f

/*! Size of the starting area in simulation space. */
#define scene_scale 100.0f

/*! Ration of width of each cell to neighborhood distance. (either 1 or 2) */
#define CELL_WIDTH_MAX_DIST_RATIO 2

/***********************************************
* Kernel state (pointers are device pointers) *
***********************************************/

int numObjects;
dim3 threadsPerBlock(blockSize);

// LOOK-1.2 - These buffers are here to hold all your boid information.
// These get allocated for you in Boids::initSimulation.
// Consider why you would need two velocity buffers in a simulation where each
// boid cares about its neighbors' velocities.
// These are called ping-pong buffers.
glm::vec3 *dev_pos;
glm::vec3 *dev_vel1;
glm::vec3 *dev_vel2;

// LOOK-2.1 - these are NOT allocated for you. You'll have to set up the thrust
// pointers on your own too.

// For efficient sorting and the uniform grid. These should always be parallel.
int *dev_particleArrayIndices; // What index in dev_pos and dev_velX represents this particle?
int *dev_particleGridIndices; // What grid cell is this particle in?
// needed for use with thrust
thrust::device_ptr<int> dev_thrust_particleArrayIndices;
thrust::device_ptr<int> dev_thrust_particleGridIndices;

int *dev_gridCellStartIndices; // What part of dev_particleArrayIndices belongs
int *dev_gridCellEndIndices;   // to this cell?

// TODO-2.3 - consider what additional buffers you might need to reshuffle
// the position and velocity data to be coherent within cells.

// LOOK-2.1 - Grid parameters based on simulation parameters.
// These are automatically computed for you in Boids::initSimulation
int gridCellCount;
int gridSideCount;
float gridCellWidth;
float gridInverseCellWidth;
glm::vec3 gridMinimum;

/******************
* initSimulation *
******************/

__host__ __device__ unsigned int hash(unsigned int a) {
  a = (a + 0x7ed55d16) + (a << 12);
  a = (a ^ 0xc761c23c) ^ (a >> 19);
  a = (a + 0x165667b1) + (a << 5);
  a = (a + 0xd3a2646c) ^ (a << 9);
  a = (a + 0xfd7046c5) + (a << 3);
  a = (a ^ 0xb55a4f09) ^ (a >> 16);
  return a;
}

/**
* LOOK-1.2 - this is a typical helper function for a CUDA kernel.
* Function for generating a random vec3.
*/
__host__ __device__ glm::vec3 generateRandomVec3(float time, int index) {
  thrust::default_random_engine rng(hash((int)(index * time)));
  thrust::uniform_real_distribution<float> unitDistrib(-1, 1);

  return glm::vec3((float)unitDistrib(rng), (float)unitDistrib(rng), (float)unitDistrib(rng));
}

/**
* LOOK-1.2 - This is a basic CUDA kernel.
* CUDA kernel for generating boids with a specified mass randomly around the star.
*/
__global__ void kernGenerateRandomPosArray(int time, int N, glm::vec3 * arr, float scale) {
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (index < N) {
    glm::vec3 rand = generateRandomVec3(time, index);
    arr[index].x = scale * rand.x;
    arr[index].y = scale * rand.y;
    arr[index].z = scale * rand.z;
  }
}

/**
* Initialize memory, update some globals
*/
void Boids::initSimulation(int N) {
  numObjects = N;
  dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);

  // LOOK-1.2 - This is basic CUDA memory management and error checking.
  // Don't forget to hipFree in  Boids::endSimulation.
  hipMalloc((void**)&dev_pos, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_pos failed!");

  hipMalloc((void**)&dev_vel1, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_vel1 failed!");

  hipMalloc((void**)&dev_vel2, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_vel2 failed!");

  // LOOK-1.2 - This is a typical CUDA kernel invocation.
  kernGenerateRandomPosArray<<<fullBlocksPerGrid, blockSize>>>(1, numObjects,
    dev_pos, scene_scale);
  checkCUDAErrorWithLine("kernGenerateRandomPosArray failed!");

  // LOOK-2.1 computing grid params

  gridCellWidth = float(CELL_WIDTH_MAX_DIST_RATIO) * std::max(std::max(rule1Distance, rule2Distance), rule3Distance);
  int halfSideCount = (int)(scene_scale / gridCellWidth) + 1;
  gridSideCount = 2 * halfSideCount;

  gridCellCount = gridSideCount * gridSideCount * gridSideCount;
  gridInverseCellWidth = 1.0f / gridCellWidth;
  float halfGridWidth = gridCellWidth * halfSideCount;
  gridMinimum.x -= halfGridWidth;
  gridMinimum.y -= halfGridWidth;
  gridMinimum.z -= halfGridWidth;

  // TODO-2.1 TODO-2.3 - Allocate additional buffers here.
  hipMalloc((void**)&dev_gridCellStartIndices, gridCellCount * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_gridCellStartIndices failed!");

  hipMalloc((void**)&dev_gridCellEndIndices, gridCellCount * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_gridCellEndIndices failed!");

  hipMalloc((void**)&dev_particleGridIndices, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_particleGridIndices failed!");

  hipMalloc((void**)&dev_particleArrayIndices, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_particleArrayIndices failed!");

  dev_thrust_particleArrayIndices = thrust::device_ptr<int>(dev_particleArrayIndices);
  dev_thrust_particleGridIndices = thrust::device_ptr<int>(dev_particleGridIndices);

  hipDeviceSynchronize();
}


/******************
* copyBoidsToVBO *
******************/

/**
* Copy the boid positions into the VBO so that they can be drawn by OpenGL.
*/
__global__ void kernCopyPositionsToVBO(int N, glm::vec3 *pos, float *vbo, float s_scale) {
  int index = threadIdx.x + (blockIdx.x * blockDim.x);

  float c_scale = -1.0f / s_scale;

  if (index < N) {
    vbo[4 * index + 0] = pos[index].x * c_scale;
    vbo[4 * index + 1] = pos[index].y * c_scale;
    vbo[4 * index + 2] = pos[index].z * c_scale;
    vbo[4 * index + 3] = 1.0f;
  }
}

__global__ void kernCopyVelocitiesToVBO(int N, glm::vec3 *vel, float *vbo, float s_scale) {
  int index = threadIdx.x + (blockIdx.x * blockDim.x);

  if (index < N) {
    vbo[4 * index + 0] = vel[index].x + 0.3f;
    vbo[4 * index + 1] = vel[index].y + 0.3f;
    vbo[4 * index + 2] = vel[index].z + 0.3f;
    vbo[4 * index + 3] = 1.0f;
  }
}

/**
* Wrapper for call to the kernCopyboidsToVBO CUDA kernel.
*/
void Boids::copyBoidsToVBO(float *vbodptr_positions, float *vbodptr_velocities) {
  dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);

  kernCopyPositionsToVBO << <fullBlocksPerGrid, blockSize >> >(numObjects, dev_pos, vbodptr_positions, scene_scale);
  kernCopyVelocitiesToVBO << <fullBlocksPerGrid, blockSize >> >(numObjects, dev_vel1, vbodptr_velocities, scene_scale);

  checkCUDAErrorWithLine("copyBoidsToVBO failed!");

  hipDeviceSynchronize();
}


/******************
* stepSimulation *
******************/

/**
* LOOK-1.2 You can use this as a helper for kernUpdateVelocityBruteForce.
* __device__ code can be called from a __global__ context
* Compute the new velocity on the body with index `iSelf` due to the `N` boids
* in the `pos` and `vel` arrays.
*/
__device__ glm::vec3 computeVelocityChange(int N, int iSelf, const glm::vec3 *pos, const glm::vec3 *vel) {
 
    glm::vec3 perceived_center = glm::vec3(0.0f);
    glm::vec3 c = glm::vec3(0.0f);
    glm::vec3 perceived_velocity = glm::vec3(0.0f);

    int neighbors1 = 0, neighbors3 = 0;

    for (int i = 0; i < N; i++) {

        if (i == iSelf) {
            continue;
        }

        float dist = glm::distance(pos[i], pos[iSelf]);

      // Rule 1: boids fly towards their local perceived center of mass, which excludes themselves
        if (dist < rule1Distance) {
            perceived_center += pos[i];
            neighbors1++;
        }

      // Rule 2: boids try to stay a distance d away from each other
        if (dist < rule2Distance) {
            c -= pos[i] - pos[iSelf];
        }

      // Rule 3: boids try to match the speed of surrounding boids
        if (dist < rule3Distance) {
            perceived_velocity += vel[i];
            neighbors3++;
        }
    }

    if (neighbors1 > 0)
        perceived_center /= neighbors1;

    glm::vec3 v1 = (perceived_center - pos[iSelf]) * rule1Scale;
    glm::vec3 v2 = c * rule2Scale;

    if (neighbors3 > 0)
        perceived_velocity /= neighbors3;

    glm::vec3 v3 = perceived_velocity * rule3Scale;
 
    return v1 + v2 + v3;
}

/**
* TODO-1.2 implement basic flocking
* For each of the `N` bodies, update its position based on its current velocity.
*/
__global__ void kernUpdateVelocityBruteForce(int N, glm::vec3 *pos,
  glm::vec3 *vel1, glm::vec3 *vel2) {
  // Compute a new velocity based on pos and vel1

    int index = threadIdx.x + (blockIdx.x * blockDim.x);
    if (index >= N) {
        return;
    }

    glm::vec3 v_new = vel1[index] + computeVelocityChange(N, index, pos, vel1);

    // Clamp the speed
    if (glm::length(v_new) > maxSpeed) {
        v_new = glm::normalize(v_new) * maxSpeed;
    }

    // Record the new velocity into vel2. Question: why NOT vel1?
    vel2[index] = v_new;
}

/**
* LOOK-1.2 Since this is pretty trivial, we implemented it for you.
* For each of the `N` bodies, update its position based on its current velocity.
*/
__global__ void kernUpdatePos(int N, float dt, glm::vec3 *pos, glm::vec3 *vel) {
  // Update position by velocity
  int index = threadIdx.x + (blockIdx.x * blockDim.x);
  if (index >= N) {
    return;
  }
  glm::vec3 thisPos = pos[index];
  thisPos += vel[index] * dt;

  // Wrap the boids around so we don't lose them
  thisPos.x = thisPos.x < -scene_scale ? scene_scale : thisPos.x;
  thisPos.y = thisPos.y < -scene_scale ? scene_scale : thisPos.y;
  thisPos.z = thisPos.z < -scene_scale ? scene_scale : thisPos.z;

  thisPos.x = thisPos.x > scene_scale ? -scene_scale : thisPos.x;
  thisPos.y = thisPos.y > scene_scale ? -scene_scale : thisPos.y;
  thisPos.z = thisPos.z > scene_scale ? -scene_scale : thisPos.z;

  pos[index] = thisPos;
}

// LOOK-2.1 Consider this method of computing a 1D index from a 3D grid index.
// LOOK-2.3 Looking at this method, what would be the most memory efficient
//          order for iterating over neighboring grid cells?
//          for(x)
//            for(y)
//             for(z)? Or some other order?
__device__ int gridIndex3Dto1D(int x, int y, int z, int gridResolution) {
  return x + y * gridResolution + z * gridResolution * gridResolution;
} 

__global__ void kernComputeIndices(int N, int gridResolution,
  glm::vec3 gridMin, float inverseCellWidth,
  glm::vec3 *pos, int *indices, int *gridIndices) {

    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= N) {
        return;
    }

    // - Set up a parallel array of integer indices as pointers to the actual
    //   boid data in pos and vel1/vel2
    indices[index] = index;

    // - Label each boid with the index of its grid cell.
    glm::ivec3 netPos3D = glm::floor((pos[index] - gridMin) *inverseCellWidth);
    gridIndices[index] = (gridIndex3Dto1D(netPos3D.x, netPos3D.y, netPos3D.z, gridResolution));
}

// LOOK-2.1 Consider how this could be useful for indicating that a cell
//          does not enclose any boids
__global__ void kernResetIntBuffer(int N, int *intBuffer, int value) {
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (index < N) {
    intBuffer[index] = value;
  }
}

__global__ void kernIdentifyCellStartEnd(int N, int *particleGridIndices,
  int *gridCellStartIndices, int *gridCellEndIndices) {
  // TODO-2.1
    dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);
  // Identify the start point of each cell in the gridIndices array.
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= N) {
        return;
    }

    int currGridIndex = particleGridIndices[index];

    if (index == 0){
        gridCellStartIndices[currGridIndex] = index;
        return;
    }

    if (index == N - 1) {
        gridCellEndIndices[currGridIndex] = index;
    }

    int prevGridIndex = particleGridIndices[index - 1];

    if (prevGridIndex != currGridIndex) {
        gridCellStartIndices[currGridIndex] = index;
        gridCellEndIndices[currGridIndex - 1] = index - 1;
    } 
  // This is basically a parallel unrolling of a loop that goes
  // "this index doesn't match the one before it, must be a new cell!"
}

__global__ void kernUpdateVelNeighborSearchScattered(
  int N, int gridResolution, glm::vec3 gridMin,
  float inverseCellWidth, float cellWidth,
  int *gridCellStartIndices, int *gridCellEndIndices,
  int *particleArrayIndices,
  glm::vec3 *pos, glm::vec3 *vel1, glm::vec3 *vel2) {
  // TODO-2.1 - Update a boid's velocity using the uniform grid to reduce
  // the number of boids that need to be checked.
  // - Identify the grid cell that this particle is in
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (index >= N) {
        return;
    }

    glm::vec3 currPos = pos[index];

    // - Label each boid with the index of its grid cell.
    glm::ivec3 gridIdx3D = glm::floor((currPos - gridMin) * inverseCellWidth);

    // - Identify which cells may contain neighbors. This isn't always 8.
    glm::vec3 cellCenter3D = (glm::vec3(gridIdx3D) + 0.5f) * cellWidth + gridMin;
    glm::ivec3 startIdx;
    glm::ivec3 endIdx;
    if (CELL_WIDTH_MAX_DIST_RATIO == 1) {
        startIdx = glm::ivec3(-1);
        endIdx = glm::ivec3(1);
    }
    else {
        glm::bvec3 checkIdx = glm::lessThan(currPos, cellCenter3D);
        startIdx = -glm::ivec3(checkIdx);
        endIdx = 1 - glm::ivec3(checkIdx);
    }

    glm::vec3 perceived_center = glm::vec3(0.0f);
    glm::vec3 perceived_velocity = glm::vec3(0.0f);
    glm::vec3 c = glm::vec3(0.0f);
    int neighbors1 = 0;
    int neighbors3 = 0;

    // - For each cell, read the start/end indices in the boid pointer array.
    for (int i = startIdx.x; i <= endIdx.x; i++) {
        for (int j = startIdx.y; j <= endIdx.y; j++) {
            for (int k = startIdx.z; k <= endIdx.z; k++) {

                glm::ivec3 currGridIdx = gridIdx3D + glm::ivec3{ i,j,k };
                glm::ivec3 upperLimit = glm::ivec3(gridResolution - 1);

                if (!(upperLimit.x <= currGridIdx.x <= 0 || upperLimit.y <= currGridIdx.y <= 0 || upperLimit.z <= currGridIdx.z <= 0))
                    return;

                int gridIdx1D = gridIndex3Dto1D(currGridIdx.x, currGridIdx.y,
                    currGridIdx.z + k, gridResolution);

                // starting and ending indices of boids
                int startBoidIdx = gridCellStartIndices[gridIdx1D], endBoidIdx = gridCellEndIndices[gridIdx1D];

                // - Access each boid in the cell and compute velocity change from
                //   the boids rules, if this boid is within the neighborhood distance.
                if (startBoidIdx >= 0 && endBoidIdx <= N - 1) {
                    for (int* p = &particleArrayIndices[startBoidIdx];
                        p <= &particleArrayIndices[endBoidIdx]; p++) {
                        int idx = *p;

                        if (idx == index)
                            continue;

                        float dist = glm::distance(currPos, pos[idx]);

                        // Rule - 1

                        if (dist < rule1Distance) {
                            perceived_center += pos[idx];
                            neighbors1++;
                        }

                        // Rule 2: boids try to stay a distance d away from each other
                        if (dist < rule2Distance)
                            c -= (pos[idx] - currPos);

                        // Rule 3: boids try to match the speed of surrounding boids
                        if (dist < rule3Distance) {
                            perceived_velocity += vel1[idx];
                            neighbors3++;
                        }
                    }
                }
            }
        }
    }
    if (neighbors1 > 0)
        perceived_center /= neighbors1;
    glm::vec3 v1 = (perceived_center - currPos) * rule1Scale;

    glm::vec3 v2 = c * rule2Scale;

    if (neighbors3 > 0)
        perceived_velocity /= neighbors3;
    glm::vec3 v3 = perceived_velocity * rule3Scale;

    glm::vec3 v_new = vel1[index] + v1 + v2 + v3;

    // - Clamp the speed change before putting the new speed in vel2
    if (glm::length(v_new) > maxSpeed) {
        v_new = glm::normalize(v_new) * maxSpeed;
    }
    vel2[index] = v_new;
}

__global__ void kernUpdateVelNeighborSearchCoherent(
  int N, int gridResolution, glm::vec3 gridMin,
  float inverseCellWidth, float cellWidth,
  int *gridCellStartIndices, int *gridCellEndIndices,
  glm::vec3 *pos, glm::vec3 *vel1, glm::vec3 *vel2) {
  // TODO-2.3 - This should be very similar to kernUpdateVelNeighborSearchScattered,
  // except with one less level of indirection.
  // This should expect gridCellStartIndices and gridCellEndIndices to refer
  // directly to pos and vel1.
  // - Identify the grid cell that this particle is in
  // - Identify which cells may contain neighbors. This isn't always 8.
  // - For each cell, read the start/end indices in the boid pointer array.
  //   DIFFERENCE: For best results, consider what order the cells should be
  //   checked in to maximize the memory benefits of reordering the boids data.
  // - Access each boid in the cell and compute velocity change from
  //   the boids rules, if this boid is within the neighborhood distance.
  // - Clamp the speed change before putting the new speed in vel2
}

/**
* Step the entire N-body simulation by `dt` seconds.
*/
void Boids::stepSimulationNaive(float dt) {
  // TODO-1.2 - use the kernels you wrote to step the simulation forward in time.
  // TODO-1.2 ping-pong the velocity buffers
    
    dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);

    kernUpdateVelocityBruteForce << <fullBlocksPerGrid, blockSize >> > (numObjects, dev_pos, dev_vel1, dev_vel2);

    kernUpdatePos << <fullBlocksPerGrid, blockSize >> > (numObjects, dt, dev_pos, dev_vel2);

    std::swap(dev_vel1, dev_vel2);
} 

void Boids::stepSimulationScatteredGrid(float dt) {
  // TODO-2.1
  // Uniform Grid Neighbor search using Thrust sort.
  // In Parallel:
  // - label each particle with its array index as well as its grid index.

    dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);
    dim3 gridCellThread((gridCellCount + blockSize - 1) / blockSize);

    kernComputeIndices << <fullBlocksPerGrid, blockSize >> > (numObjects, gridSideCount, gridMinimum,
        gridInverseCellWidth, dev_pos, dev_particleArrayIndices, dev_particleGridIndices);
    
  //   Use 2x width grids.
  // - Unstable key sort using Thrust. A stable sort isn't necessary, but you
  //   are welcome to do a performance comparison.
    thrust::sort_by_key(dev_thrust_particleGridIndices, dev_thrust_particleGridIndices + numObjects, dev_thrust_particleArrayIndices);
  // - Naively unroll the loop for finding the start and end indices of each
  //   cell's data pointers in the array of boid indices
    kernResetIntBuffer << <gridCellThread, blockSize >> > (gridCellCount, dev_gridCellStartIndices, -1);
    kernResetIntBuffer << <gridCellThread, blockSize >> > (gridCellCount, dev_gridCellEndIndices, -1);

    kernIdentifyCellStartEnd << <fullBlocksPerGrid, blockSize >> > (numObjects, dev_particleGridIndices,
        dev_gridCellStartIndices, dev_gridCellEndIndices);

  // - Perform velocity updates using neighbor search
    kernUpdateVelNeighborSearchScattered << < fullBlocksPerGrid, blockSize >> > (numObjects, gridSideCount,
        gridMinimum, gridInverseCellWidth, gridCellWidth, dev_gridCellStartIndices, dev_gridCellEndIndices,
        dev_particleArrayIndices, dev_pos, dev_vel1, dev_vel2);
  // 
  // - Update positions
    kernUpdatePos << <fullBlocksPerGrid, blockSize >> > (numObjects, dt, dev_pos, dev_vel2);
  // - Ping-pong buffers as needed
    std::swap(dev_vel1, dev_vel2);
}

void Boids::stepSimulationCoherentGrid(float dt) {
  // TODO-2.3 - start by copying Boids::stepSimulationNaiveGrid
  // Uniform Grid Neighbor search using Thrust sort on cell-coherent data.
  // In Parallel:
  // - Label each particle with its array index as well as its grid index.
  //   Use 2x width grids
  // - Unstable key sort using Thrust. A stable sort isn't necessary, but you
  //   are welcome to do a performance comparison.
  // - Naively unroll the loop for finding the start and end indices of each
  //   cell's data pointers in the array of boid indices
  // - BIG DIFFERENCE: use the rearranged array index buffer to reshuffle all
  //   the particle data in the simulation array.
  //   CONSIDER WHAT ADDITIONAL BUFFERS YOU NEED
  // - Perform velocity updates using neighbor search
  // - Update positions
  // - Ping-pong buffers as needed. THIS MAY BE DIFFERENT FROM BEFORE.
}

void Boids::endSimulation() {
  hipFree(dev_vel1);
  hipFree(dev_vel2);
  hipFree(dev_pos);

  // TODO-2.1 TODO-2.3 - Free any additional buffers here.
  hipFree(dev_gridCellEndIndices);
  hipFree(dev_gridCellStartIndices);
  hipFree(dev_particleArrayIndices);
  hipFree(dev_particleGridIndices);
}

void Boids::unitTest() {
  // LOOK-1.2 Feel free to write additional tests here.

  // test unstable sort
  int *dev_intKeys;
  int *dev_intValues;
  int N = 10;

  std::unique_ptr<int[]>intKeys{ new int[N] };
  std::unique_ptr<int[]>intValues{ new int[N] };

  intKeys[0] = 0; intValues[0] = 0;
  intKeys[1] = 1; intValues[1] = 1;
  intKeys[2] = 0; intValues[2] = 2;
  intKeys[3] = 3; intValues[3] = 3;
  intKeys[4] = 0; intValues[4] = 4;
  intKeys[5] = 2; intValues[5] = 5;
  intKeys[6] = 2; intValues[6] = 6;
  intKeys[7] = 0; intValues[7] = 7;
  intKeys[8] = 5; intValues[8] = 8;
  intKeys[9] = 6; intValues[9] = 9;

  hipMalloc((void**)&dev_intKeys, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_intKeys failed!");

  hipMalloc((void**)&dev_intValues, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_intValues failed!");

  dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);

  std::cout << "before unstable sort: " << std::endl;
  for (int i = 0; i < N; i++) {
    std::cout << "  key: " << intKeys[i];
    std::cout << " value: " << intValues[i] << std::endl;
  }

  // How to copy data to the GPU
  hipMemcpy(dev_intKeys, intKeys.get(), sizeof(int) * N, hipMemcpyHostToDevice);
  hipMemcpy(dev_intValues, intValues.get(), sizeof(int) * N, hipMemcpyHostToDevice);

  // Wrap device vectors in thrust iterators for use with thrust.
  thrust::device_ptr<int> dev_thrust_keys(dev_intKeys);
  thrust::device_ptr<int> dev_thrust_values(dev_intValues);
  // LOOK-2.1 Example for using thrust::sort_by_key
  thrust::sort_by_key(dev_thrust_keys, dev_thrust_keys + N, dev_thrust_values);

  // How to copy data back to the CPU side from the GPU
  hipMemcpy(intKeys.get(), dev_intKeys, sizeof(int) * N, hipMemcpyDeviceToHost);
  hipMemcpy(intValues.get(), dev_intValues, sizeof(int) * N, hipMemcpyDeviceToHost);
  checkCUDAErrorWithLine("memcpy back failed!");

  std::cout << "after unstable sort: " << std::endl;
  for (int i = 0; i < N; i++) {
    std::cout << "  key: " << intKeys[i];
    std::cout << " value: " << intValues[i] << std::endl;
  }

  // cleanup
  hipFree(dev_intKeys);
  hipFree(dev_intValues);
  checkCUDAErrorWithLine("hipFree failed!");
  return;
}
