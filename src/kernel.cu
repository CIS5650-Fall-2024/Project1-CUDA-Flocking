#include "hip/hip_runtime.h"
#define GLM_FORCE_CUDA
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <glm/glm.hpp>
#include "utilityCore.hpp"
#include "kernel.h"

// LOOK-2.1 potentially useful for doing grid-based neighbor search
#ifndef imax
#define imax( a, b ) ( ((a) > (b)) ? (a) : (b) )
#endif

#ifndef imin
#define imin( a, b ) ( ((a) < (b)) ? (a) : (b) )
#endif

#define checkCUDAErrorWithLine(msg) checkCUDAError(msg, __LINE__)

/**
* Check for CUDA errors; print and exit if there was a problem.
*/
void checkCUDAError(const char *msg, int line = -1) {
  hipError_t err = hipGetLastError();
  if (hipSuccess != err) {
    if (line >= 0) {
      fprintf(stderr, "Line %d: ", line);
    }
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}

/**
 * Utility functions
*/
template<typename T>
void static inline swap(T* &a, T* &b) {
  auto *const temp = a;
  a = b;
  b = temp;
}

template<typename T>
__global__ void gather(const int* map, int N, const T* values, T* output) {
  const auto index = threadIdx.x + (blockIdx.x * blockDim.x);
  if (index >= N) return;

  output[index] = values[map[index]];
}

/*****************
* Configuration *
*****************/

/*! Block size used for CUDA kernel launch. */
#define blockSize 128

// LOOK-1.2 Parameters for the boids algorithm.
// These worked well in our reference implementation.
#define rule1Distance 5.0f
#define rule2Distance 3.0f
#define rule3Distance 9.0f

#define rule1Scale 0.01f
#define rule2Scale 0.1f
#define rule3Scale 0.1f

#define maxSpeed 1.0f

#define FULL_NEIGHBOR_CHECK 1

/*! Size of the starting area in simulation space. */
#define scene_scale 100.0f

/***********************************************
* Kernel state (pointers are device pointers) *
***********************************************/

int numObjects;
dim3 threadsPerBlock(blockSize);

// LOOK-1.2 - These buffers are here to hold all your boid information.
// These get allocated for you in Boids::initSimulation.
// Consider why you would need two velocity buffers in a simulation where each
// boid cares about its neighbors' velocities.
// These are called ping-pong buffers.
glm::vec3 *dev_pos;
glm::vec3 *dev_vel1;
glm::vec3 *dev_vel2;

// LOOK-2.1 - these are NOT allocated for you. You'll have to set up the thrust
// pointers on your own too.

// For efficient sorting and the uniform grid. These should always be parallel.
int *dev_particleArrayIndices; // What index in dev_pos and dev_velX represents this particle?
int *dev_particleGridIndices; // What grid cell is this particle in?
// needed for use with thrust
thrust::device_ptr<int> dev_thrust_particleArrayIndices;
thrust::device_ptr<int> dev_thrust_particleGridIndices;

int *dev_gridCellStartIndices; // What part of dev_particleArrayIndices belongs
int *dev_gridCellEndIndices;   // to this cell?

// TODO-2.3 - consider what additional buffers you might need to reshuffle
// the position and velocity data to be coherent within cells.
glm::vec3 *dev_pos_rearranged;
glm::vec3 *dev_vel1_rearranged;

// LOOK-2.1 - Grid parameters based on simulation parameters.
// These are automatically computed for you in Boids::initSimulation
int gridCellCount;
int gridSideCount;
float gridCellWidth;
float gridInverseCellWidth;
glm::vec3 gridMinimum;

/******************
* initSimulation *
******************/

__host__ __device__ unsigned int hash(unsigned int a) {
  a = (a + 0x7ed55d16) + (a << 12);
  a = (a ^ 0xc761c23c) ^ (a >> 19);
  a = (a + 0x165667b1) + (a << 5);
  a = (a + 0xd3a2646c) ^ (a << 9);
  a = (a + 0xfd7046c5) + (a << 3);
  a = (a ^ 0xb55a4f09) ^ (a >> 16);
  return a;
}

/**
* LOOK-1.2 - this is a typical helper function for a CUDA kernel.
* Function for generating a random vec3.
*/
__host__ __device__ glm::vec3 generateRandomVec3(float time, int index) {
  thrust::default_random_engine rng(hash((int)(index * time)));
  thrust::uniform_real_distribution<float> unitDistrib(-1, 1);

  return glm::vec3((float)unitDistrib(rng), (float)unitDistrib(rng), (float)unitDistrib(rng));
}

/**
* LOOK-1.2 - This is a basic CUDA kernel.
* CUDA kernel for generating boids with a specified mass randomly around the star.
*/
__global__ void kernGenerateRandomPosArray(int time, int N, glm::vec3 * arr, float scale) {
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (index < N) {
    glm::vec3 rand = generateRandomVec3(time, index);
    arr[index].x = scale * rand.x;
    arr[index].y = scale * rand.y;
    arr[index].z = scale * rand.z;
  }
}

/**
* Initialize memory, update some globals
*/
void Boids::initSimulation(int N) {
  numObjects = N;
  dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);

  // LOOK-1.2 - This is basic CUDA memory management and error checking.
  // Don't forget to hipFree in  Boids::endSimulation.
  hipMalloc((void**)&dev_pos, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_pos failed!");

  hipMalloc((void**)&dev_vel1, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_vel1 failed!");

  hipMalloc((void**)&dev_vel2, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_vel2 failed!");

  // LOOK-1.2 - This is a typical CUDA kernel invocation.
  kernGenerateRandomPosArray<<<fullBlocksPerGrid, blockSize>>>(1, numObjects,
    dev_pos, scene_scale);
  checkCUDAErrorWithLine("kernGenerateRandomPosArray failed!");

  // LOOK-2.1 computing grid params
  #if FULL_NEIGHBOR_CHECK
    gridCellWidth = std::max(std::max(rule1Distance, rule2Distance), rule3Distance);
  #else
    gridCellWidth = 2.0f * std::max(std::max(rule1Distance, rule2Distance), rule3Distance);
  #endif
  
  int halfSideCount = (int)(scene_scale / gridCellWidth) + 1;
  gridSideCount = 2 * halfSideCount;

  gridCellCount = gridSideCount * gridSideCount * gridSideCount;
  gridInverseCellWidth = 1.0f / gridCellWidth;
  float halfGridWidth = gridCellWidth * halfSideCount;
  gridMinimum.x -= halfGridWidth;
  gridMinimum.y -= halfGridWidth;
  gridMinimum.z -= halfGridWidth;

  // TODO-2.1 TODO-2.3 - Allocate additional buffers here.
  hipMalloc((void**)&dev_particleArrayIndices, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_particleArrayIndices failed!");

  hipMalloc((void**)&dev_particleGridIndices, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_particleGridIndices failed!");

  dev_thrust_particleArrayIndices = thrust::device_pointer_cast(dev_particleArrayIndices);
  dev_thrust_particleGridIndices = thrust::device_pointer_cast(dev_particleGridIndices);

  hipMalloc((void**)&dev_gridCellStartIndices, gridCellCount * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_gridCellStartIndices failed!");
  hipMalloc((void**)&dev_gridCellEndIndices, gridCellCount * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_gridCellEndIndices failed!");

  hipMalloc((void**)&dev_pos_rearranged, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_pos_rearranged failed!");

  hipMalloc((void**)&dev_vel1_rearranged, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_vel1_rearranged failed!");

  hipDeviceSynchronize();
}


/******************
* copyBoidsToVBO *
******************/

/**
* Copy the boid positions into the VBO so that they can be drawn by OpenGL.
*/
__global__ void kernCopyPositionsToVBO(int N, glm::vec3 *pos, float *vbo, float s_scale) {
  int index = threadIdx.x + (blockIdx.x * blockDim.x);

  float c_scale = -1.0f / s_scale;

  if (index < N) {
    vbo[4 * index + 0] = pos[index].x * c_scale;
    vbo[4 * index + 1] = pos[index].y * c_scale;
    vbo[4 * index + 2] = pos[index].z * c_scale;
    vbo[4 * index + 3] = 1.0f;
  }
}

__global__ void kernCopyVelocitiesToVBO(int N, glm::vec3 *vel, float *vbo, float s_scale) {
  int index = threadIdx.x + (blockIdx.x * blockDim.x);

  if (index < N) {
    vbo[4 * index + 0] = vel[index].x + 0.3f;
    vbo[4 * index + 1] = vel[index].y + 0.3f;
    vbo[4 * index + 2] = vel[index].z + 0.3f;
    vbo[4 * index + 3] = 1.0f;
  }
}

/**
* Wrapper for call to the kernCopyboidsToVBO CUDA kernel.
*/
void Boids::copyBoidsToVBO(float *vbodptr_positions, float *vbodptr_velocities) {
  dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);

  kernCopyPositionsToVBO << <fullBlocksPerGrid, blockSize >> >(numObjects, dev_pos, vbodptr_positions, scene_scale);
  kernCopyVelocitiesToVBO << <fullBlocksPerGrid, blockSize >> >(numObjects, dev_vel1, vbodptr_velocities, scene_scale);

  checkCUDAErrorWithLine("copyBoidsToVBO failed!");

  hipDeviceSynchronize();
}

__device__ static inline glm::vec3 compute_new_vel(
  const glm::vec3 &pos, const glm::vec3 &vel,
  const glm::vec3 &perceived_center, int rule_1_neighbors,
  const glm::vec3 &repulsion,
  const glm::vec3 &perceived_velocity, int rule_3_neighbors
) {

  auto new_velocity = vel + repulsion * rule2Scale;
  if (rule_1_neighbors > 0) new_velocity += (perceived_center / (float)rule_1_neighbors - pos) * rule1Scale;
  if (rule_3_neighbors > 0) new_velocity += perceived_velocity / (float)rule_3_neighbors * rule3Scale;

  return glm::clamp(new_velocity, -maxSpeed, maxSpeed);
}


/******************
* stepSimulation *
******************/

/**
* LOOK-1.2 You can use this as a helper for kernUpdateVelocityBruteForce.
* __device__ code can be called from a __global__ context
* Compute the new velocity on the body with index `iSelf` due to the `N` boids
* in the `pos` and `vel` arrays.
*/
__device__ glm::vec3 computeVelocityChange(int N, int iSelf, const glm::vec3 *pos, const glm::vec3 *vel) {
  // Rule 1: boids fly towards their local perceived center of mass, which excludes themselves
  // Rule 2: boids try to stay a distance d away from each other
  // Rule 3: boids try to match the speed of surrounding boids

  int rule_1_neighbors = 0;
  glm::vec3 perceived_center; // rule 1

  glm::vec3 repulsion; // rule 2

  int rule_3_neighbors = 0;
  glm::vec3 perceived_velocity; // rule 3

  for (auto i = 0; i < N; i++) {
    if (i == iSelf) continue;

    const auto relative_position = pos[i] - pos[iSelf];
    const auto squared_distance = glm::dot(relative_position, relative_position);

    // Rule 1
    if (squared_distance < rule1Distance * rule1Distance) {
      perceived_center += pos[i];
      rule_1_neighbors++;
    }

    // Rule 2
    if (squared_distance < rule2Distance * rule2Distance) {
      repulsion -= relative_position;
    }
  
    // Rule 3
    if (squared_distance < rule3Distance * rule3Distance) {
      perceived_velocity += vel[i];
      rule_3_neighbors++;
    }
  }

  return compute_new_vel(
    pos[iSelf], vel[iSelf],
    perceived_center, rule_1_neighbors,
    repulsion,
    perceived_velocity, rule_3_neighbors
  );

}

/**
* TODO-1.2 implement basic flocking
* For each of the `N` bodies, update its position based on its current velocity.
*/
__global__ void kernUpdateVelocityBruteForce(int N, glm::vec3 *pos,
  glm::vec3 *vel1, glm::vec3 *vel2) {
  const auto index = threadIdx.x + (blockIdx.x * blockDim.x);
  if (index >= N) return;

  vel2[index] = computeVelocityChange(N, index, pos, vel1);
}

/**
* LOOK-1.2 Since this is pretty trivial, we implemented it for you.
* For each of the `N` bodies, update its position based on its current velocity.
*/
__global__ void kernUpdatePos(int N, float dt, glm::vec3 *pos, glm::vec3 *vel) {
  // Update position by velocity
  int index = threadIdx.x + (blockIdx.x * blockDim.x);
  if (index >= N) {
    return;
  }
  glm::vec3 thisPos = pos[index];
  thisPos += vel[index] * dt;

  // Wrap the boids around so we don't lose them
  thisPos.x = thisPos.x < -scene_scale ? scene_scale : thisPos.x;
  thisPos.y = thisPos.y < -scene_scale ? scene_scale : thisPos.y;
  thisPos.z = thisPos.z < -scene_scale ? scene_scale : thisPos.z;

  thisPos.x = thisPos.x > scene_scale ? -scene_scale : thisPos.x;
  thisPos.y = thisPos.y > scene_scale ? -scene_scale : thisPos.y;
  thisPos.z = thisPos.z > scene_scale ? -scene_scale : thisPos.z;

  pos[index] = thisPos;
}

// LOOK-2.1 Consider this method of computing a 1D index from a 3D grid index.
// LOOK-2.3 Looking at this method, what would be the most memory efficient
//          order for iterating over neighboring grid cells?
//          for(x)
//            for(y)
//             for(z)? Or some other order?
__device__ int gridIndex3Dto1D(int x, int y, int z, int gridResolution) {
  return x + y * gridResolution + z * gridResolution * gridResolution;
}

__global__ void kernComputeIndices(int N, int gridResolution,
  glm::vec3 gridMin, float inverseCellWidth,
  glm::vec3 *pos, int *indices, int *gridIndices) {
    // TODO-2.1
    // - Label each boid with the index of its grid cell.
    // - Set up a parallel array of integer indices as pointers to the actual
    //   boid data in pos and vel1/vel2
  const auto index = threadIdx.x + (blockIdx.x * blockDim.x);
  if (index >= N) return;

  const glm::ivec3 grid_loc_3d{(pos[index] - gridMin) * inverseCellWidth};
  const auto grid_loc_1d = gridIndex3Dto1D(grid_loc_3d.x, grid_loc_3d.y, grid_loc_3d.z, gridResolution);

  gridIndices[index] = grid_loc_1d;
  indices[index] = index;
}

// LOOK-2.1 Consider how this could be useful for indicating that a cell
//          does not enclose any boids
__global__ void kernResetIntBuffer(int N, int *intBuffer, int value) {
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (index < N) {
    intBuffer[index] = value;
  }
}

__global__ void kernIdentifyCellStartEnd(int N, int *particleGridIndices,
  int *gridCellStartIndices, int *gridCellEndIndices) {
  // TODO-2.1
  // Identify the start point of each cell in the gridIndices array.
  // This is basically a parallel unrolling of a loop that goes
  // "this index doesn't match the one before it, must be a new cell!"

  const auto index = threadIdx.x + (blockIdx.x * blockDim.x);
  if (index >= N) return;

  const auto grid_index = particleGridIndices[index];
  if (index == 0) {
    gridCellStartIndices[grid_index] = index;
    return;
  }

  const auto prev_grid_index = particleGridIndices[index - 1];
  if (grid_index != prev_grid_index) {
    gridCellEndIndices[prev_grid_index] = index;
    gridCellStartIndices[grid_index] = index;
  }

  if (index == N - 1) gridCellEndIndices[grid_index] = index + 1;

}

__global__ void kernUpdateVelNeighborSearchScattered(
  int N, int gridResolution, glm::vec3 gridMin,
  float inverseCellWidth, float cellWidth,
  int *gridCellStartIndices, int *gridCellEndIndices,
  int *particleArrayIndices,
  glm::vec3 *pos, glm::vec3 *vel1, glm::vec3 *vel2) {
  // TODO-2.1 - Update a boid's velocity using the uniform grid to reduce
  // the number of boids that need to be checked.
  // - Identify the grid cell that this particle is in
  // - Identify which cells may contain neighbors. This isn't always 8.
  // - For each cell, read the start/end indices in the boid pointer array.
  // - Access each boid in the cell and compute velocity change from
  //   the boids rules, if this boid is within the neighborhood distance.
  // - Clamp the speed change before putting the new speed in vel2

  const auto index = threadIdx.x + (blockIdx.x * blockDim.x);
  if (index >= N) return;


#if FULL_NEIGHBOR_CHECK
  const glm::ivec3 grid_loc_3d{(pos[index] - gridMin) * inverseCellWidth};
#else
  const glm::ivec3 grid_loc_3d{glm::round((pos[index] - gridMin) * inverseCellWidth)};
#endif

  int rule_1_neighbors = 0;
  glm::vec3 perceived_center; // rule 1

  glm::vec3 repulsion; // rule 2

  int rule_3_neighbors = 0;
  glm::vec3 perceived_velocity; // rule 3

#if FULL_NEIGHBOR_CHECK
  for (auto x = -1; x <= 1; x++) {
    for (auto y = -1; y <= 1; y++) {
        for (auto z = -1; z <= 1; z++) {
#else
  for (auto x = -1; x <= 0; x++) {
    for (auto y = -1; y <= 0; y++) {
        for (auto z = -1; z <= 0; z++) {
#endif
            const auto neighbor_grid_loc_3d = grid_loc_3d + glm::ivec3{x, y, z};
            if (
                glm::any(glm::lessThan(neighbor_grid_loc_3d, glm::ivec3{0}))
                || glm::any(glm::greaterThanEqual(neighbor_grid_loc_3d, glm::ivec3{gridResolution}))
            ) continue;

            const auto neighbor_grid_loc_1d = gridIndex3Dto1D(
                neighbor_grid_loc_3d.x,
                neighbor_grid_loc_3d.y,
                neighbor_grid_loc_3d.z,
                gridResolution
            );


            // implicitly handles cells with no boids because these will both be -1.
            for (auto b = gridCellStartIndices[neighbor_grid_loc_1d]; b < gridCellEndIndices[neighbor_grid_loc_1d]; b++) {
                const auto boid_index = particleArrayIndices[b];
                if (index == boid_index) continue;

                const auto relative_position = pos[boid_index] - pos[index];
                const auto squared_distance = glm::dot(relative_position, relative_position);

                // Rule 1
                if (squared_distance < rule1Distance * rule1Distance) {
                    perceived_center += pos[boid_index];
                    rule_1_neighbors++;
                }

                // Rule 2
                if (squared_distance < rule2Distance * rule2Distance) {
                    repulsion -= relative_position;
                }
            
                // Rule 3
                if (squared_distance < rule3Distance * rule3Distance) {
                    perceived_velocity += vel1[boid_index];
                    rule_3_neighbors++;
                }
            }
        }
    }
  }

  vel2[index] = compute_new_vel(
    pos[index], vel1[index],
    perceived_center, rule_1_neighbors,
    repulsion,
    perceived_velocity, rule_3_neighbors
  );
}

__global__ void kernUpdateVelNeighborSearchCoherent(
  int N, int gridResolution, glm::vec3 gridMin,
  float inverseCellWidth, float cellWidth,
  int *gridCellStartIndices, int *gridCellEndIndices,
  glm::vec3 *pos, glm::vec3 *vel1, glm::vec3 *vel2) {
  // TODO-2.3 - This should be very similar to kernUpdateVelNeighborSearchScattered,
  // except with one less level of indirection.
  // This should expect gridCellStartIndices and gridCellEndIndices to refer
  // directly to pos and vel1.
  // - Identify the grid cell that this particle is in
  // - Identify which cells may contain neighbors. This isn't always 8.
  // - For each cell, read the start/end indices in the boid pointer array.
  //   DIFFERENCE: For best results, consider what order the cells should be
  //   checked in to maximize the memory benefits of reordering the boids data.
  // - Access each boid in the cell and compute velocity change from
  //   the boids rules, if this boid is within the neighborhood distance.
  // - Clamp the speed change before putting the new speed in vel2

  const auto index = threadIdx.x + (blockIdx.x * blockDim.x);
  if (index >= N) return;


#if FULL_NEIGHBOR_CHECK
  const glm::ivec3 grid_loc_3d{(pos[index] - gridMin) * inverseCellWidth};
#else
  const glm::ivec3 grid_loc_3d{glm::round((pos[index] - gridMin) * inverseCellWidth)};
#endif

  int rule_1_neighbors = 0;
  glm::vec3 perceived_center; // rule 1

  glm::vec3 repulsion; // rule 2

  int rule_3_neighbors = 0;
  glm::vec3 perceived_velocity; // rule 3

#if FULL_NEIGHBOR_CHECK
  for (auto x = -1; x <= 1; x++) {
    for (auto y = -1; y <= 1; y++) {
        for (auto z = -1; z <= 1; z++) {
#else
  for (auto x = -1; x <= 0; x++) {
    for (auto y = -1; y <= 0; y++) {
        for (auto z = -1; z <= 0; z++) {
#endif
            const auto neighbor_grid_loc_3d = grid_loc_3d + glm::ivec3{x, y, z};
            if (
                glm::any(glm::lessThan(neighbor_grid_loc_3d, glm::ivec3{0}))
                || glm::any(glm::greaterThanEqual(neighbor_grid_loc_3d, glm::ivec3{gridResolution}))
            ) continue;

            const auto neighbor_grid_loc_1d = gridIndex3Dto1D(
                neighbor_grid_loc_3d.x,
                neighbor_grid_loc_3d.y,
                neighbor_grid_loc_3d.z,
                gridResolution
            );


            // implicitly handles cells with no boids because these will both be -1.
            for (auto b = gridCellStartIndices[neighbor_grid_loc_1d]; b < gridCellEndIndices[neighbor_grid_loc_1d]; b++) {
                if (index == b) continue;

                const auto relative_position = pos[b] - pos[index];
                const auto squared_distance = glm::dot(relative_position, relative_position);

                // Rule 1
                if (squared_distance < rule1Distance * rule1Distance) {
                    perceived_center += pos[b];
                    rule_1_neighbors++;
                }

                // Rule 2
                if (squared_distance < rule2Distance * rule2Distance) {
                    repulsion -= relative_position;
                }
            
                // Rule 3
                if (squared_distance < rule3Distance * rule3Distance) {
                    perceived_velocity += vel1[b];
                    rule_3_neighbors++;
                }
            }
        }
    }
  }

  vel2[index] = compute_new_vel(
    pos[index], vel1[index],
    perceived_center, rule_1_neighbors,
    repulsion,
    perceived_velocity, rule_3_neighbors
  );
}

/**
* Step the entire N-body simulation by `dt` seconds.
*/
void Boids::stepSimulationNaive(float dt) {
  dim3 full_blocks_per_grid{(numObjects + blockSize - 1) / blockSize};
  kernUpdateVelocityBruteForce<<<full_blocks_per_grid, blockSize>>>(numObjects, dev_pos, dev_vel1, dev_vel2);
  kernUpdatePos<<<full_blocks_per_grid, blockSize>>>(numObjects, dt, dev_pos, dev_vel1);
  swap(dev_vel1, dev_vel2);
}

void Boids::stepSimulationScatteredGrid(float dt) {
  // TODO-2.1
  // Uniform Grid Neighbor search using Thrust sort.
  // In Parallel:
  // - label each particle with its array index as well as its grid index.
  //   Use 2x width grids.
  // - Unstable key sort using Thrust. A stable sort isn't necessary, but you
  //   are welcome to do a performance comparison.
  // - Naively unroll the loop for finding the start and end indices of each
  //   cell's data pointers in the array of boid indices
  // - Perform velocity updates using neighbor search
  // - Update positions
  // - Ping-pong buffers as needed

  dim3 full_blocks_per_grid{(numObjects + blockSize - 1) / blockSize};

  kernComputeIndices<<<full_blocks_per_grid, blockSize>>>(
    numObjects,
    gridSideCount,
    gridMinimum,
    gridInverseCellWidth,
    dev_pos,
    dev_particleArrayIndices,
    dev_particleGridIndices
  );

  thrust::sort_by_key(dev_thrust_particleGridIndices, dev_thrust_particleGridIndices + numObjects, dev_particleArrayIndices);
  
  dim3 grid_blocks{(gridCellCount + blockSize - 1) / blockSize};
  kernResetIntBuffer<<<grid_blocks, blockSize>>>(gridCellCount, dev_gridCellStartIndices, -1);
  kernResetIntBuffer<<<grid_blocks, blockSize>>>(gridCellCount, dev_gridCellEndIndices, -1);

  kernIdentifyCellStartEnd<<<full_blocks_per_grid, blockSize>>>(
    numObjects,
    dev_particleGridIndices,
    dev_gridCellStartIndices,
    dev_gridCellEndIndices
  );

  kernUpdateVelNeighborSearchScattered<<<full_blocks_per_grid, blockSize>>>(
    numObjects,
    gridSideCount, gridMinimum, gridInverseCellWidth, gridCellWidth,
    dev_gridCellStartIndices,
    dev_gridCellEndIndices,
    dev_particleArrayIndices,
    dev_pos, dev_vel1, dev_vel2
  );

  kernUpdatePos<<<full_blocks_per_grid, blockSize>>>(numObjects, dt, dev_pos, dev_vel2);

  swap(dev_vel1, dev_vel2);
}

void Boids::stepSimulationCoherentGrid(float dt) {
  // TODO-2.3 - start by copying Boids::stepSimulationNaiveGrid
  // Uniform Grid Neighbor search using Thrust sort on cell-coherent data.
  // In Parallel:
  // - Label each particle with its array index as well as its grid index.
  //   Use 2x width grids
  // - Unstable key sort using Thrust. A stable sort isn't necessary, but you
  //   are welcome to do a performance comparison.
  // - Naively unroll the loop for finding the start and end indices of each
  //   cell's data pointers in the array of boid indices
  // - BIG DIFFERENCE: use the rearranged array index buffer to reshuffle all
  //   the particle data in the simulation array.
  //   CONSIDER WHAT ADDITIONAL BUFFERS YOU NEED
  // - Perform velocity updates using neighbor search
  // - Update positions
  // - Ping-pong buffers as needed. THIS MAY BE DIFFERENT FROM BEFORE.

  dim3 full_blocks_per_grid{(numObjects + blockSize - 1) / blockSize};

  kernComputeIndices<<<full_blocks_per_grid, blockSize>>>(
    numObjects,
    gridSideCount,
    gridMinimum,
    gridInverseCellWidth,
    dev_pos,
    dev_particleArrayIndices,
    dev_particleGridIndices
  );

  thrust::sort_by_key(dev_thrust_particleGridIndices, dev_thrust_particleGridIndices + numObjects, dev_particleArrayIndices);
  
  dim3 grid_blocks{(gridCellCount + blockSize - 1) / blockSize};
  kernResetIntBuffer<<<grid_blocks, blockSize>>>(gridCellCount, dev_gridCellStartIndices, -1);
  kernResetIntBuffer<<<grid_blocks, blockSize>>>(gridCellCount, dev_gridCellEndIndices, -1);

  kernIdentifyCellStartEnd<<<full_blocks_per_grid, blockSize>>>(
    numObjects,
    dev_particleGridIndices,
    dev_gridCellStartIndices,
    dev_gridCellEndIndices
  );

  gather<<<full_blocks_per_grid, blockSize>>>(dev_particleArrayIndices, numObjects, dev_pos, dev_pos_rearranged);
  gather<<<full_blocks_per_grid, blockSize>>>(dev_particleArrayIndices, numObjects, dev_vel1, dev_vel1_rearranged);

  kernUpdateVelNeighborSearchCoherent<<<full_blocks_per_grid, blockSize>>>(
    numObjects,
    gridSideCount, gridMinimum, gridInverseCellWidth, gridCellWidth,
    dev_gridCellStartIndices,
    dev_gridCellEndIndices,
    dev_pos_rearranged, dev_vel1_rearranged, dev_vel2
  );

  kernUpdatePos<<<full_blocks_per_grid, blockSize>>>(numObjects, dt, dev_pos_rearranged, dev_vel2);

  swap(dev_pos, dev_pos_rearranged);
  swap(dev_vel1, dev_vel2);
}

void Boids::endSimulation() {
  hipFree(dev_vel1);
  hipFree(dev_vel2);
  hipFree(dev_pos);

  // TODO-2.1 TODO-2.3 - Free any additional buffers here.
  hipFree(dev_particleArrayIndices);
  hipFree(dev_particleGridIndices);
  hipFree(dev_gridCellStartIndices);
  hipFree(dev_gridCellEndIndices);

  hipFree(dev_pos_rearranged);
  hipFree(dev_vel1_rearranged);
}

void Boids::unitTest() {
  // LOOK-1.2 Feel free to write additional tests here.

  // test unstable sort
  int *dev_intKeys;
  int *dev_intValues;
  int N = 10;

  std::unique_ptr<int[]>intKeys{ new int[N] };
  std::unique_ptr<int[]>intValues{ new int[N] };

  intKeys[0] = 0; intValues[0] = 0;
  intKeys[1] = 1; intValues[1] = 1;
  intKeys[2] = 0; intValues[2] = 2;
  intKeys[3] = 3; intValues[3] = 3;
  intKeys[4] = 0; intValues[4] = 4;
  intKeys[5] = 2; intValues[5] = 5;
  intKeys[6] = 2; intValues[6] = 6;
  intKeys[7] = 0; intValues[7] = 7;
  intKeys[8] = 5; intValues[8] = 8;
  intKeys[9] = 6; intValues[9] = 9;

  hipMalloc((void**)&dev_intKeys, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_intKeys failed!");

  hipMalloc((void**)&dev_intValues, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_intValues failed!");

  dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);

  std::cout << "before unstable sort: " << std::endl;
  for (int i = 0; i < N; i++) {
    std::cout << "  key: " << intKeys[i];
    std::cout << " value: " << intValues[i] << std::endl;
  }

  // How to copy data to the GPU
  hipMemcpy(dev_intKeys, intKeys.get(), sizeof(int) * N, hipMemcpyHostToDevice);
  hipMemcpy(dev_intValues, intValues.get(), sizeof(int) * N, hipMemcpyHostToDevice);

  // Wrap device vectors in thrust iterators for use with thrust.
  thrust::device_ptr<int> dev_thrust_keys(dev_intKeys);
  thrust::device_ptr<int> dev_thrust_values(dev_intValues);
  // LOOK-2.1 Example for using thrust::sort_by_key
  thrust::sort_by_key(dev_thrust_keys, dev_thrust_keys + N, dev_thrust_values);

  // How to copy data back to the CPU side from the GPU
  hipMemcpy(intKeys.get(), dev_intKeys, sizeof(int) * N, hipMemcpyDeviceToHost);
  hipMemcpy(intValues.get(), dev_intValues, sizeof(int) * N, hipMemcpyDeviceToHost);
  checkCUDAErrorWithLine("memcpy back failed!");

  std::cout << "after unstable sort: " << std::endl;
  for (int i = 0; i < N; i++) {
    std::cout << "  key: " << intKeys[i];
    std::cout << " value: " << intValues[i] << std::endl;
  }

  // cleanup
  hipFree(dev_intKeys);
  hipFree(dev_intValues);
  checkCUDAErrorWithLine("hipFree failed!");
  return;
}
