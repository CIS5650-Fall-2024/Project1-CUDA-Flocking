#include "hip/hip_runtime.h"
#define GLM_FORCE_CUDA
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <glm/glm.hpp>
#include "utilityCore.hpp"
#include "kernel.h"

// LOOK-2.1 potentially useful for doing grid-based neighbor search
#ifndef imax
#define imax( a, b ) ( ((a) > (b)) ? (a) : (b) )
#endif

#ifndef imin
#define imin( a, b ) ( ((a) < (b)) ? (a) : (b) )
#endif

#define checkCUDAErrorWithLine(msg) checkCUDAError(msg, __LINE__)

/**
* Check for CUDA errors; print and exit if there was a problem.
*/
void checkCUDAError(const char *msg, int line = -1) {
  hipError_t err = hipGetLastError();
  if (hipSuccess != err) {
    if (line >= 0) {
      fprintf(stderr, "Line %d: ", line);
    }
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}


/*****************
* Configuration *
*****************/

/*! Block size used for CUDA kernel launch. */
#define blockSize 128

// LOOK-1.2 Parameters for the boids algorithm.
// These worked well in our reference implementation.
#define rule1Distance 5.0f
#define rule2Distance 3.0f
#define rule3Distance 5.0f

#define rule1Scale 0.01f
#define rule2Scale 0.1f
#define rule3Scale 0.1f

#define maxSpeed 1.0f

/*! Size of the starting area in simulation space. */
#define scene_scale 100.0f

/***********************************************
* Kernel state (pointers are device pointers) *
***********************************************/

int numObjects;
dim3 threadsPerBlock(blockSize);

// LOOK-1.2 - These buffers are here to hold all your boid information.
// These get allocated for you in Boids::initSimulation.
// Consider why you would need two velocity buffers in a simulation where each
// boid cares about its neighbors' velocities.
// These are called ping-pong buffers.
glm::vec3 *dev_pos;
glm::vec3 *dev_vel1;
glm::vec3 *dev_vel2;

// LOOK-2.1 - these are NOT allocated for you. You'll have to set up the thrust
// pointers on your own too.

// For efficient sorting and the uniform grid. These should always be parallel.
int *dev_particleArrayIndices; // What index in dev_pos and dev_velX represents this particle?
int *dev_particleGridIndices; // What grid cell is this particle in?
// needed for use with thrust
thrust::device_ptr<int> dev_thrust_particleArrayIndices;
thrust::device_ptr<int> dev_thrust_particleGridIndices;

int *dev_gridCellStartIndices; // What part of dev_particleArrayIndices belongs
int *dev_gridCellEndIndices;   // to this cell?

// TODO-2.3 - consider what additional buffers you might need to reshuffle
// the position and velocity data to be coherent within cells.

// LOOK-2.1 - Grid parameters based on simulation parameters.
// These are automatically computed for you in Boids::initSimulation
int gridCellCount;
int gridSideCount;
float gridCellWidth;
float gridInverseCellWidth;
glm::vec3 gridMinimum;

/******************
* initSimulation *
******************/

__host__ __device__ unsigned int hash(unsigned int a) {
  a = (a + 0x7ed55d16) + (a << 12);
  a = (a ^ 0xc761c23c) ^ (a >> 19);
  a = (a + 0x165667b1) + (a << 5);
  a = (a + 0xd3a2646c) ^ (a << 9);
  a = (a + 0xfd7046c5) + (a << 3);
  a = (a ^ 0xb55a4f09) ^ (a >> 16);
  return a;
}

/**
* LOOK-1.2 - this is a typical helper function for a CUDA kernel.
* Function for generating a random vec3.
*/
__host__ __device__ glm::vec3 generateRandomVec3(float time, int index) {
  thrust::default_random_engine rng(hash((int)(index * time)));
  thrust::uniform_real_distribution<float> unitDistrib(-1, 1);

  return glm::vec3((float)unitDistrib(rng), (float)unitDistrib(rng), (float)unitDistrib(rng));
}

/**
* LOOK-1.2 - This is a basic CUDA kernel.
* CUDA kernel for generating boids with a specified mass randomly around the star.
*/
__global__ void kernGenerateRandomPosArray(int time, int N, glm::vec3 * arr, float scale) {
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (index < N) {
    glm::vec3 rand = generateRandomVec3(time, index);
    arr[index].x = scale * rand.x;
    arr[index].y = scale * rand.y;
    arr[index].z = scale * rand.z;
  }
}

/**
* Initialize memory, update some globals
*/
void Boids::initSimulation(int N) {
  numObjects = N;
  std::cout << N << std::endl;
  dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);
  std::cout << "Block Size " << fullBlocksPerGrid.x << " " << fullBlocksPerGrid.y << std::endl;
  // LOOK-1.2 - This is basic CUDA memory management and error checking.
  // Don't forget to hipFree in  Boids::endSimulation.
  hipMalloc((void**)&dev_pos, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_pos failed!");

  hipMalloc((void**)&dev_vel1, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_vel1 failed!");

  hipMalloc((void**)&dev_vel2, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_vel2 failed!");

  // LOOK-1.2 - This is a typical CUDA kernel invocation.
  kernGenerateRandomPosArray<<<fullBlocksPerGrid, blockSize>>>(1, numObjects,
    dev_pos, scene_scale);
  checkCUDAErrorWithLine("kernGenerateRandomPosArray failed!");

  // LOOK-2.1 computing grid params
  gridCellWidth = 2.0f * std::max(std::max(rule1Distance, rule2Distance), rule3Distance);
  int halfSideCount = (int)(scene_scale / gridCellWidth) + 1;
  gridSideCount = 2 * halfSideCount;

  gridCellCount = gridSideCount * gridSideCount * gridSideCount;
  gridInverseCellWidth = 1.0f / gridCellWidth;
  float halfGridWidth = gridCellWidth * halfSideCount;
  gridMinimum.x -= halfGridWidth;
  gridMinimum.y -= halfGridWidth;
  gridMinimum.z -= halfGridWidth;

  // TODO-2.1 TODO-2.3 - Allocate additional buffers here.
  hipMalloc((void**)&dev_particleArrayIndices, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_particleArrayIndices failed!");

  hipMalloc((void**)&dev_particleGridIndices, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_particleGridIndices failed!");

  hipMalloc((void**)&dev_gridCellStartIndices, gridCellCount * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_gridCellStartIndices failed!");

  hipMalloc((void**)&dev_gridCellEndIndices, gridCellCount * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_gridCellEndIndices failed!");
  hipDeviceSynchronize();
}


/******************
* copyBoidsToVBO *
******************/

/**
* Copy the boid positions into the VBO so that they can be drawn by OpenGL.
*/
__global__ void kernCopyPositionsToVBO(int N, glm::vec3 *pos, float *vbo, float s_scale) {
  int index = threadIdx.x + (blockIdx.x * blockDim.x);

  float c_scale = -1.0f / s_scale;

  if (index < N) {
    vbo[4 * index + 0] = pos[index].x * c_scale;
    vbo[4 * index + 1] = pos[index].y * c_scale;
    vbo[4 * index + 2] = pos[index].z * c_scale;
    vbo[4 * index + 3] = 1.0f;
  }
}

__global__ void kernCopyVelocitiesToVBO(int N, glm::vec3 *vel, float *vbo, float s_scale) {
  int index = threadIdx.x + (blockIdx.x * blockDim.x);

  if (index < N) {
    vbo[4 * index + 0] = vel[index].x + 0.3f;
    vbo[4 * index + 1] = vel[index].y + 0.3f;
    vbo[4 * index + 2] = vel[index].z + 0.3f;
    vbo[4 * index + 3] = 1.0f;
  }
}

/**
* Wrapper for call to the kernCopyboidsToVBO CUDA kernel.
*/
void Boids::copyBoidsToVBO(float *vbodptr_positions, float *vbodptr_velocities) {
  dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);

  kernCopyPositionsToVBO << <fullBlocksPerGrid, blockSize >> >(numObjects, dev_pos, vbodptr_positions, scene_scale);
  kernCopyVelocitiesToVBO << <fullBlocksPerGrid, blockSize >> >(numObjects, dev_vel1, vbodptr_velocities, scene_scale);

  checkCUDAErrorWithLine("copyBoidsToVBO failed!");

  hipDeviceSynchronize();
}


/******************
* stepSimulation *
******************/

/**
* LOOK-1.2 You can use this as a helper for kernUpdateVelocityBruteForce.
* __device__ code can be called from a __global__ context
* Compute the new velocity on the body with index `iSelf` due to the `N` boids
* in the `pos` and `vel` arrays.
*/
__device__ glm::vec3 computeVelocityChange(int N, int iSelf, const glm::vec3 *pos, const glm::vec3 *vel) {
    glm::vec3 res(0.0f);
  // Rule 1: boids fly towards their local perceived center of mass, which excludes themselves
    
    
    glm::vec3 perceived_center(0);
    glm::vec3 c(0);
    glm::vec3 perceived_velocity(0);
    int numOfNeighbors1 = 0;
    int numOfNeighbors2 = 0;
    for (int i = 0; i < N; i++) {
        float distance = glm::distance(pos[i], pos[iSelf]);
        if (i != iSelf && distance < rule1Distance) {
            perceived_center += pos[i];
            numOfNeighbors1++;
        }
        if (i != iSelf && distance < rule2Distance) {
            c -= (pos[i] - pos[iSelf]);
        }
        if (i != iSelf && distance < rule3Distance) {
            perceived_velocity += vel[i];
            numOfNeighbors2++;
        }
    }
    if (numOfNeighbors1 > 0) { 
        perceived_center /= numOfNeighbors1;
        res += (perceived_center - pos[iSelf]) * rule1Scale;
    }
    if (numOfNeighbors2 > 0) { 
        perceived_velocity /= numOfNeighbors2; 
        res += perceived_velocity * rule3Scale;
    }
    res += c * rule2Scale;
    return res;
}

/**
* TODO-1.2 implement basic flocking
* For each of the `N` bodies, update its position based on its current velocity.
*/
__global__ void kernUpdateVelocityBruteForce(int N, glm::vec3 *pos,
  glm::vec3 *vel1, glm::vec3 *vel2) {
  // Compute a new velocity based on pos and vel1
  // Clamp the speed
  // Record the new velocity into vel2. Question: why NOT vel1?
    int index = threadIdx.x + (blockIdx.x * blockDim.x);
    if (index < N) {
        glm::vec3 velocity = computeVelocityChange(N, index, pos, vel1) + vel1[index];
        float len = glm::length(velocity);
        if (len > maxSpeed) {
            vel2[index] = velocity / len * maxSpeed;
        }else{
            vel2[index] = velocity;
        }
    }
}

/**
* LOOK-1.2 Since this is pretty trivial, we implemented it for you.
* For each of the `N` bodies, update its position based on its current velocity.
*/
__global__ void kernUpdatePos(int N, float dt, glm::vec3 *pos, glm::vec3 *vel) {
  // Update position by velocity
  int index = threadIdx.x + (blockIdx.x * blockDim.x);
  if (index >= N) {
    return;
  }
  glm::vec3 thisPos = pos[index];
  thisPos += vel[index] * dt;

  // Wrap the boids around so we don't lose them
  thisPos.x = thisPos.x < -scene_scale ? scene_scale : thisPos.x;
  thisPos.y = thisPos.y < -scene_scale ? scene_scale : thisPos.y;
  thisPos.z = thisPos.z < -scene_scale ? scene_scale : thisPos.z;

  thisPos.x = thisPos.x > scene_scale ? -scene_scale : thisPos.x;
  thisPos.y = thisPos.y > scene_scale ? -scene_scale : thisPos.y;
  thisPos.z = thisPos.z > scene_scale ? -scene_scale : thisPos.z;

  pos[index] = thisPos;
}

// LOOK-2.1 Consider this method of computing a 1D index from a 3D grid index.
// LOOK-2.3 Looking at this method, what would be the most memory efficient
//          order for iterating over neighboring grid cells?
//          for(x)
//            for(y)
//             for(z)? Or some other order?
__device__ int gridIndex3Dto1D(int x, int y, int z, int gridResolution) {
  return x + y * gridResolution + z * gridResolution * gridResolution;
}

__global__ void kernComputeIndices(int N, int gridResolution,
  glm::vec3 gridMin, float inverseCellWidth,
  glm::vec3 *pos, int *indices, int *gridIndices) {
    // TODO-2.1
    // - Label each boid with the index of its grid cell.
    // - Set up a parallel array of integer indices as pointers to the actual
    //   boid data in pos and vel1/vel2
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index < N) {
        indices[index] = index;
        glm::vec3 gridPos = (pos[index] - gridMin) * inverseCellWidth;
        gridIndices[index] = gridIndex3Dto1D(gridPos.x, gridPos.y, gridPos.z, gridResolution);
    }
}

// LOOK-2.1 Consider how this could be useful for indicating that a cell
//          does not enclose any boids
__global__ void kernResetIntBuffer(int N, int *intBuffer, int value) {
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (index < N) {
    intBuffer[index] = value;
  }
}

__global__ void kernIdentifyCellStartEnd(int N, int *particleGridIndices,
  int *gridCellStartIndices, int *gridCellEndIndices) {
  // TODO-2.1
  // Identify the start point of each cell in the gridIndices array.
  // This is basically a parallel unrolling of a loop that goes
  // "this index doesn't match the one before it, must be a new cell!"
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index < N) {
        int gridIndex = particleGridIndices[index];
        if (gridCellStartIndices[gridIndex] < 0) {
            gridCellStartIndices[gridIndex] = index;
        }
        else if(gridCellStartIndices[gridIndex] > index) {
            gridCellStartIndices[gridIndex] = index;
        }
        if (gridCellEndIndices[gridIndex] < 0) {
            gridCellEndIndices[gridIndex] = index;
        }
        else if (gridCellEndIndices[gridIndex] < index) {
            gridCellEndIndices[gridIndex] = index;
        }
    }
}

__global__ void kernUpdateVelNeighborSearchScattered(
  int N, int gridResolution, glm::vec3 gridMin,
  float inverseCellWidth, float cellWidth,
  int *gridCellStartIndices, int *gridCellEndIndices,
  int *particleArrayIndices,
  glm::vec3 *pos, glm::vec3 *vel1, glm::vec3 *vel2) {
  // TODO-2.1 - Update a boid's velocity using the uniform grid to reduce
  // the number of boids that need to be checked.
  // - Identify the grid cell that this particle is in
  // - Identify which cells may contain neighbors. This isn't always 8.
  // - For each cell, read the start/end indices in the boid pointer array.
  // - Access each boid in the cell and compute velocity change from
  //   the boids rules, if this boid is within the neighborhood distance.
  // - Clamp the speed change before putting the new speed in vel2
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index < N) {
        glm::vec3 res(0.0f);
        glm::vec3 perceived_center(0);
        glm::vec3 c(0);
        glm::vec3 perceived_velocity(0);
        int numOfNeighbors1 = 0;
        int numOfNeighbors2 = 0;

        glm::vec3 position = pos[index];
        glm::vec3 gridIndex = (position - gridMin) * inverseCellWidth;
        glm::vec3 gridStart = gridIndex - glm::vec3(0.5f);
        glm::vec3 gridEnd = gridIndex + glm::vec3(0.5f);
        for (int x = imax(gridStart.x, 0); x <= imin(gridEnd.x, gridResolution - 1); x++) {
            for (int y = imax(gridStart.y, 0); y <= imin(gridEnd.y, gridResolution - 1); y++) {
                for (int z = imax(gridStart.z, 0); z <= imin(gridEnd.z, gridResolution - 1); z++) {
                    int gridId = gridIndex3Dto1D(x, y, z, gridResolution);
                    if (gridCellStartIndices[gridId] < 0) {
                        continue;
                    }
                    for (int i = gridCellStartIndices[gridId]; i <= gridCellEndIndices[gridId]; i++) {
                        int boidIndex = particleArrayIndices[i];
                        float distance = glm::distance(pos[boidIndex], pos[index]);
                        if (boidIndex != index && distance < rule1Distance) {
                            perceived_center += pos[boidIndex];
                            numOfNeighbors1++;
                        }
                        if (boidIndex != index && distance < rule2Distance) {
                            c -= (pos[boidIndex] - pos[index]);
                        }
                        if (boidIndex != index && distance < rule3Distance) {
                            perceived_velocity += vel1[boidIndex];
                            numOfNeighbors2++;
                        }
                    }
                }
            }
        }
        if (numOfNeighbors1 > 0) {
            perceived_center /= numOfNeighbors1;
            res += (perceived_center - pos[index]) * rule1Scale;
        }
        if (numOfNeighbors2 > 0) {
            perceived_velocity /= numOfNeighbors2;
            res += perceived_velocity * rule3Scale;
        }
        res += c * rule2Scale;
        glm::vec3 velocity = res + vel1[index];
        float len = glm::length(velocity);
        if (len > maxSpeed) {
            vel2[index] = velocity / len * maxSpeed;
        }else {
            vel2[index] = velocity;
        }
    }
}

__global__ void kernUpdateVelNeighborSearchCoherent(
  int N, int gridResolution, glm::vec3 gridMin,
  float inverseCellWidth, float cellWidth,
  int *gridCellStartIndices, int *gridCellEndIndices,
  glm::vec3 *pos, glm::vec3 *vel1, glm::vec3 *vel2) {
  // TODO-2.3 - This should be very similar to kernUpdateVelNeighborSearchScattered,
  // except with one less level of indirection.
  // This should expect gridCellStartIndices and gridCellEndIndices to refer
  // directly to pos and vel1.
  // - Identify the grid cell that this particle is in
  // - Identify which cells may contain neighbors. This isn't always 8.
  // - For each cell, read the start/end indices in the boid pointer array.
  //   DIFFERENCE: For best results, consider what order the cells should be
  //   checked in to maximize the memory benefits of reordering the boids data.
  // - Access each boid in the cell and compute velocity change from
  //   the boids rules, if this boid is within the neighborhood distance.
  // - Clamp the speed change before putting the new speed in vel2
}

/**
* Step the entire N-body simulation by `dt` seconds.
*/
void Boids::stepSimulationNaive(float dt) {
  // TODO-1.2 - use the kernels you wrote to step the simulation forward in time.
  // TODO-1.2 ping-pong the velocity buffers
    dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);
    kernUpdateVelocityBruteForce<<<fullBlocksPerGrid, blockSize>>>(numObjects, dev_pos, dev_vel1, dev_vel2);
    checkCUDAErrorWithLine("kernUpdateVelocityBruteForce Failed");
    kernUpdatePos <<<fullBlocksPerGrid, blockSize >>> (numObjects, dt, dev_pos, dev_vel2);
    checkCUDAErrorWithLine("kernUpdatePos Failed");
    std::swap(dev_vel1, dev_vel2);
}

void Boids::stepSimulationScatteredGrid(float dt) {
  // TODO-2.1
  // Uniform Grid Neighbor search using Thrust sort.
  // In Parallel:
  // - label each particle with its array index as well as its grid index.
  //   Use 2x width grids.
  // - Unstable key sort using Thrust. A stable sort isn't necessary, but you
  //   are welcome to do a performance comparison.
  // - Naively unroll the loop for finding the start and end indices of each
  //   cell's data pointers in the array of boid indices
  // - Perform velocity updates using neighbor search
  // - Update positions
  // - Ping-pong buffers as needed
    dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);
    dim3 fullCellPerGrid((gridCellCount + blockSize - 1) / blockSize);

    //compute indices
    kernComputeIndices <<<fullBlocksPerGrid, blockSize >>> (numObjects, gridSideCount, gridMinimum, gridInverseCellWidth, dev_pos, dev_particleArrayIndices, dev_particleGridIndices);
    checkCUDAErrorWithLine("kernComputeIndices Failed");
    //sort and copy
    thrust::device_ptr<int> dev_thrust_keys(dev_particleGridIndices);
    thrust::device_ptr<int> dev_thrust_values(dev_particleArrayIndices);
    thrust::sort_by_key(dev_thrust_keys, dev_thrust_keys + numObjects, dev_thrust_values);

    kernResetIntBuffer<<<fullCellPerGrid, blockSize>>>(gridCellCount, dev_gridCellStartIndices, -1);
    checkCUDAErrorWithLine("kernResetIntBuffer on Start Failed");
    kernResetIntBuffer<<<fullCellPerGrid, blockSize >>> (gridCellCount, dev_gridCellEndIndices, -1);
    checkCUDAErrorWithLine("kernResetIntBuffer on End Failed");
    kernIdentifyCellStartEnd<<<fullBlocksPerGrid, blockSize >>> (numObjects, dev_particleGridIndices, dev_gridCellStartIndices, dev_gridCellEndIndices);
    checkCUDAErrorWithLine("kernIdentifyCellStartEnd Failed");

    kernUpdateVelNeighborSearchScattered <<<fullBlocksPerGrid, blockSize >>> (numObjects, gridSideCount, gridMinimum, gridInverseCellWidth, gridCellWidth, dev_gridCellStartIndices, dev_gridCellEndIndices, dev_particleArrayIndices, dev_pos, dev_vel1, dev_vel2);
    checkCUDAErrorWithLine("kernUpdateVelNeighborSearchScattered Failed");

    kernUpdatePos<<<fullBlocksPerGrid, blockSize>>> (numObjects, dt, dev_pos, dev_vel2);
    checkCUDAErrorWithLine("kernUpdatePos Failed");
    std::swap(dev_vel1, dev_vel2);
}

void Boids::stepSimulationCoherentGrid(float dt) {
  // TODO-2.3 - start by copying Boids::stepSimulationNaiveGrid
  // Uniform Grid Neighbor search using Thrust sort on cell-coherent data.
  // In Parallel:
  // - Label each particle with its array index as well as its grid index.
  //   Use 2x width grids
  // - Unstable key sort using Thrust. A stable sort isn't necessary, but you
  //   are welcome to do a performance comparison.
  // - Naively unroll the loop for finding the start and end indices of each
  //   cell's data pointers in the array of boid indices
  // - BIG DIFFERENCE: use the rearranged array index buffer to reshuffle all
  //   the particle data in the simulation array.
  //   CONSIDER WHAT ADDITIONAL BUFFERS YOU NEED
  // - Perform velocity updates using neighbor search
  // - Update positions
  // - Ping-pong buffers as needed. THIS MAY BE DIFFERENT FROM BEFORE.
}

void Boids::endSimulation() {
  hipFree(dev_vel1);
  hipFree(dev_vel2);
  hipFree(dev_pos);

  // TODO-2.1 TODO-2.3 - Free any additional buffers here.
  hipFree(dev_particleArrayIndices);
  hipFree(dev_particleGridIndices);
  hipFree(dev_gridCellStartIndices);
  hipFree(dev_gridCellEndIndices);
}

void Boids::unitTest() {
  // LOOK-1.2 Feel free to write additional tests here.

  // test unstable sort
  int *dev_intKeys;
  int *dev_intValues;
  int N = 10;

  std::unique_ptr<int[]>intKeys{ new int[N] };
  std::unique_ptr<int[]>intValues{ new int[N] };

  intKeys[0] = 0; intValues[0] = 0;
  intKeys[1] = 1; intValues[1] = 1;
  intKeys[2] = 0; intValues[2] = 2;
  intKeys[3] = 3; intValues[3] = 3;
  intKeys[4] = 0; intValues[4] = 4;
  intKeys[5] = 2; intValues[5] = 5;
  intKeys[6] = 2; intValues[6] = 6;
  intKeys[7] = 0; intValues[7] = 7;
  intKeys[8] = 5; intValues[8] = 8;
  intKeys[9] = 6; intValues[9] = 9;

  hipMalloc((void**)&dev_intKeys, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_intKeys failed!");

  hipMalloc((void**)&dev_intValues, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_intValues failed!");

  dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);

  std::cout << "before unstable sort: " << std::endl;
  for (int i = 0; i < N; i++) {
    std::cout << "  key: " << intKeys[i];
    std::cout << " value: " << intValues[i] << std::endl;
  }
  // How to copy data to the GPU
  hipMemcpy(dev_intKeys, intKeys.get(), sizeof(int) * N, hipMemcpyHostToDevice);
  hipMemcpy(dev_intValues, intValues.get(), sizeof(int) * N, hipMemcpyHostToDevice);

  // Wrap device vectors in thrust iterators for use with thrust.
  thrust::device_ptr<int> dev_thrust_keys(dev_intKeys);
  thrust::device_ptr<int> dev_thrust_values(dev_intValues);
  // LOOK-2.1 Example for using thrust::sort_by_key
  thrust::sort_by_key(dev_thrust_keys, dev_thrust_keys + N, dev_thrust_values);

  // How to copy data back to the CPU side from the GPU
  hipMemcpy(intKeys.get(), dev_intKeys, sizeof(int) * N, hipMemcpyDeviceToHost);
  hipMemcpy(intValues.get(), dev_intValues, sizeof(int) * N, hipMemcpyDeviceToHost);
  checkCUDAErrorWithLine("memcpy back failed!");

  std::cout << "after unstable sort: " << std::endl;
  for (int i = 0; i < N; i++) {
    std::cout << "  key: " << intKeys[i];
    std::cout << " value: " << intValues[i] << std::endl;
  }

  // cleanup
  hipFree(dev_intKeys);
  hipFree(dev_intValues);
  checkCUDAErrorWithLine("hipFree failed!");
  return;
}
