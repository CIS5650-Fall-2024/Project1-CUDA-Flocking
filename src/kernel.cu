#include "hip/hip_runtime.h"
#define GLM_FORCE_CUDA
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <glm/glm.hpp>
#include "utilityCore.hpp"
#include "kernel.h"

// LOOK-2.1 potentially useful for doing grid-based neighbor search
#ifndef imax
#define imax( a, b ) ( ((a) > (b)) ? (a) : (b) )
#endif

#ifndef imin
#define imin( a, b ) ( ((a) < (b)) ? (a) : (b) )
#endif

#define checkCUDAErrorWithLine(msg) checkCUDAError(msg, __LINE__)

/**
* Check for CUDA errors; print and exit if there was a problem.
*/
void checkCUDAError(const char *msg, int line = -1) {
  hipError_t err = hipGetLastError();
  if (hipSuccess != err) {
    if (line >= 0) {
      fprintf(stderr, "Line %d: ", line);
    }
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}


/*****************
* Configuration *
*****************/

/*! Block size used for CUDA kernel launch. */
#define blockSize 128

// LOOK-1.2 Parameters for the boids algorithm.
// These worked well in our reference implementation.
#define rule1Distance 5.0f
#define rule2Distance 3.0f
#define rule3Distance 5.0f

#define rule1Scale 0.01f
#define rule2Scale 0.1f
#define rule3Scale 0.1f

#define maxSpeed 1.0f

/*! Size of the starting area in simulation space. */
#define scene_scale 100.0f

/***********************************************
* Kernel state (pointers are device pointers) *
***********************************************/

int numObjects;
dim3 threadsPerBlock(blockSize);

// LOOK-1.2 - These buffers are here to hold all your boid information.
// These get allocated for you in Boids::initSimulation.
// Consider why you would need two velocity buffers in a simulation where each
// boid cares about its neighbors' velocities.
// These are called ping-pong buffers.
glm::vec3 *dev_pos;
glm::vec3 *dev_vel1;
glm::vec3 *dev_vel2;

// LOOK-2.1 - these are NOT allocated for you. You'll have to set up the thrust
// pointers on your own too.

// For efficient sorting and the uniform grid. These should always be parallel.
int *dev_particleArrayIndices; // What index in dev_pos and dev_velX represents this particle?
int *dev_particleGridIndices; // What grid cell is this particle in?
// needed for use with thrust
thrust::device_ptr<int> dev_thrust_particleArrayIndices;
thrust::device_ptr<int> dev_thrust_particleGridIndices;

int *dev_gridCellStartIndices; // What part of dev_particleArrayIndices belongs
int *dev_gridCellEndIndices;   // to this cell?

// TODO-2.3 - consider what additional buffers you might need to reshuffle
// the position and velocity data to be coherent within cells.

glm::vec3 *extraBuffer;

// LOOK-2.1 - Grid parameters based on simulation parameters.
// These are automatically computed for you in Boids::initSimulation
int gridCellCount;
int gridSideCount;
float gridCellWidth;
float gridInverseCellWidth;
glm::vec3 gridMinimum;

/******************
* initSimulation *
******************/

__host__ __device__ unsigned int hash(unsigned int a) {
  a = (a + 0x7ed55d16) + (a << 12);
  a = (a ^ 0xc761c23c) ^ (a >> 19);
  a = (a + 0x165667b1) + (a << 5);
  a = (a + 0xd3a2646c) ^ (a << 9);
  a = (a + 0xfd7046c5) + (a << 3);
  a = (a ^ 0xb55a4f09) ^ (a >> 16);
  return a;
}

/**
* LOOK-1.2 - this is a typical helper function for a CUDA kernel.
* Function for generating a random vec3.
*/
__host__ __device__ glm::vec3 generateRandomVec3(float time, int index) {
  thrust::default_random_engine rng(hash((int)(index * time)));
  thrust::uniform_real_distribution<float> unitDistrib(-1, 1);

  return glm::vec3((float)unitDistrib(rng), (float)unitDistrib(rng), (float)unitDistrib(rng));
}

__device__ glm::vec3 operator-(glm::vec3 a, glm::vec3 b)
{
    return glm::vec3(a.x - b.x, a.y - b.y, a.z - b.z);
}

__device__ float dot(glm::vec3 a, glm::vec3 b)
{
    return a.x * b.x + a.y * b.y + a.z * b.z;
}

__device__ float distance(glm::vec3 v)
{
    return sqrtf(dot(v, v));
}

/**
* LOOK-1.2 - This is a basic CUDA kernel.
* CUDA kernel for generating boids with a specified mass randomly around the star.
*/
__global__ void kernGenerateRandomPosArray(int time, int N, glm::vec3 * arr, float scale) {
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (index < N) {
    glm::vec3 rand = generateRandomVec3(time, index);
    arr[index].x = scale * rand.x;
    arr[index].y = scale * rand.y;
    arr[index].z = scale * rand.z;
  }
}

/**
* Initialize memory, update some globals
*/
void Boids::initSimulation(int N) {
  numObjects = N;
  dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);

  // LOOK-1.2 - This is basic CUDA memory management and error checking.
  // Don't forget to hipFree in  Boids::endSimulation.
  hipMalloc((void**)&dev_pos, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_pos failed!");

  hipMalloc((void**)&dev_vel1, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_vel1 failed!");

  hipMalloc((void**)&dev_vel2, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_vel2 failed!");

  // LOOK-1.2 - This is a typical CUDA kernel invocation.
  kernGenerateRandomPosArray<<<fullBlocksPerGrid, blockSize>>>(1, numObjects,
    dev_pos, scene_scale);
  checkCUDAErrorWithLine("kernGenerateRandomPosArray failed!");

  // LOOK-2.1 computing grid params
  gridCellWidth = 2.0f * std::max(std::max(rule1Distance, rule2Distance), rule3Distance);
  int halfSideCount = (int)(scene_scale / gridCellWidth) + 1;
  gridSideCount = 2 * halfSideCount;

  gridCellCount = gridSideCount * gridSideCount * gridSideCount;
  gridInverseCellWidth = 1.0f / gridCellWidth;
  float halfGridWidth = gridCellWidth * halfSideCount;
  gridMinimum.x -= halfGridWidth;
  gridMinimum.y -= halfGridWidth;
  gridMinimum.z -= halfGridWidth;

  // TODO-2.1 TODO-2.3 - Allocate additional buffers here.

  hipMalloc((void**)&dev_particleArrayIndices, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_particleArrayIndices failed!");

  hipMalloc((void**)&dev_particleGridIndices, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_particleGridIndices failed!");

  hipMalloc((void**)&dev_gridCellStartIndices, gridCellCount * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_gridCellStartIndices failed!");

  hipMalloc((void**)&dev_gridCellEndIndices, gridCellCount * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_gridCellEndIndices failed!");

  hipMalloc((void**)&extraBuffer, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc extraBuffer failed!");

  hipDeviceSynchronize();
}


/******************
* copyBoidsToVBO *
******************/

/**
* Copy the boid positions into the VBO so that they can be drawn by OpenGL.
*/
__global__ void kernCopyPositionsToVBO(int N, glm::vec3 *pos, float *vbo, float s_scale) {
  int index = threadIdx.x + (blockIdx.x * blockDim.x);

  float c_scale = -1.0f / s_scale;

  if (index < N) {
    vbo[4 * index + 0] = pos[index].x * c_scale;
    vbo[4 * index + 1] = pos[index].y * c_scale;
    vbo[4 * index + 2] = pos[index].z * c_scale;
    vbo[4 * index + 3] = 1.0f;
  }
}

__global__ void kernCopyVelocitiesToVBO(int N, glm::vec3 *vel, float *vbo, float s_scale) {
  int index = threadIdx.x + (blockIdx.x * blockDim.x);

  if (index < N) {
    vbo[4 * index + 0] = vel[index].x + 0.3f;
    vbo[4 * index + 1] = vel[index].y + 0.3f;
    vbo[4 * index + 2] = vel[index].z + 0.3f;
    vbo[4 * index + 3] = 1.0f;
  }
}

/**
* Wrapper for call to the kernCopyboidsToVBO CUDA kernel.
*/
void Boids::copyBoidsToVBO(float *vbodptr_positions, float *vbodptr_velocities) {
  dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);

  kernCopyPositionsToVBO <<<fullBlocksPerGrid, blockSize >>>(numObjects, dev_pos, vbodptr_positions, scene_scale);
  kernCopyVelocitiesToVBO <<<fullBlocksPerGrid, blockSize >>>(numObjects, dev_vel1, vbodptr_velocities, scene_scale);

  checkCUDAErrorWithLine("copyBoidsToVBO failed!");

  hipDeviceSynchronize();
}


/******************
* stepSimulation *
******************/

/**
* LOOK-1.2 You can use this as a helper for kernUpdateVelocityBruteForce.
* __device__ code can be called from a __global__ context
* Compute the new velocity on the body with index `iSelf` due to the `N` boids
* in the `pos` and `vel` arrays.
*/
__device__ glm::vec3 computeVelocityChange(int N, int iSelf, const glm::vec3 *pos, const glm::vec3 *vel) {
  int index = threadIdx.x + (blockIdx.x * blockDim.x);
  if (index >= N) {
      return { 0,0,0 };
  }
  glm::vec3 v1 = { 0,0,0 };
  glm::vec3 v2 = { 0,0,0 };
  glm::vec3 v3 = { 0,0,0 };
  // Rule 1: boids fly towards their local perceived center of mass, which excludes themselves
  glm::vec3 perceived_center = { 0,0,0 };
  int neighbor_count_1 = 0;
  for (int i = 0; i < N; i++) {
      if (i != iSelf) {
          float d = distance(pos[iSelf] - pos[i]);
          //float d = glm::distance(pos[iSelf], pos[i]);
          if (d < rule1Distance) {
              perceived_center += pos[i];
              neighbor_count_1++;
          }
      }
  }
  if (neighbor_count_1 > 0) {
      perceived_center /= (float)neighbor_count_1;
      v1 = (perceived_center - pos[iSelf]) * rule1Scale;
  }

  // Rule 2: boids try to stay a distance d away from each other
  glm::vec3 c = { 0,0,0 };
  int neighbor_count_2 = 0;
  for (int i = 0; i < N; i++) {
      if (i != iSelf) {
          //float d = glm::distance(pos[iSelf], pos[i]);
          float d = distance(pos[iSelf] - pos[i]);
          if (d < rule2Distance) {
              c -= (pos[iSelf] - pos[i]);
              neighbor_count_2++;
          }
      }
  }
  if (neighbor_count_2 > 0) {
      v2 = c * rule2Scale;
  }
  // Rule 3: boids try to match the speed of surrounding boids
  int neighbor_count_3 = 0;
  glm::vec3 perceived_velocity = { 0, 0, 0 };
  for (int i = 0; i < N; i++) {
      if (i != iSelf) {
          float d = distance(pos[iSelf] - pos[i]);
          //float d = glm::distance(pos[iSelf], pos[i]);
          if (d < rule3Distance) {
              perceived_velocity += vel[i];
              neighbor_count_3++;
          }
      }
  }
  if (neighbor_count_3 > 0) {
      v3 = perceived_velocity * rule3Scale;
  }

  return vel[iSelf] + (v1 + v2 + v3);
}

/**
* TODO-1.2 implement basic flocking
* For each of the `N` bodies, update its position based on its current velocity.
*/
__global__ void kernUpdateVelocityBruteForce(int N, glm::vec3 *pos,
  glm::vec3 *vel1, glm::vec3 *vel2) {
  int index = threadIdx.x + (blockIdx.x * blockDim.x);
  if (index >= N) {
    return;
  }
  // Compute a new velocity based on pos and vel1
  glm::vec3 newVel = computeVelocityChange(N, index, pos, vel1);
  float newSpeed = glm::length(newVel);
  // Clamp the speed
  if (newSpeed > maxSpeed && newSpeed != 0) {
      newVel *= (maxSpeed / newSpeed);
  }
  // Record the new velocity into vel2. 
  vel2[index] = newVel;
}

/**
* LOOK-1.2 Since this is pretty trivial, we implemented it for you.
* For each of the `N` bodies, update its position based on its current velocity.
*/
__global__ void kernUpdatePos(int N, float dt, glm::vec3 *pos, glm::vec3 *vel) {
  // Update position by velocity
  int index = threadIdx.x + (blockIdx.x * blockDim.x);
  if (index >= N) {
    return;
  }
  glm::vec3 thisPos = pos[index];
  thisPos += vel[index] * dt;

  // Wrap the boids around so we don't lose them - i.e. out-of-bounds check
  thisPos.x = thisPos.x < -scene_scale ? scene_scale : thisPos.x;
  thisPos.y = thisPos.y < -scene_scale ? scene_scale : thisPos.y;
  thisPos.z = thisPos.z < -scene_scale ? scene_scale : thisPos.z;

  thisPos.x = thisPos.x > scene_scale ? -scene_scale : thisPos.x;
  thisPos.y = thisPos.y > scene_scale ? -scene_scale : thisPos.y;
  thisPos.z = thisPos.z > scene_scale ? -scene_scale : thisPos.z;

  pos[index] = thisPos;
}

// LOOK-2.1 Consider this method of computing a 1D index from a 3D grid index.
// LOOK-2.3 Looking at this method, what would be the most memory efficient
//          order for iterating over neighboring grid cells?
//          for(x)
//            for(y)
//             for(z)? Or some other order?
__device__ int gridIndex3Dto1D(int x, int y, int z, int gridResolution) {
  return x + y * gridResolution + z * gridResolution * gridResolution;
}

__global__ void kernComputeIndices(int N, int gridResolution,
  glm::vec3 gridMin, float inverseCellWidth,
  glm::vec3 *pos, int *indices, int *gridIndices) {
    int boid = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (boid >= N) return;
    // TODO-2.1
    // - Label each boid with the index of its grid cell.
    glm::vec3 curPos = pos[boid];
    glm::vec3 gridIndex3D = glm::floor((curPos - gridMin) * inverseCellWidth);
    int gridIndex = gridIndex3Dto1D(gridIndex3D.x, gridIndex3D.y, gridIndex3D.z, gridResolution);
    gridIndices[boid] = gridIndex;
    // - Set up a parallel array of integer indices as pointers to the actual
    //   boid data in pos and vel1/vel2
    indices[boid] = boid;
}

// LOOK-2.1 Consider how this could be useful for indicating that a cell
//          does not enclose any boids
__global__ void kernResetIntBuffer(int N, int *intBuffer, int value) {
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (index < N) {
    intBuffer[index] = value;
  }
}

__global__ void kernIdentifyCellStartEnd(int N, int *particleGridIndices,
  int *gridCellStartIndices, int *gridCellEndIndices) {
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= N) return;
  // TODO-2.1
  // Identify the start point of each cell in the gridIndices array.
  // This is basically a parallel unrolling of a loop that goes
  // "this index doesn't match the one before it, must be a new cell!"
    int gridIndex = particleGridIndices[index];
    //check if start
    if (gridCellStartIndices[gridIndex] == 0 || particleGridIndices[index - 1] != particleGridIndices[index]) {
        gridCellStartIndices[gridIndex] = index;
    }
    //check if end
    if (index == N - 1 || particleGridIndices[index + 1] != particleGridIndices[index]) {
        gridCellEndIndices[gridIndex] = index;
    }
}

__global__ void kernUpdateVelNeighborSearchScattered(
  int N, int gridResolution, glm::vec3 gridMin,
  float inverseCellWidth, float cellWidth,
  int *gridCellStartIndices, int *gridCellEndIndices,
  int *particleArrayIndices,
  glm::vec3 *pos, glm::vec3 *vel1, glm::vec3 *vel2) {
    int selfBoid = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (selfBoid >= N) return;

    // Behavior Params
    glm::vec3 v1 = { 0,0,0 };
    glm::vec3 v2 = { 0,0,0 };
    glm::vec3 v3 = { 0,0,0 };
    glm::vec3 perceived_center   = { 0,0,0 };
    glm::vec3 c                  = { 0,0,0 };
    glm::vec3 perceived_velocity = { 0,0,0 };
    int neighbor_count_1 = 0;
    int neighbor_count_3 = 0;

    // Find Bounds
    glm::vec3 curPos = pos[selfBoid];
    glm::vec3 gridIndex3D = glm::floor((curPos - gridMin) * inverseCellWidth);
    glm::vec3 startCell = gridIndex3D - glm::vec3(0.5f);
    glm::vec3 endCell = gridIndex3D + glm::vec3(0.5f);
    float minCell = 0;
    float maxCell = gridResolution - 1.f;
    //CellMin is 0,0,0, and max is (gridResolution-1.f, gridResolution-1.f, gridResolution-1.f)

    // Iterate
    for (int z = glm::max(startCell.z, minCell); z < glm::min(endCell.z, maxCell) + 1; z++) {
        for (int y = glm::max(startCell.y, minCell); y < glm::min(endCell.y, maxCell) + 1; y++) {
            for (int x = glm::max(startCell.x, minCell); x < glm::min(endCell.x, maxCell) + 1; x++) {
                int curGridIndex = gridIndex3Dto1D(x, y, z, gridResolution);
                if (gridCellStartIndices[curGridIndex] < 0) {
                    continue;
                }
                int startIndex = gridCellStartIndices[curGridIndex];
                int endIndex = gridCellEndIndices[curGridIndex];
                for (int j = startIndex; j <= endIndex; j++) {
                    int curBoid = particleArrayIndices[j];

                    // Rule 1: boids fly towards their local perceived center of mass, which excludes themselves
                    if (curBoid != selfBoid) {
                        float d = distance(pos[selfBoid] - pos[curBoid]);
                        if (d < rule1Distance) {
                            perceived_center += pos[curBoid];
                            neighbor_count_1++;
                        }
                    }

                    // Rule 2: boids try to stay a distance d away from each other
                    if (curBoid != selfBoid) {
                        float d = distance(pos[selfBoid] - pos[curBoid]);
                        if (d < rule2Distance) {
                            c -= (pos[selfBoid] - pos[curBoid]);
                        }
                    }

                    // Rule 3: boids try to match the speed of surrounding boids
                    if (curBoid != selfBoid) {
                        float d = distance(pos[selfBoid] - pos[curBoid]);
                        if (d < rule3Distance) {
                            perceived_velocity += vel1[curBoid];
                            neighbor_count_3++;
                        }
                    }
                }
            }
        }
    }

    // Rule 1
    if (neighbor_count_1 > 0) {
        perceived_center /= neighbor_count_1;
        v1 = (perceived_center - pos[selfBoid]) * rule1Scale;
    }

    // Rule 2
    {
        v2 = c * rule2Scale;
    }

    // Rule 3
    if (neighbor_count_3 > 0) {
        v3 = perceived_velocity * rule3Scale;
    }

    // Clamping
    glm::vec3 newVel = vel1[selfBoid] + (v1 + v2 + v3);
    float newSpeed = glm::length(newVel);
    if (newSpeed > maxSpeed && newSpeed != 0) {
        newVel *= (maxSpeed / newSpeed);
    }

    // Finish
    vel2[selfBoid] = newVel;
}

__global__ void kernUpdateVelNeighborSearchCoherent(
  int N, int gridResolution, glm::vec3 gridMin,
  float inverseCellWidth, float cellWidth,
  int *gridCellStartIndices, int *gridCellEndIndices,
  glm::vec3 *pos, glm::vec3 *vel1, glm::vec3 *vel2) {
  // TODO-2.3 - This should be very similar to kernUpdateVelNeighborSearchScattered,
  // except with one less level of indirection.
  // This should expect gridCellStartIndices and gridCellEndIndices to refer
  // directly to pos and vel1.
  // - Identify the grid cell that this particle is in
  // - Identify which cells may contain neighbors. This isn't always 8.
  // - For each cell, read the start/end indices in the boid pointer array.
  //   DIFFERENCE: For best results, consider what order the cells should be
  //   checked in to maximize the memory benefits of reordering the boids data.
  // - Access each boid in the cell and compute velocity change from
  //   the boids rules, if this boid is within the neighborhood distance.
  // - Clamp the speed change before putting the new speed in vel2


    int selfBoid = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (selfBoid >= N) return;

    // Behavior Params
    glm::vec3 v1 = { 0,0,0 };
    glm::vec3 v2 = { 0,0,0 };
    glm::vec3 v3 = { 0,0,0 };
    glm::vec3 perceived_center = { 0,0,0 };
    glm::vec3 c = { 0,0,0 };
    glm::vec3 perceived_velocity = { 0,0,0 };
    int neighbor_count_1 = 0;
    int neighbor_count_3 = 0;

    // Find Bounds
    glm::vec3 curPos = pos[selfBoid];
    glm::vec3 gridIndex3D = glm::floor((curPos - gridMin) * inverseCellWidth);
    glm::vec3 startCell = gridIndex3D - glm::vec3(0.5f);
    glm::vec3 endCell = gridIndex3D + glm::vec3(0.5f);
    float minCell = 0;
    float maxCell = gridResolution - 1.f;
    //CellMin is 0,0,0, and max is (gridResolution-1.f, gridResolution-1.f, gridResolution-1.f)

    // Iterate
    for (int z = glm::max(startCell.z, minCell); z < glm::min(endCell.z, maxCell) + 1; z++) {
        for (int y = glm::max(startCell.y, minCell); y < glm::min(endCell.y, maxCell) + 1; y++) {
            for (int x = glm::max(startCell.x, minCell); x < glm::min(endCell.x, maxCell) + 1; x++) {
                int curGridIndex = gridIndex3Dto1D(x, y, z, gridResolution);
                if (gridCellStartIndices[curGridIndex] < 0) {
                    continue;
                }
                int startIndex = gridCellStartIndices[curGridIndex];
                int endIndex = gridCellEndIndices[curGridIndex];
                for (int curBoid = startIndex; curBoid <= endIndex; curBoid++) {

                    // Rule 1: boids fly towards their local perceived center of mass, which excludes themselves
                    if (curBoid != selfBoid) {
                        float d = distance(pos[selfBoid] - pos[curBoid]);
                        if (d < rule1Distance) {
                            perceived_center += pos[curBoid];
                            neighbor_count_1++;
                        }
                    }

                    // Rule 2: boids try to stay a distance d away from each other
                    if (curBoid != selfBoid) {
                        float d = distance(pos[selfBoid] - pos[curBoid]);
                        if (d < rule2Distance) {
                            c -= (pos[selfBoid] - pos[curBoid]);
                        }
                    }

                    // Rule 3: boids try to match the speed of surrounding boids
                    if (curBoid != selfBoid) {
                        float d = distance(pos[selfBoid] - pos[curBoid]);
                        if (d < rule3Distance) {
                            perceived_velocity += vel1[curBoid];
                            neighbor_count_3++;
                        }
                    }
                }
            }
        }
    }

    // Rule 1
    if (neighbor_count_1 > 0) {
        perceived_center /= neighbor_count_1;
        v1 = (perceived_center - pos[selfBoid]) * rule1Scale;
    }

    // Rule 2
    {
        v2 = c * rule2Scale;
    }

    // Rule 3
    if (neighbor_count_3 > 0) {
        v3 = perceived_velocity * rule3Scale;
    }

    // Clamping
    glm::vec3 newVel = vel1[selfBoid] + (v1 + v2 + v3);
    float newSpeed = glm::length(newVel);
    if (newSpeed > maxSpeed && newSpeed != 0) {
        newVel *= (maxSpeed / newSpeed);
    }

    // Finish
    vel2[selfBoid] = newVel;

}

__global__ void reshuffle(int N, int* keys, glm::vec3* original, glm::vec3* extra) {
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= N) return;

    int boid = keys[index];
    extra[index] = original[boid];
}
/**
* Step the entire N-body simulation by `dt` seconds.
*/
void Boids::stepSimulationNaive(float dt) {
  // TODO-1.2 - use the kernels you wrote to step the simulation forward in time.
    dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);
    kernUpdateVelocityBruteForce <<<fullBlocksPerGrid, blockSize >>> (numObjects, dev_pos, dev_vel1, dev_vel2);
    kernUpdatePos<<<fullBlocksPerGrid, blockSize >>>(numObjects, dt, dev_pos, dev_vel2);
  // TODO-1.2 ping-pong the velocity buffers
    std::swap(dev_vel1, dev_vel2);
}

void Boids::stepSimulationScatteredGrid(float dt) {
    
    dim3 fullBoidBlocksPerGrid((numObjects + blockSize - 1) / blockSize);
// TODO-2.1
// Uniform Grid Neighbor search using Thrust sort.


// - label each particle with its array index as well as its grid index.
    kernComputeIndices<<<fullBoidBlocksPerGrid, blockSize>>>(numObjects, gridSideCount, gridMinimum, gridInverseCellWidth, dev_pos,
        dev_particleArrayIndices, dev_particleGridIndices);
    checkCUDAErrorWithLine("kernComputeIndices failed!");
// - Sorting
    thrust::device_ptr<int> dev_thrust_keys(dev_particleGridIndices);
    thrust::device_ptr<int> dev_thrust_values(dev_particleArrayIndices);
    //(key, key + N, value)
    thrust::sort_by_key(dev_thrust_keys, dev_thrust_keys + numObjects, dev_thrust_values);
    checkCUDAErrorWithLine("thrust sort failed!");
// - Reset start/end buffers for null boid cell check
    {
        dim3 fullCellBlocksPerGrid((gridCellCount + blockSize - 1) / blockSize);
        kernResetIntBuffer<<<fullCellBlocksPerGrid, blockSize >>>(gridCellCount, dev_gridCellStartIndices, -1);
        checkCUDAErrorWithLine("kernResetIntBuffer on StartIndices failed!");
    }

// - Naively unroll the loop for finding the start and end indices of each
//   cell's data pointers in the array of boid indices
    kernIdentifyCellStartEnd<<<fullBoidBlocksPerGrid, blockSize>>>(numObjects, dev_particleGridIndices, dev_gridCellStartIndices, dev_gridCellEndIndices);
    checkCUDAErrorWithLine("kernIdentifyCellStartEnd failed!");
    
// - Perform velocity updates using neighbor search
    kernUpdateVelNeighborSearchScattered<<<fullBoidBlocksPerGrid, blockSize>>>(numObjects, gridSideCount, gridMinimum, gridInverseCellWidth, gridCellWidth,
        dev_gridCellStartIndices, dev_gridCellEndIndices, dev_particleArrayIndices, 
        dev_pos, dev_vel1, dev_vel2);
    checkCUDAErrorWithLine("kernUpdateVelNeighborSearchScattered failed!");
// - Update positions
    kernUpdatePos<<<fullBoidBlocksPerGrid, blockSize >>>(numObjects, dt, dev_pos, dev_vel2);
// - Ping-pong buffers as needed
    std::swap(dev_vel1, dev_vel2);
    checkCUDAErrorWithLine("ping pong failed!");
}

void Boids::stepSimulationCoherentGrid(float dt) {
  // TODO-2.3 - start by copying Boids::stepSimulationNaiveGrid
  // Uniform Grid Neighbor search using Thrust sort on cell-coherent data.
  // In Parallel:
  // - Label each particle with its array index as well as its grid index.
  //   Use 2x width grids
  // - Unstable key sort using Thrust. A stable sort isn't necessary, but you
  //   are welcome to do a performance comparison.
  // - Naively unroll the loop for finding the start and end indices of each
  //   cell's data pointers in the array of boid indices
  // - BIG DIFFERENCE: use the rearranged array index buffer to reshuffle all
  //   the particle data in the simulation array.
  //   CONSIDER WHAT ADDITIONAL BUFFERS YOU NEED
  // - Perform velocity updates using neighbor search
  // - Update positions
  // - Ping-pong buffers as needed. THIS MAY BE DIFFERENT FROM BEFORE.

    dim3 fullBoidBlocksPerGrid((numObjects + blockSize - 1) / blockSize);
    // - label each particle with its array index as well as its grid index.
    kernComputeIndices << <fullBoidBlocksPerGrid, blockSize >> > (numObjects, gridSideCount, gridMinimum, gridInverseCellWidth, dev_pos,
        dev_particleArrayIndices, dev_particleGridIndices);
    checkCUDAErrorWithLine("kernComputeIndices failed!");
    // - Sorting
    thrust::device_ptr<int> dev_thrust_keys(dev_particleGridIndices);
    thrust::device_ptr<int> dev_thrust_values(dev_particleArrayIndices);
    //(key, key + N, value)
    thrust::sort_by_key(dev_thrust_keys, dev_thrust_keys + numObjects, dev_thrust_values);
    checkCUDAErrorWithLine("thrust sort failed!");
    // - Reset start/end buffers for null boid cell check
    {
        dim3 fullCellBlocksPerGrid((gridCellCount + blockSize - 1) / blockSize);
        kernResetIntBuffer << <fullCellBlocksPerGrid, blockSize >> > (gridCellCount, dev_gridCellStartIndices, -1);
        checkCUDAErrorWithLine("kernResetIntBuffer on StartIndices failed!");
    }

    // - Naively unroll the loop for finding the start and end indices of each
    //   cell's data pointers in the array of boid indices
    kernIdentifyCellStartEnd << <fullBoidBlocksPerGrid, blockSize >> > (numObjects, dev_particleGridIndices, dev_gridCellStartIndices, dev_gridCellEndIndices);
    checkCUDAErrorWithLine("kernIdentifyCellStartEnd failed!");

    {
        //RESHUFFLE!
        //Similar to vel2, use extraBuffer as temp storage so that the reshuffling doesn't have feedback
        reshuffle <<<fullBoidBlocksPerGrid, blockSize >> > (numObjects, dev_particleArrayIndices, dev_pos, extraBuffer);
        std::swap(dev_pos, extraBuffer);
        reshuffle <<<fullBoidBlocksPerGrid, blockSize >> > (numObjects, dev_particleArrayIndices, dev_vel1, extraBuffer);
        std::swap(dev_vel1, extraBuffer);
        reshuffle <<<fullBoidBlocksPerGrid, blockSize >> > (numObjects, dev_particleArrayIndices, dev_vel2, extraBuffer);
        std::swap(dev_vel2, extraBuffer);
        //As long as we shuffle all 3 buffers, together, it doesnt matter that we lost the original boid structuring.
        //Each buffer is still N elements long, and they are all still parallel. That is all that matters. ..[0] doesnt have to be the 0th boid in our
        //app because we dont care about specific ID boids ever.
    }

    // - Perform velocity updates using neighbor search
    kernUpdateVelNeighborSearchCoherent<<<fullBoidBlocksPerGrid, blockSize >> > (numObjects, gridSideCount, gridMinimum, gridInverseCellWidth, gridCellWidth,
        dev_gridCellStartIndices, dev_gridCellEndIndices,
        dev_pos, dev_vel1, dev_vel2);
    checkCUDAErrorWithLine("kernUpdateVelNeighborSearchScattered failed!");
    // - Update positions
    kernUpdatePos << <fullBoidBlocksPerGrid, blockSize >> > (numObjects, dt, dev_pos, dev_vel2);
    // - Ping-pong buffers as needed
    std::swap(dev_vel1, dev_vel2);
    checkCUDAErrorWithLine("ping pong failed!");
}

void Boids::endSimulation() {
  hipFree(dev_vel1);
  hipFree(dev_vel2);
  hipFree(dev_pos);

  // TODO-2.1 TODO-2.3 - Free any additional buffers here.
  hipFree(dev_particleArrayIndices);
  hipFree(dev_particleGridIndices);
  hipFree(dev_gridCellStartIndices);
  hipFree(dev_gridCellEndIndices);
  hipFree(extraBuffer);
}

void Boids::unitTest() {
  // LOOK-1.2 Feel free to write additional tests here.

  // test unstable sort
  int *dev_intKeys;
  int *dev_intValues;
  int N = 10;

  std::unique_ptr<int[]>intKeys{ new int[N] };
  std::unique_ptr<int[]>intValues{ new int[N] };

  intKeys[0] = 0; intValues[0] = 0;
  intKeys[1] = 1; intValues[1] = 1;
  intKeys[2] = 0; intValues[2] = 2;
  intKeys[3] = 3; intValues[3] = 3;
  intKeys[4] = 0; intValues[4] = 4;
  intKeys[5] = 2; intValues[5] = 5;
  intKeys[6] = 2; intValues[6] = 6;
  intKeys[7] = 0; intValues[7] = 7;
  intKeys[8] = 5; intValues[8] = 8;
  intKeys[9] = 6; intValues[9] = 9;

  hipMalloc((void**)&dev_intKeys, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_intKeys failed!");

  hipMalloc((void**)&dev_intValues, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_intValues failed!");

  dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);

  std::cout << "before unstable sort: " << std::endl;
  for (int i = 0; i < N; i++) {
    std::cout << "  key: " << intKeys[i];
    std::cout << " value: " << intValues[i] << std::endl;
  }

  // How to copy data to the GPU
  hipMemcpy(dev_intKeys, intKeys.get(), sizeof(int) * N, hipMemcpyHostToDevice);
  hipMemcpy(dev_intValues, intValues.get(), sizeof(int) * N, hipMemcpyHostToDevice);

  // Wrap device vectors in thrust iterators for use with thrust.
  thrust::device_ptr<int> dev_thrust_keys(dev_intKeys);
  thrust::device_ptr<int> dev_thrust_values(dev_intValues);
  // LOOK-2.1 Example for using thrust::sort_by_key
  thrust::sort_by_key(dev_thrust_keys, dev_thrust_keys + N, dev_thrust_values);

  // How to copy data back to the CPU side from the GPU
  hipMemcpy(intKeys.get(), dev_intKeys, sizeof(int) * N, hipMemcpyDeviceToHost);
  hipMemcpy(intValues.get(), dev_intValues, sizeof(int) * N, hipMemcpyDeviceToHost);
  checkCUDAErrorWithLine("memcpy back failed!");

  std::cout << "after unstable sort: " << std::endl;
  for (int i = 0; i < N; i++) {
    std::cout << "  key: " << intKeys[i];
    std::cout << " value: " << intValues[i] << std::endl;
  }

  // cleanup
  hipFree(dev_intKeys);
  hipFree(dev_intValues);
  checkCUDAErrorWithLine("hipFree failed!");
  return;
}
