#include "hip/hip_runtime.h"
#define GLM_FORCE_CUDA
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <glm/glm.hpp>
#include "utilityCore.hpp"
#include "kernel.h"
#include ""

// LOOK-2.1 potentially useful for doing grid-based neighbor search
#ifndef imax
#define imax( a, b ) ( ((a) > (b)) ? (a) : (b) )
#endif

#ifndef imin
#define imin( a, b ) ( ((a) < (b)) ? (a) : (b) )
#endif

#define checkCUDAErrorWithLine(msg) checkCUDAError(msg, __LINE__)

/**
* Check for CUDA errors; print and exit if there was a problem.
*/
void checkCUDAError(const char* msg, int line = -1) {
	hipError_t err = hipGetLastError();
	if (hipSuccess != err) {
		if (line >= 0) {
			fprintf(stderr, "Line %d: ", line);
		}
		fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}


/*****************
* Configuration *
*****************/

/*! Block size used for CUDA kernel launch. */
#define blockSize 128

// LOOK-1.2 Parameters for the boids algorithm.
// These worked well in our reference implementation.
#define rule1Distance 5.0f
#define rule2Distance 3.0f
#define rule3Distance 5.0f

#define rule1Scale 0.01f
#define rule2Scale 0.1f
#define rule3Scale 0.1f

#define maxSpeed 1.0f

/*! Size of the starting area in simulation space. */
#define scene_scale 100.0f

/***********************************************
* Kernel state (pointers are device pointers) *
***********************************************/

unsigned int numObjects;
dim3 threadsPerBlock(blockSize);

// LOOK-1.2 - These buffers are here to hold all your boid information.
// These get allocated for you in Boids::initSimulation.
// Consider why you would need two velocity buffers in a simulation where each
// boid cares about its neighbors' velocities.
// These are called ping-pong buffers.
glm::vec3* dev_pos;
glm::vec3* dev_vel1;
glm::vec3* dev_vel2;

// LOOK-2.1 - these are NOT allocated for you. You'll have to set up the thrust
// pointers on your own too.

// For efficient sorting and the uniform grid. These should always be parallel.
int* dev_particleArrayIndices; // What index in dev_pos and dev_velX represents this particle?
int* dev_particleGridIndices; // What grid cell is this particle in?
// needed for use with thrust
thrust::device_ptr<int> dev_thrust_particleArrayIndices;
thrust::device_ptr<int> dev_thrust_particleGridIndices;

int* dev_gridCellStartIndices; // What part of dev_particleArrayIndices belongs
int* dev_gridCellEndIndices;   // to this cell?

// TODO-2.3 - consider what additional buffers you might need to reshuffle
// the position and velocity data to be coherent within cells.

// LOOK-2.1 - Grid parameters based on simulation parameters.
// These are automatically computed for you in Boids::initSimulation
int gridCellCount;
int gridSideCount;
float gridCellWidth;
float gridInverseCellWidth;
glm::vec3 gridMinimum;

/******************
* initSimulation *
******************/

__host__ __device__ unsigned int hash(unsigned int a) {
	a = (a + 0x7ed55d16) + (a << 12);
	a = (a ^ 0xc761c23c) ^ (a >> 19);
	a = (a + 0x165667b1) + (a << 5);
	a = (a + 0xd3a2646c) ^ (a << 9);
	a = (a + 0xfd7046c5) + (a << 3);
	a = (a ^ 0xb55a4f09) ^ (a >> 16);
	return a;
}

/**
* LOOK-1.2 - this is a typical helper function for a CUDA kernel.
* Function for generating a random vec3.
*/
__host__ __device__ glm::vec3 generateRandomVec3(float time, int index) {
	thrust::default_random_engine rng(hash((int)(index * time)));
	thrust::uniform_real_distribution<float> unitDistrib(-1, 1);

	return glm::vec3((float)unitDistrib(rng), (float)unitDistrib(rng), (float)unitDistrib(rng));
}

/**
* LOOK-1.2 - This is a basic CUDA kernel.
* CUDA kernel for generating boids with a specified mass randomly around the star.
*/
__global__ void kernGenerateRandomPosArray(int time, int N, glm::vec3* arr, float scale) {
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (index < N) {
		glm::vec3 rand = generateRandomVec3(time, index);
		arr[index].x = scale * rand.x;
		arr[index].y = scale * rand.y;
		arr[index].z = scale * rand.z;
	}
}

/**
* Initialize memory, update some globals
*/
void Boids::initSimulation(int N) {
	numObjects = N;
	dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);

	// LOOK-1.2 - This is basic CUDA memory management and error checking.
	// Don't forget to hipFree in  Boids::endSimulation.
	hipMalloc((void**)&dev_pos, N * sizeof(glm::vec3));
	checkCUDAErrorWithLine("hipMalloc dev_pos failed!");

	hipMalloc((void**)&dev_vel1, N * sizeof(glm::vec3));
	checkCUDAErrorWithLine("hipMalloc dev_vel1 failed!");

	hipMalloc((void**)&dev_vel2, N * sizeof(glm::vec3));
	checkCUDAErrorWithLine("hipMalloc dev_vel2 failed!");

	// LOOK-1.2 - This is a typical CUDA kernel invocation.
	kernGenerateRandomPosArray << <fullBlocksPerGrid, blockSize >> > (1, numObjects,
		dev_pos, scene_scale);
	checkCUDAErrorWithLine("kernGenerateRandomPosArray failed!");

	// LOOK-2.1 computing grid params
	gridCellWidth = 2.0f * std::max(std::max(rule1Distance, rule2Distance), rule3Distance);
	int halfSideCount = (int)(scene_scale / gridCellWidth) + 1;
	gridSideCount = 2 * halfSideCount;

	gridCellCount = gridSideCount * gridSideCount * gridSideCount;
	gridInverseCellWidth = 1.0f / gridCellWidth;
	float halfGridWidth = gridCellWidth * halfSideCount;
	gridMinimum.x -= halfGridWidth;
	gridMinimum.y -= halfGridWidth;
	gridMinimum.z -= halfGridWidth;

	// TODO-2.1 TODO-2.3 - Allocate additional buffers here.
	hipMalloc((void**)&dev_particleArrayIndices, N * sizeof(int));
	checkCUDAErrorWithLine("hipMalloc dev_particleArrayIndices failed!");

	hipMalloc((void**)&dev_particleGridIndices, N * sizeof(int));
	checkCUDAErrorWithLine("hipMalloc dev_particleGridIndices failed!");

	hipMalloc((void**)&dev_gridCellStartIndices, gridCellCount * sizeof(int));
	checkCUDAErrorWithLine("hipMalloc dev_gridCellStartIndices failed!");

	hipMalloc((void**)&dev_gridCellEndIndices, gridCellCount * sizeof(int));
	checkCUDAErrorWithLine("hipMalloc dev_gridCellEndIndices failed!");

	hipDeviceSynchronize();

	dev_thrust_particleArrayIndices = thrust::device_pointer_cast(dev_particleArrayIndices);
	dev_thrust_particleGridIndices = thrust::device_pointer_cast(dev_particleGridIndices);
}


/******************
* copyBoidsToVBO *
******************/

/**
* Copy the boid positions into the VBO so that they can be drawn by OpenGL.
*/
__global__ void kernCopyPositionsToVBO(int N, glm::vec3* pos, float* vbo, float s_scale) {
	int index = threadIdx.x + (blockIdx.x * blockDim.x);

	float c_scale = -1.0f / s_scale;

	if (index < N) {
		vbo[4 * index + 0] = pos[index].x * c_scale;
		vbo[4 * index + 1] = pos[index].y * c_scale;
		vbo[4 * index + 2] = pos[index].z * c_scale;
		vbo[4 * index + 3] = 1.0f;
	}
}

__global__ void kernCopyVelocitiesToVBO(int N, glm::vec3* vel, float* vbo, float s_scale) {
	int index = threadIdx.x + (blockIdx.x * blockDim.x);

	if (index < N) {
		vbo[4 * index + 0] = vel[index].x + 0.3f;
		vbo[4 * index + 1] = vel[index].y + 0.3f;
		vbo[4 * index + 2] = vel[index].z + 0.3f;
		vbo[4 * index + 3] = 1.0f;
	}
}

/**
* Wrapper for call to the kernCopyboidsToVBO CUDA kernel.
*/
void Boids::copyBoidsToVBO(float* vbodptr_positions, float* vbodptr_velocities) {
	dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);

	kernCopyPositionsToVBO<<<fullBlocksPerGrid, blockSize>>>(numObjects, dev_pos, vbodptr_positions, scene_scale);
	kernCopyVelocitiesToVBO<<<fullBlocksPerGrid, blockSize>>>(numObjects, dev_vel1, vbodptr_velocities, scene_scale);

	checkCUDAErrorWithLine("copyBoidsToVBO failed!");

	hipDeviceSynchronize();
}


/******************
* stepSimulation *
******************/
/**
* LOOK-1.2 You can use this as a helper for kernUpdateVelocityBruteForce.
* __device__ code can be called from a __global__ context
* Compute the new velocity on the body with index `iSelf` due to the `N` boids
* in the `pos` and `vel` arrays.
*/
__device__ glm::vec3 computeVelocityChange(int N, int iSelf, const glm::vec3* pos, const glm::vec3* vel) {
	glm::vec3 perceivedCenter  { 0.0f, 0.0f, 0.0f };
	glm::vec3 avoidingDirection{ 0.0f, 0.0f, 0.0f };
	glm::vec3 perceivedVelocity{ 0.0f, 0.0f, 0.0f };

	glm::vec3 position = pos[iSelf];

	int rule1Neighbours = 0;
	int rule3Neighbours = 0;

	for (int i = 0; i < N; i++) {
		// boid itself does not influence itself
		if (i == iSelf) continue;

		// compute distance to neighbour
		glm::vec3 distanceVector = pos[i] - position;
		float distance = glm::length(distanceVector);

		// Rule 1: boids fly towards their local perceived center of mass, which excludes themselves
		if (distance < rule1Distance) {
			perceivedCenter += pos[i];
			rule1Neighbours++;
		}

		// Rule 2: boids try to stay a distance d away from each other
		if (distance < rule2Distance) avoidingDirection -= distanceVector;

		// Rule 3: boids try to match the speed of surrounding boids
		if (distance < rule3Distance) {
			perceivedVelocity += vel[i];
			rule3Neighbours++;
		}
	}

	if (rule1Neighbours > 0) perceivedCenter /= rule1Neighbours;
	if (rule3Neighbours > 0) perceivedVelocity /= rule3Neighbours;

	// Return sum of velocity changes according to each rule
	return {
		(perceivedCenter - position) * rule1Scale +
		avoidingDirection * rule2Scale +
		perceivedVelocity * rule3Scale
	};
}

__device__ glm::vec3 clampVelocity(glm::vec3 velocity) {
	// Clamping the velocity
	float speed = glm::length(velocity);
	return (speed > maxSpeed) ? glm::normalize(velocity) * maxSpeed : velocity;
}

/**
* TODO-1.2 implement basic flocking
* For each of the `N` bodies, update its position based on its current velocity.
*/
__global__ void kernUpdateVelocityBruteForce(int N, glm::vec3* pos,
	glm::vec3* vel1, glm::vec3* vel2) {
	// Compute a new velocity based on pos and vel1
	// Clamp the speed
	// Record the new velocity into vel2. Question: why NOT vel1?
	int index = threadIdx.x + (blockIdx.x * blockDim.x);
	if (index >= N) return;
	
	glm::vec3 velocity = vel1[index];
	velocity += computeVelocityChange(N, index, pos, vel1);

	vel2[index] = clampVelocity(velocity);
}

/**
* LOOK-1.2 Since this is pretty trivial, we implemented it for you.
* For each of the `N` bodies, update its position based on its current velocity.
*/
__global__ void kernUpdatePos(int N, float dt, glm::vec3* pos, glm::vec3* vel) {
	// Update position by velocity
	int index = threadIdx.x + (blockIdx.x * blockDim.x);
	if (index >= N) {
		return;
	}
	glm::vec3 thisPos = pos[index];
	thisPos += vel[index] * dt;

	// Wrap the boids around so we don't lose them
	thisPos.x = thisPos.x < -scene_scale ? scene_scale : thisPos.x;
	thisPos.y = thisPos.y < -scene_scale ? scene_scale : thisPos.y;
	thisPos.z = thisPos.z < -scene_scale ? scene_scale : thisPos.z;

	thisPos.x = thisPos.x > scene_scale ? -scene_scale : thisPos.x;
	thisPos.y = thisPos.y > scene_scale ? -scene_scale : thisPos.y;
	thisPos.z = thisPos.z > scene_scale ? -scene_scale : thisPos.z;

	pos[index] = thisPos;
}

// LOOK-2.1 Consider this method of computing a 1D index from a 3D grid index.
// LOOK-2.3 Looking at this method, what would be the most memory efficient
//          order for iterating over neighboring grid cells?
//          for(x)
//            for(y)
//             for(z)? Or some other order?
__device__ int gridIndex3Dto1D(int x, int y, int z, int gridResolution) {
	return x + y * gridResolution + z * gridResolution * gridResolution;
}

__global__ void kernComputeIndices(int N, int gridResolution,
	glm::vec3 gridMin, float inverseCellWidth,
	glm::vec3* pos, int* indices, int* gridIndices) {
	// TODO-2.1
	// - Label each boid with the index of its grid cell.
	// - Set up a parallel array of integer indices as pointers to the actual
	//   boid data in pos and vel1/vel2
	int index = threadIdx.x + (blockIdx.x * blockDim.x);
	if (index >= N) return;

	indices[index] = index;

	glm::ivec3 idx3D = glm::floor((pos[index] - gridMin) * inverseCellWidth);
	gridIndices[index] = gridIndex3Dto1D(idx3D.x, idx3D.y, idx3D.z, gridResolution);
}

// LOOK-2.1 Consider how this could be useful for indicating that a cell
//          does not enclose any boids
__global__ void kernResetIntBuffer(int N, int* intBuffer, int value) {
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (index < N) {
		intBuffer[index] = value;
	}
}

__global__ void kernIdentifyCellStartEnd(int N, int* particleGridIndices,
	int* gridCellStartIndices, int* gridCellEndIndices) {
	// TODO-2.1
	// Identify the start point of each cell in the gridIndices array.
	// This is basically a parallel unrolling of a loop that goes
	// "this index doesn't match the one before it, must be a new cell!"
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (index >= N) return;
	
	if (index == 0) gridCellStartIndices[particleGridIndices[0]] = 0;
	if (index == N - 1) gridCellEndIndices[particleGridIndices[N - 1]] = N - 1;

	if (particleGridIndices[index] != particleGridIndices[index - 1]) {
		gridCellStartIndices[particleGridIndices[index]] = index;
		gridCellEndIndices[particleGridIndices[index - 1]] = index - 1;
	}
}

__global__ void kernUpdateVelNeighborSearchScattered(
	int N, int gridResolution, glm::vec3 gridMin,
	float inverseCellWidth, float cellWidth,
	int* gridCellStartIndices, int* gridCellEndIndices,
	int* particleArrayIndices,
	glm::vec3* pos, glm::vec3* vel1, glm::vec3* vel2) {
	// TODO-2.1 - Update a boid's velocity using the uniform grid to reduce
	// the number of boids that need to be checked.
	// - Identify the grid cell that this particle is in
	// - Identify which cells may contain neighbors. This isn't always 8.
	// - For each cell, read the start/end indices in the boid pointer array.
	// - Access each boid in the cell and compute velocity change from
	//   the boids rules, if this boid is within the neighborhood distance.
	// - Clamp the speed change before putting the new speed in vel2
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (index >= N) return;

	glm::vec3 velocity = vel1[index];
	glm::vec3 position = pos[index];

	glm::ivec3 idxCenter = glm::floor((pos[index] - gridMin) * inverseCellWidth);

	glm::vec3 gridCenter = (glm::vec3(idxCenter) + 0.5f) * cellWidth + gridMin;

	glm::bvec3 idxOffset = glm::greaterThan(pos[index], gridCenter);

	glm::vec3 perceivedCenter  { 0.0f, 0.0f, 0.0f };
	glm::vec3 perceivedVelocity{ 0.0f, 0.0f, 0.0f };
	glm::vec3 avoidingDirection{ 0.0f, 0.0f, 0.0f };
	int rule1Neighbours = 0;
	int rule3Neighbours = 0;

	glm::ivec3 localIdx;
	for (localIdx.z = idxOffset.z - 1; localIdx.z < idxOffset.z + 1; localIdx.z++) {
		for (localIdx.y = idxOffset.y - 1; localIdx.y < idxOffset.y + 1; localIdx.y++) {
			for (localIdx.x = idxOffset.x - 1; localIdx.x < idxOffset.x + 1; localIdx.x++) {
				// calculate x, y, z index for center and neighbouring cells
				glm::ivec3 globalIdx = idxCenter + localIdx;

				// skip calculation if cell index is out of grid range
				if (glm::any(glm::lessThan(globalIdx, glm::ivec3(0))) ||
					glm::any(glm::greaterThan(globalIdx, glm::ivec3(gridResolution)))) continue;

				// compute 1D index of cell
				int gridIndex = gridIndex3Dto1D(
					globalIdx.x, globalIdx.y, globalIdx.z, gridResolution
				);

				int gridStartIdx = gridCellStartIndices[gridIndex];
				int gridEndIdx = gridCellEndIndices[gridIndex];

				// grid cell does not contain any boids
				if (gridStartIdx == -1) continue;

				for (int* ptr = &particleArrayIndices[gridStartIdx]; ptr <= &particleArrayIndices[gridEndIdx]; ptr++) {
					int idx = *ptr;
					// boid itself does not influence itself
					if (idx == index) continue;

					// compute distance to neighbour
					glm::vec3 distanceVector = pos[idx] - position;
					float distance = glm::length(distanceVector);

					// Rule 1: boids fly towards their local perceived center of mass, which excludes themselves
					if (distance < rule1Distance) {
						perceivedCenter += pos[idx];
						rule1Neighbours++;
					}

					// Rule 2: boids try to stay a distance d away from each other
					if (distance < rule2Distance) avoidingDirection -= distanceVector;

					// Rule 3: boids try to match the speed of surrounding boids
					if (distance < rule3Distance) {
						perceivedVelocity += vel1[idx];
						rule3Neighbours++;
					}
				}
			}
		}
	}

	if (rule1Neighbours > 0) perceivedCenter /= rule1Neighbours;
	if (rule3Neighbours > 0) perceivedVelocity /= rule3Neighbours;

	// Update velocity with velocity changes according to each rule
	velocity += (perceivedCenter - position) * rule1Scale +
		avoidingDirection * rule2Scale +
		perceivedVelocity * rule3Scale;

	vel2[index] = clampVelocity(velocity);
}

__global__ void kernUpdateVelNeighborSearchCoherent(
	int N, int gridResolution, glm::vec3 gridMin,
	float inverseCellWidth, float cellWidth,
	int* gridCellStartIndices, int* gridCellEndIndices,
	glm::vec3* pos, glm::vec3* vel1, glm::vec3* vel2) {
	// TODO-2.3 - This should be very similar to kernUpdateVelNeighborSearchScattered,
	// except with one less level of indirection.
	// This should expect gridCellStartIndices and gridCellEndIndices to refer
	// directly to pos and vel1.
	// - Identify the grid cell that this particle is in
	// - Identify which cells may contain neighbors. This isn't always 8.
	// - For each cell, read the start/end indices in the boid pointer array.
	//   DIFFERENCE: For best results, consider what order the cells should be
	//   checked in to maximize the memory benefits of reordering the boids data.
	// - Access each boid in the cell and compute velocity change from
	//   the boids rules, if this boid is within the neighborhood distance.
	// - Clamp the speed change before putting the new speed in vel2
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (index >= N) return;

	glm::vec3 velocity = vel1[index];
	glm::vec3 position = pos[index];

	glm::ivec3 idxCenter = glm::floor((pos[index] - gridMin) * inverseCellWidth);

	glm::vec3 gridCenter = (glm::vec3(idxCenter) + 0.5f) * cellWidth + gridMin;

	glm::bvec3 idxOffset = glm::greaterThan(pos[index], gridCenter);

	glm::vec3 perceivedCenter  { 0.0f, 0.0f, 0.0f };
	glm::vec3 perceivedVelocity{ 0.0f, 0.0f, 0.0f };
	glm::vec3 avoidingDirection{ 0.0f, 0.0f, 0.0f };
	int rule1Neighbours = 0;
	int rule3Neighbours = 0;

	glm::ivec3 localIdx;
	for (localIdx.z = idxOffset.z - 1; localIdx.z < idxOffset.z + 1; localIdx.z++) {
		for (localIdx.y = idxOffset.y - 1; localIdx.y < idxOffset.y + 1; localIdx.y++) {
			for (localIdx.x = idxOffset.x - 1; localIdx.x < idxOffset.x + 1; localIdx.x++) {
				// calculate x, y, z index for center and neighbouring cells
				glm::ivec3 globalIdx = idxCenter + localIdx;

				// skip calculation if cell index is out of grid range
				if (glm::any(glm::lessThan(globalIdx, glm::ivec3(0))) ||
					glm::any(glm::greaterThan(globalIdx, glm::ivec3(gridResolution)))) continue;

				// compute 1D index of cell
				int gridIndex = gridIndex3Dto1D(
					globalIdx.x, globalIdx.y, globalIdx.z, gridResolution
				);

				int gridStartIdx = gridCellStartIndices[gridIndex];
				int gridEndIdx = gridCellEndIndices[gridIndex];

				// grid cell does not contain any boids
				if (gridStartIdx == -1) continue;

				for (int idx = gridStartIdx; idx <= gridEndIdx; idx++) {
					// boid itself does not influence itself
					if (idx == index) continue;

					// compute distance to neighbour
					glm::vec3 distanceVector = pos[idx] - position;
					float distance = glm::length(distanceVector);

					// Rule 1: boids fly towards their local perceived center of mass, which excludes themselves
					if (distance < rule1Distance) {
						perceivedCenter += pos[idx];
						rule1Neighbours++;
					}

					// Rule 2: boids try to stay a distance d away from each other
					if (distance < rule2Distance) avoidingDirection -= distanceVector;

					// Rule 3: boids try to match the speed of surrounding boids
					if (distance < rule3Distance) {
						perceivedVelocity += vel1[idx];
						rule3Neighbours++;
					}
				}
			}
		}
	}

	if (rule1Neighbours > 0) perceivedCenter /= rule1Neighbours;
	if (rule3Neighbours > 0) perceivedVelocity /= rule3Neighbours;

	// Update velocity with velocity changes according to each rule
	velocity += (perceivedCenter - position) * rule1Scale +
		avoidingDirection * rule2Scale +
		perceivedVelocity * rule3Scale;

	vel2[index] = clampVelocity(velocity);
}

template <typename T>
__global__ void kernReshuffleArray(int N, int* indices, T* shuffleArray, T* bufferArray) {
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (index >= N) return;

	bufferArray[index] = shuffleArray[indices[index]];
}

/**
* Step the entire N-body simulation by `dt` seconds.
*/
void Boids::stepSimulationNaive(float dt) {
	// TODO-1.2 - use the kernels you wrote to step the simulation forward in time.
	// TODO-1.2 ping-pong the velocity buffers
	dim3 fullBlocksPerGrid{ (numObjects + blockSize - 1) / blockSize };

	kernUpdateVelocityBruteForce<<<fullBlocksPerGrid, blockSize>>>(
		numObjects, dev_pos, dev_vel1, dev_vel2
	);
	checkCUDAErrorWithLine("kernUpdateVelocityBruteForce failed!");

	kernUpdatePos<<<fullBlocksPerGrid, blockSize>>>(numObjects, dt, dev_pos, dev_vel2);
	checkCUDAErrorWithLine("kernUpdatePos failed!");

	std::swap(dev_vel1, dev_vel2);
}

void Boids::stepSimulationScatteredGrid(float dt) {
	// TODO-2.1
	// Uniform Grid Neighbor search using Thrust sort.
	// In Parallel:
	// - label each particle with its array index as well as its grid index.
	//   Use 2x width grids.
	// - Unstable key sort using Thrust. A stable sort isn't necessary, but you
	//   are welcome to do a performance comparison.
	// - Naively unroll the loop for finding the start and end indices of each
	//   cell's data pointers in the array of boid indices
	// - Perform velocity updates using neighbor search
	// - Update positions
	// - Ping-pong buffers as needed
	dim3 fullBlocksPerGrid{ (numObjects + blockSize - 1) / blockSize };

	kernResetIntBuffer<<<fullBlocksPerGrid, blockSize>>>(numObjects, dev_gridCellStartIndices, -1);
	kernResetIntBuffer<<<fullBlocksPerGrid, blockSize>>>(numObjects, dev_gridCellEndIndices, -1);

	kernComputeIndices<<<fullBlocksPerGrid, blockSize>>>(
		numObjects, gridSideCount, gridMinimum, gridInverseCellWidth,
		dev_pos, dev_particleArrayIndices, dev_particleGridIndices
	);
	checkCUDAErrorWithLine("kernComputeIndices failed!");

	thrust::sort_by_key(dev_thrust_particleGridIndices, dev_thrust_particleGridIndices + numObjects,
		dev_thrust_particleArrayIndices);

	kernIdentifyCellStartEnd<<<fullBlocksPerGrid, blockSize>>>(gridCellCount, dev_particleGridIndices,
		dev_gridCellStartIndices, dev_gridCellEndIndices);
	checkCUDAErrorWithLine("kernIdentifyCellStartEnd failed!");

	kernUpdateVelNeighborSearchScattered<<<fullBlocksPerGrid, blockSize>>>(
		numObjects, gridSideCount, gridMinimum,
		gridInverseCellWidth, gridCellWidth,
		dev_gridCellStartIndices, dev_gridCellEndIndices,
		dev_particleArrayIndices,
		dev_pos, dev_vel1, dev_vel2);
	checkCUDAErrorWithLine("kernUpdateVelNeighborSearchScattered failed!");

	kernUpdatePos<<<fullBlocksPerGrid, blockSize>>>(numObjects, dt, dev_pos, dev_vel2);
	checkCUDAErrorWithLine("kernUpdatePos failed!");

	std::swap(dev_vel1, dev_vel2);
}

auto num_frames = 0;
auto total_time = 0.0f;
bool printed = 0;

void Boids::stepSimulationCoherentGrid(float dt) {
	// TODO-2.3 - start by copying Boids::stepSimulationNaiveGrid
	// Uniform Grid Neighbor search using Thrust sort on cell-coherent data.
	// In Parallel:
	// - Label each particle with its array index as well as its grid index.
	//   Use 2x width grids
	// - Unstable key sort using Thrust. A stable sort isn't necessary, but you
	//   are welcome to do a performance comparison.
	// - Naively unroll the loop for finding the start and end indices of each
	//   cell's data pointers in the array of boid indices
	// - BIG DIFFERENCE: use the rearranged array index buffer to reshuffle all
	//   the particle data in the simulation array.
	//   CONSIDER WHAT ADDITIONAL BUFFERS YOU NEED
	// - Perform velocity updates using neighbor search
	// - Update positions
	// - Ping-pong buffers as needed. THIS MAY BE DIFFERENT FROM BEFORE.

	dim3 fullBlocksPerGrid{ (numObjects + blockSize - 1) / blockSize };

	kernResetIntBuffer<<<fullBlocksPerGrid, blockSize>>>(numObjects, dev_gridCellStartIndices, -1);
	checkCUDAErrorWithLine("kernResetIntBuffer failed!");

	kernComputeIndices<<<fullBlocksPerGrid, blockSize>>>(
		numObjects, gridSideCount, gridMinimum, gridInverseCellWidth,
		dev_pos, dev_particleArrayIndices, dev_particleGridIndices
		);
	checkCUDAErrorWithLine("kernComputeIndices failed!");

	thrust::sort_by_key(dev_thrust_particleGridIndices, dev_thrust_particleGridIndices + numObjects,
		dev_thrust_particleArrayIndices);

	// Reordering dev_pos array, using dev_vel2 as buffer vector
	kernReshuffleArray<<<fullBlocksPerGrid, blockSize>>>(
		numObjects, dev_particleArrayIndices, dev_pos, dev_vel2
	);
	checkCUDAErrorWithLine("kernReshuffleArray(dev_pos) failed!");
	std::swap(dev_pos, dev_vel2);
	// Reordering dev_vel1 into dev_vel2
	kernReshuffleArray<<<fullBlocksPerGrid, blockSize>>>(
		numObjects, dev_particleArrayIndices, dev_vel1, dev_vel2
	);
	checkCUDAErrorWithLine("kernReshuffleArray(dev_vel1) failed!");

	kernIdentifyCellStartEnd<<<fullBlocksPerGrid, blockSize>>>(gridCellCount, dev_particleGridIndices,
		dev_gridCellStartIndices, dev_gridCellEndIndices);
	checkCUDAErrorWithLine("kernIdentifyCellStartEnd failed!");

	// 
	kernUpdateVelNeighborSearchCoherent<<<fullBlocksPerGrid, blockSize>>>(
		numObjects, gridSideCount, gridMinimum,
		gridInverseCellWidth, gridCellWidth,
		dev_gridCellStartIndices, dev_gridCellEndIndices,
		dev_pos, dev_vel2, dev_vel1);
	checkCUDAErrorWithLine("kernUpdateVelNeighborSearchCoherent failed!");

	kernUpdatePos<<<fullBlocksPerGrid, blockSize>>>(numObjects, dt, dev_pos, dev_vel1);
	checkCUDAErrorWithLine("kernUpdatePos failed!");

	// No additional swap necessary
}

void Boids::endSimulation() {
	hipFree(dev_vel1);
	hipFree(dev_vel2);
	hipFree(dev_pos);

	// TODO-2.1 TODO-2.3 - Free any additional buffers here.
	hipFree(dev_particleArrayIndices);
	hipFree(dev_particleGridIndices);
	hipFree(dev_gridCellStartIndices);
	hipFree(dev_gridCellEndIndices);
}

void Boids::unitTest() {
	// LOOK-1.2 Feel free to write additional tests here.

	// test unstable sort
	int* dev_intKeys;
	int* dev_intValues;
	int N = 10;

	std::unique_ptr<int[]>intKeys{ new int[N] };
	std::unique_ptr<int[]>intValues{ new int[N] };

	intKeys[0] = 0; intValues[0] = 0;
	intKeys[1] = 1; intValues[1] = 1;
	intKeys[2] = 0; intValues[2] = 2;
	intKeys[3] = 3; intValues[3] = 3;
	intKeys[4] = 0; intValues[4] = 4;
	intKeys[5] = 2; intValues[5] = 5;
	intKeys[6] = 2; intValues[6] = 6;
	intKeys[7] = 0; intValues[7] = 7;
	intKeys[8] = 5; intValues[8] = 8;
	intKeys[9] = 6; intValues[9] = 9;

	hipMalloc((void**)&dev_intKeys, N * sizeof(int));
	checkCUDAErrorWithLine("hipMalloc dev_intKeys failed!");

	hipMalloc((void**)&dev_intValues, N * sizeof(int));
	checkCUDAErrorWithLine("hipMalloc dev_intValues failed!");

	dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);

	std::cout << "before unstable sort: " << std::endl;
	for (int i = 0; i < N; i++) {
		std::cout << "  key: " << intKeys[i];
		std::cout << " value: " << intValues[i] << std::endl;
	}

	// How to copy data to the GPU
	hipMemcpy(dev_intKeys, intKeys.get(), sizeof(int) * N, hipMemcpyHostToDevice);
	hipMemcpy(dev_intValues, intValues.get(), sizeof(int) * N, hipMemcpyHostToDevice);

	// Wrap device vectors in thrust iterators for use with thrust.
	thrust::device_ptr<int> dev_thrust_keys(dev_intKeys);
	thrust::device_ptr<int> dev_thrust_values(dev_intValues);
	// LOOK-2.1 Example for using thrust::sort_by_key
	thrust::sort_by_key(dev_thrust_keys, dev_thrust_keys + N, dev_thrust_values);

	// How to copy data back to the CPU side from the GPU
	hipMemcpy(intKeys.get(), dev_intKeys, sizeof(int) * N, hipMemcpyDeviceToHost);
	hipMemcpy(intValues.get(), dev_intValues, sizeof(int) * N, hipMemcpyDeviceToHost);
	checkCUDAErrorWithLine("memcpy back failed!");

	std::cout << "after unstable sort: " << std::endl;
	for (int i = 0; i < N; i++) {
		std::cout << "  key: " << intKeys[i];
		std::cout << " value: " << intValues[i] << std::endl;
	}

	// cleanup
	hipFree(dev_intKeys);
	hipFree(dev_intValues);
	checkCUDAErrorWithLine("hipFree failed!");
	return;
}
